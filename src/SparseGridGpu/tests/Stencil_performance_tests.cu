#include "hip/hip_runtime.h"
//
// Created by tommaso on 4/07/19.
//

#define BOOST_TEST_DYN_LINK

#include <boost/test/unit_test.hpp>
#include "SparseGridGpu/SparseGridGpu.hpp"

// Error handling from https://stackoverflow.com/a/13245319
static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

template<unsigned int p, unsigned int chunksPerBlock=1, typename SparseGridType, typename ScalarT>
__global__ void insertConstantValue(SparseGridType sparseGrid, ScalarT value)
{
    constexpr unsigned int pMask = SparseGridType::pMask;
    typedef BlockTypeOf<typename SparseGridType::AggregateType, p> BlockT;
    typedef BlockTypeOf<typename SparseGridType::AggregateType, pMask> MaskBlockT;

    sparseGrid.init();

    __shared__ BlockT *blocks[chunksPerBlock];
    __shared__ MaskBlockT *masks[chunksPerBlock];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    auto pos = sparseGrid.getLinId(coord);
    unsigned int dataBlockId = pos / BlockT::size;
    unsigned int offset = pos % BlockT::size;
    unsigned int dataBlockNum = dataBlockId % chunksPerBlock;

    if (offset == 0) // Just one thread per data block
    {
        auto encap = sparseGrid.insertBlock(dataBlockId);
        blocks[dataBlockNum] = &(encap.template get<p>());
        masks[dataBlockNum] = &(encap.template get<pMask>());
    }

    __syncthreads();
    blocks[dataBlockNum]->block[offset] = value;
    BlockMapGpu_ker<>::setExist(masks[dataBlockNum]->block[offset]);

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType, typename ValueT>
__global__ void insertOneValue(SparseGridType sparseGrid, dim3 pt, ValueT value)
{
    sparseGrid.init();

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    dim3 thCoord(x, y, z);
    if (thCoord.x == pt.x && thCoord.y == pt.y && thCoord.z == pt.z)
    {
        grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});
        sparseGrid.template insert<p>(coord) = value;
    }
    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType, typename VectorOutType>
__global__ void copyBlocksToOutput(SparseGridType sparseGrid, VectorOutType output)
{
    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    size_t pos = sparseGrid.getLinId(coord);

    auto value = sparseGrid.template get<p>(coord);

//    printf("copyBlocksToOutput: bDim=(%d,%d), bId=(%d,%d), tId=(%d,%d) : "
//           "pos=%ld, coord={%d,%d}, value=%d\n",
//           bDimX, bDimY,
//           bIdX, bIdY,
//           tIdX, tIdY,
//           pos,
//           x, y,
//           static_cast<int>(value)); //debug

    output.template get<p>(pos) = value;

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int dim, unsigned int p>
struct HeatStencilOrig
{
    // This is an example of a laplacian smoothing stencil to apply using the apply stencil facility of SparseGridGpu

    static constexpr unsigned int flops = 3 + 2*dim;

    static constexpr unsigned int supportRadius = 1;

    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __device__ void stencil(
            SparseGridT & sparseGrid,
            grid_key_dx<dim> & dataBlockCoord,
            unsigned int offset,
            grid_key_dx<dim> & pointCoord,
            DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore,
            float dt)
    {
        typedef typename SparseGridT::AggregateBlockType AggregateT;
        typedef ScalarTypeOf<AggregateT, p> ScalarT;

        constexpr unsigned int enlargedBlockSize = IntPow<
                SparseGridT::getBlockEdgeSize() + 2 * supportRadius, dim>::value;

        __shared__ ScalarT enlargedBlock[enlargedBlockSize];
        sparseGrid.loadBlock<p>(dataBlockLoad, enlargedBlock);
        sparseGrid.loadGhost<p>(dataBlockCoord, enlargedBlock);
        __syncthreads();

        const auto coord = sparseGrid.getCoordInEnlargedBlock(offset);
        const auto linId = sparseGrid.getLinIdInEnlargedBlock(offset);
        ScalarT cur = enlargedBlock[linId];
        ScalarT laplacian = -2.0 * dim * cur; // The central part of the stencil
        for (int d = 0; d < dim; ++d)
        {
            auto nPlusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, 1);
            auto nMinusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, -1);
            ScalarT neighbourPlus = enlargedBlock[nPlusId];
            ScalarT neighbourMinus = enlargedBlock[nMinusId];
            laplacian += neighbourMinus + neighbourPlus;
        }
        enlargedBlock[linId] = cur + dt * laplacian;

        __syncthreads();
        sparseGrid.storeBlock<p>(dataBlockStore, enlargedBlock);
    }

    template <typename SparseGridT, typename CtxT>
    static inline void __host__ flush(SparseGridT & sparseGrid, CtxT & ctx)
    {
        sparseGrid.template flush <smin_<0>> (ctx, flush_type::FLUSH_ON_DEVICE);
    }
};

template<unsigned int dim, unsigned int p>
struct HeatStencil
{
    // This is an example of a laplacian smoothing stencil to apply using the apply stencil facility of SparseGridGpu

    static constexpr unsigned int flops = 3 + 2*dim;

    static constexpr unsigned int supportRadius = 1;

    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __device__ void stencil(
            SparseGridT & sparseGrid,
            grid_key_dx<dim> & dataBlockCoord,
            unsigned int offset,
            grid_key_dx<dim> & pointCoord,
            DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore,
            float dt, unsigned int maxIter=1000)
    {
        typedef typename SparseGridT::AggregateBlockType AggregateT;
        typedef ScalarTypeOf<AggregateT, p> ScalarT;
        typedef BlockTypeOf<AggregateT, p> BlockT;
        constexpr unsigned int blockSize = BlockT::size;

        constexpr unsigned int enlargedBlockSize = IntPow<
                SparseGridT::getBlockEdgeSize() + 2 * supportRadius, dim>::value;

        const auto coord = sparseGrid.getCoordInEnlargedBlock(offset);
        const auto linId = sparseGrid.getLinIdInEnlargedBlock(offset);
        char boundaryDirection[dim];
        bool isBoundary = sparseGrid.getIfBoundaryElementInEnlargedBlock(coord, boundaryDirection);

        unsigned int nPlusId[dim], nMinusId[dim];
        for (int d=0; d<dim; ++d)
        {
            nPlusId[d] = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, 1);
            nMinusId[d] = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, -1);
        }

        __shared__ ScalarT enlargedBlock[enlargedBlockSize];

        ScalarT * nPlus[dim];
        ScalarT * nMinus[dim];
        for (int d=0; d<dim; ++d)
        {
            const auto boundaryDir = boundaryDirection[d];
            const auto nCoord = sparseGrid.getNeighbour(pointCoord, d, boundaryDir);
            const auto nOffset = sparseGrid.getLinId(nCoord) % blockSize;
            nPlus[d] = &(enlargedBlock[nPlusId[d]]);
            nMinus[d] = &(enlargedBlock[nMinusId[d]]);
            if (boundaryDir==1)
            {
//                nPlus[d] = &(sparseGrid.getBlock(nCoord).template get<p>()[nOffset]);
//                nPlus[d] = &(sparseGrid.template get<p>(nCoord));
                nPlus[d] = sparseGrid.getBlock(nCoord).template get<p>().block + nOffset;
            }
            else if (boundaryDir==-1)
            {
//                nMinus[d] = &(sparseGrid.getBlock(nCoord).template get<p>()[nOffset]);
//                nMinus[d] = &(sparseGrid.template get<p>(nCoord));
                nMinus[d] = sparseGrid.getBlock(nCoord).template get<p>().block + nOffset;
            }
        }

        sparseGrid.loadBlock<p>(dataBlockLoad, enlargedBlock);
        __syncthreads();
//            sparseGrid.loadGhost<p>(dataBlockCoord, enlargedBlock);
//            __syncthreads();

        ScalarT cur = enlargedBlock[linId];
        ScalarT laplacian = -2.0 * dim * cur; // The central part of the stencil
        for (int d = 0; d < dim; ++d)
        {
//                const auto boundary = boundaryDirection[d];
//                ScalarT neighbourPlus = enlargedBlock[nPlusId[d]];
//                ScalarT neighbourMinus = enlargedBlock[nMinusId[d]];
//                if (boundary == 1)
//                {
//                    neighbourPlus = *(nPlus[d]);
//                }
//                else if (boundary == -1)
//                {
//                    neighbourMinus = *(nMinus[d]);
//                }
            laplacian += *(nMinus[d]) + *(nPlus[d]);
//                laplacian += neighbourMinus + neighbourPlus;

        }
        enlargedBlock[linId] = cur + dt * laplacian;

        __syncthreads();
        sparseGrid.storeBlock<p>(dataBlockStore, enlargedBlock);
        __syncthreads();
    }

    template <typename SparseGridT, typename CtxT>
    static inline void __host__ flush(SparseGridT & sparseGrid, CtxT & ctx)
    {
        sparseGrid.template flush <smin_<0>> (ctx, flush_type::FLUSH_ON_DEVICE);
    }
};

BOOST_AUTO_TEST_SUITE(SparseGridGpu_Stencil_Performance_tests)
    BOOST_AUTO_TEST_CASE(testStencilHeatPerf)
    {
        constexpr unsigned int dim = 2;
        constexpr unsigned int blockEdgeSize = 8;
        unsigned int gridEdgeSize = 256;
        constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;
        typedef aggregate<float> AggregateT;
//        typedef HeatStencil<dim, 0> StencilT;
        typedef HeatStencilOrig<dim, 0> StencilT;

        unsigned int iterations = 5;
        unsigned int repetitions = 1;

        float timeInitAvg;
        float timeStencilAvg;
        float timeTotalAvg;

        for (int rep=0; rep<repetitions; ++rep)
        {

            hipEvent_t start, afterInit, stop;
            float timeInit;
            float timeStencil;
            float timeTotal;

            HANDLE_ERROR(hipEventCreate(&start));
            HANDLE_ERROR(hipEventCreate(&afterInit));
            HANDLE_ERROR(hipEventCreate(&stop));

            HANDLE_ERROR(hipEventRecord(start, 0));

            dim3 gridSize(gridEdgeSize, gridEdgeSize);
            dim3 blockSize(blockEdgeSize, blockEdgeSize);
            BlockGeometry<dim, blockEdgeSize> blockGeometry(gridSize);
            SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize> sparseGrid(blockGeometry);
            mgpu::ofp_context_t ctx;
            sparseGrid.template setBackgroundValue<0>(0);

            // Initialize the grid
            sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
            insertConstantValue<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), 0);
            sparseGrid.flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

            sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
            dim3 sourcePt(gridSize.x * blockEdgeSize / 2, gridSize.y * blockEdgeSize / 2, 0);
            insertOneValue<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), sourcePt, 100);
            sparseGrid.flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

            HANDLE_ERROR(hipEventRecord(afterInit, 0));
            HANDLE_ERROR(hipEventSynchronize(afterInit));

            for (unsigned int iter=0; iter<iterations; ++iter)
            {
                sparseGrid.applyStencils<StencilT>(STENCIL_MODE_INPLACE, 0.1);
                hipDeviceSynchronize();
            }

            HANDLE_ERROR(hipEventRecord(stop, 0));
            HANDLE_ERROR(hipEventSynchronize(stop));
            HANDLE_ERROR(hipEventElapsedTime(&timeInit, start, afterInit));
            HANDLE_ERROR(hipEventElapsedTime(&timeStencil, afterInit, stop));
            HANDLE_ERROR(hipEventElapsedTime(&timeTotal, start, stop));

            timeInitAvg += timeInit;
            timeStencilAvg += timeStencil;
            timeTotalAvg += timeTotal;
        }

        timeInitAvg /= repetitions;
        timeStencilAvg /= repetitions;
        timeTotalAvg /= repetitions;

        unsigned int numElements = gridEdgeSize*blockEdgeSize*gridEdgeSize*blockEdgeSize;
        float gElemS = numElements * iterations / (1e9 * timeStencilAvg/1000);
        float gFlopsS = gElemS * StencilT::flops;
        printf("Grid: %ux%u\n", gridEdgeSize*blockEdgeSize, gridEdgeSize*blockEdgeSize);
        printf("Iterations: %u\n", iterations);
        printf("Timing (avg on %u repetitions):\n\tInit: %f ms\n\tStencil: %f ms\n\tTotal: %f ms\n",
                repetitions, timeInitAvg, timeStencilAvg, timeTotalAvg);
        printf("Throughput:\n\t%f GElem/s\n\t%f GFlops/s\n", gElemS, gFlopsS);

    }
BOOST_AUTO_TEST_SUITE_END()
