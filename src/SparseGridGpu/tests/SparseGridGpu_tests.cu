#include "hip/hip_runtime.h"
//
// Created by tommaso on 10/06/19.
//

#define BOOST_TEST_DYN_LINK

#include <boost/test/unit_test.hpp>
#include "SparseGridGpu/SparseGridGpu.hpp"

template<unsigned int p, typename SparseGridType>
__global__ void insertValues(SparseGridType sparseGrid)
{
    sparseGrid.init();

    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    size_t pos = sparseGrid.getLinId(coord);
//    printf("insertValues: bDim=(%d,%d), bId=(%d,%d), tId=(%d,%d) : "
//           "pos=%ld, coord={%d,%d}, value=%d\n",
//           bDimX, bDimY,
//           bIdX, bIdY,
//           tIdX, tIdY,
//           pos,
//           x, y,
//           x); //debug

    sparseGrid.template insert<p>(coord) = x;

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType, typename ScalarT>
__global__ void insertConstantValue(SparseGridType sparseGrid, ScalarT value)
{
    sparseGrid.init();

    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    sparseGrid.template insert<p>(coord) = value;

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType, typename ValueT>
__global__ void insertOneValue(SparseGridType sparseGrid, dim3 pt, ValueT value)
{
    sparseGrid.init();

    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    dim3 thCoord(x, y, z);
    if (thCoord.x == pt.x && thCoord.y == pt.y && thCoord.z == pt.z)
    {
        grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});
        sparseGrid.template insert<p>(coord) = value;
    }
    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType, typename VectorOutType>
__global__ void copyBlocksToOutput(SparseGridType sparseGrid, VectorOutType output)
{
    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    size_t pos = sparseGrid.getLinId(coord);

    auto value = sparseGrid.template get<p>(coord);

//    printf("copyBlocksToOutput: bDim=(%d,%d), bId=(%d,%d), tId=(%d,%d) : "
//           "pos=%ld, coord={%d,%d}, value=%d\n",
//           bDimX, bDimY,
//           bIdX, bIdY,
//           tIdX, tIdY,
//           pos,
//           x, y,
//           static_cast<int>(value)); //debug

    output.template get<p>(pos) = value;

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType, typename VectorOutType>
__global__ void copyToOutputIfPadding(SparseGridType sparseGrid, VectorOutType output)
{
    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    size_t pos = sparseGrid.getLinId(coord);

//    auto value = sparseGrid.template get<p>(coord);
//    auto mask = sparseGrid.template get<1>(coord);
//    if (value == 1)
//    {
//        printf("copyBlocksToOutput: bDim=(%d,%d), bId=(%d,%d), tId=(%d,%d) : "
//               "pos=%ld, coord={%d,%d}, value=%d, mask=%u\n",
//               bDimX, bDimY,
//               bIdX, bIdY,
//               tIdX, tIdY,
//               pos,
//               x, y,
//               static_cast<int>(value),
//               static_cast<unsigned char>(mask)); //debug
//    }
//
//    if (sparseGrid.isPadding(coord))
//    {
//        printf("OUTPUT : Element isPadding! pos=%u\n",
//               pos); //debug
//    }

    output.template get<p>(pos) = sparseGrid.isPadding(coord) ? 1 : 0;

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType>
__global__ void insertBoundaryValuesHeat(SparseGridType sparseGrid)
{
    sparseGrid.init();

    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    float value = 0;
    if (x == 0)
    {
        value = 0;
    }
    else if (x == bDimX * gridDim.x - 1)
    {
        value = 10;
    }

    if (y == 0 || y == bDimY * gridDim.y - 1)
    {
        value = 10.0 * x / (bDimX * gridDim.x - 1);
    }

    sparseGrid.template insert<p>(coord) = value;

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int dim, unsigned int p>
struct LaplacianStencil
{
    // This is an example of a laplacian stencil to apply using the apply stencil facility of SparseGridGpu

    static constexpr unsigned int supportRadius = 1;

    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __device__ void stencil(
            SparseGridT & sparseGrid,
            grid_key_dx<dim, int> & dataBlockCoord,
            unsigned int offset,
            grid_key_dx<dim, int> & pointCoord,
            DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore)
    {
        typedef typename SparseGridT::AggregateBlockType AggregateT;
        typedef ScalarTypeOf<AggregateT, p> ScalarT;

        constexpr unsigned int enlargedBlockSize = IntPow<
                SparseGridT::getBlockEdgeSize() + 2 * supportRadius, dim>::value;

        __shared__ ScalarT enlargedBlock[enlargedBlockSize];
        sparseGrid.loadBlock<p>(dataBlockLoad, enlargedBlock);
        sparseGrid.loadGhost<p>(dataBlockCoord, enlargedBlock);
        __syncthreads();

        const auto coord = sparseGrid.getCoordInEnlargedBlock(offset);
        const auto linId = sparseGrid.getLinIdInEnlargedBlock(offset);
        ScalarT cur = enlargedBlock[linId];
        ScalarT res = -2.0*dim*cur; // The central part of the stencil
        for (int d=0; d<dim; ++d)
        {
            auto nPlusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, 1);
            auto nMinusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, -1);
            ScalarT neighbourPlus = enlargedBlock[nPlusId];
            ScalarT neighbourMinus = enlargedBlock[nMinusId];
            res += neighbourMinus + neighbourPlus;
        }
        enlargedBlock[linId] = res;

        __syncthreads();
        sparseGrid.storeBlock<p>(dataBlockStore, enlargedBlock);
    }

    template <typename SparseGridT, typename CtxT>
    static inline void __host__ flush(SparseGridT & sparseGrid, CtxT & ctx)
    {
        sparseGrid.template flush <smin_<0>> (ctx, flush_type::FLUSH_ON_DEVICE);
    }
};

template<unsigned int dim, unsigned int p>
struct HeatStencil
{
    // This is an example of a laplacian smoothing stencil to apply using the apply stencil facility of SparseGridGpu

    static constexpr unsigned int supportRadius = 1;

    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __device__ void stencil(
            SparseGridT & sparseGrid,
            const unsigned int dataBlockId,
            const int * neighboursPositions,
            unsigned int offset,
            grid_key_dx<dim, int> & pointCoord,
            DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore,
            bool applyStencilHere,
            float dt)
    {
        typedef typename SparseGridT::AggregateBlockType AggregateT;
        typedef ScalarTypeOf<AggregateT, p> ScalarT;

        constexpr unsigned int enlargedBlockSize = IntPow<
                SparseGridT::getBlockEdgeSize() + 2 * supportRadius, dim>::value;

        __shared__ ScalarT enlargedBlock[enlargedBlockSize];
        sparseGrid.loadBlock<p>(dataBlockLoad, enlargedBlock);
        sparseGrid.loadGhost<p>(dataBlockId, neighboursPositions, enlargedBlock);
//        sparseGrid.loadGhost<p>(dataBlockId, nullptr, enlargedBlock);
        __syncthreads();

        if (applyStencilHere)
        {
            const auto coord = sparseGrid.getCoordInEnlargedBlock(offset);
            const auto linId = sparseGrid.getLinIdInEnlargedBlock(offset);
            ScalarT cur = enlargedBlock[linId];
            ScalarT laplacian = -2.0 * dim * cur; // The central part of the stencil
            for (int d = 0; d < dim; ++d)
            {
                auto nPlusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, 1);
                auto nMinusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, -1);
                ScalarT neighbourPlus = enlargedBlock[nPlusId];
                ScalarT neighbourMinus = enlargedBlock[nMinusId];
                laplacian += neighbourMinus + neighbourPlus;
            }
            enlargedBlock[linId] = cur + dt * laplacian;
        }

        __syncthreads();
        sparseGrid.storeBlock<p>(dataBlockStore, enlargedBlock);
    }

    template <typename SparseGridT, typename CtxT>
    static inline void __host__ flush(SparseGridT & sparseGrid, CtxT & ctx)
    {
        sparseGrid.template flush <smax_<0>> (ctx, flush_type::FLUSH_ON_DEVICE);
    }
};

template<unsigned int dim, unsigned int p>
struct HeatStencil2
{
    // This is an example of a laplacian smoothing stencil to apply using the apply stencil facility of SparseGridGpu

    static constexpr unsigned int flops = 3 + 2*dim;

    static constexpr unsigned int supportRadius = 1;

    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __device__ void stencil(
            SparseGridT & sparseGrid,
            grid_key_dx<dim, int> & dataBlockCoord,
            unsigned int offset,
            grid_key_dx<dim, int> & pointCoord,
            DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore,
            float dt, unsigned int maxIter=1000)
    {
        typedef typename SparseGridT::AggregateBlockType AggregateT;
        typedef ScalarTypeOf<AggregateT, p> ScalarT;
        typedef BlockTypeOf<AggregateT, p> BlockT;
        constexpr unsigned int blockSize = BlockT::size;

        constexpr unsigned int enlargedBlockSize = IntPow<
                SparseGridT::getBlockEdgeSize() + 2 * supportRadius, dim>::value;

        const auto coord = sparseGrid.getCoordInEnlargedBlock(offset);
        const auto linId = sparseGrid.getLinIdInEnlargedBlock(offset);
        char boundaryDirection[dim];
        bool isBoundary = sparseGrid.getIfBoundaryElementInEnlargedBlock(coord, boundaryDirection);

        unsigned int nPlusId[dim], nMinusId[dim];
        for (int d=0; d<dim; ++d)
        {
            nPlusId[d] = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, 1);
            nMinusId[d] = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, -1);
        }

        __shared__ ScalarT enlargedBlock[enlargedBlockSize];

        ScalarT * nPlus[dim];
        ScalarT * nMinus[dim];
        for (int d=0; d<dim; ++d)
        {
            const auto boundaryDir = boundaryDirection[d];
            const auto nCoord = sparseGrid.getNeighbour(pointCoord, d, boundaryDir);
            const auto nOffset = sparseGrid.getLinId(nCoord) % blockSize;
            nPlus[d] = &(enlargedBlock[nPlusId[d]]);
            nMinus[d] = &(enlargedBlock[nMinusId[d]]);
            if (boundaryDir==1)
            {
//                nPlus[d] = &(sparseGrid.getBlock(nCoord).template get<p>()[nOffset]);
//                nPlus[d] = &(sparseGrid.template get<p>(nCoord));
//                nPlus[d] = sparseGrid.getBlock(nCoord).template getPtr<p>()->block + nOffset;
                nPlus[d] = sparseGrid.getBlock(nCoord).template get<p>().block + nOffset;
            }
            else if (boundaryDir==-1)
            {
//                nMinus[d] = &(sparseGrid.getBlock(nCoord).template get<p>()[nOffset]);
//                nMinus[d] = &(sparseGrid.template get<p>(nCoord));
//                nMinus[d] = sparseGrid.getBlock(nCoord).template getPtr<p>()->block + nOffset;
                nMinus[d] = sparseGrid.getBlock(nCoord).template get<p>().block + nOffset;
            }
        }

        sparseGrid.loadBlock<p>(dataBlockLoad, enlargedBlock);
        __syncthreads();
        for (unsigned int iter=0; iter<maxIter; ++iter)
        {
//            sparseGrid.loadGhost<p>(dataBlockCoord, enlargedBlock);
//            __syncthreads();

//todo: capisci come mai questa load non va mentre con la load ghost si!

            ScalarT cur = enlargedBlock[linId];
            ScalarT laplacian = -2.0 * dim * cur; // The central part of the stencil
            for (int d = 0; d < dim; ++d)
            {
//                const auto boundary = boundaryDirection[d];
//                ScalarT neighbourPlus = enlargedBlock[nPlusId[d]];
//                ScalarT neighbourMinus = enlargedBlock[nMinusId[d]];
//                if (boundary == 1)
//                {
//                    neighbourPlus = *(nPlus[d]);
//                }
//                else if (boundary == -1)
//                {
//                    neighbourMinus = *(nMinus[d]);
//                }
//                laplacian += neighbourMinus + neighbourPlus;
                laplacian += *(nMinus[d]) + *(nPlus[d]);
            }
            enlargedBlock[linId] = cur + dt * laplacian;

            __syncthreads();
//            sparseGrid.storeBlock<p>(dataBlockLoad, enlargedBlock);
            if (isBoundary)
            {
                dataBlockLoad.template get<p>()[offset] = enlargedBlock[linId];
            }
            __syncthreads();
        }
        __syncthreads();
        sparseGrid.storeBlock<p>(dataBlockStore, enlargedBlock);
        __syncthreads();
    }

    template <typename SparseGridT, typename CtxT>
    static inline void __host__ flush(SparseGridT & sparseGrid, CtxT & ctx)
    {
        sparseGrid.template flush <smin_<0>> (ctx, flush_type::FLUSH_ON_DEVICE);
    }
};

//todo: Here write some sort of stencil kernel to test the client interface for stencil application

BOOST_AUTO_TEST_SUITE(SparseGridGpu_tests)

    BOOST_AUTO_TEST_CASE(testInsert)
    {

        std::cout << std::endl; //debug empty line

        constexpr unsigned int dim = 2;
        constexpr unsigned int blockEdgeSize = 8;
        constexpr unsigned int dataBlockSize = blockEdgeSize * blockEdgeSize;
        typedef aggregate<DataBlock<float, dataBlockSize>> AggregateSGT;
        typedef aggregate<float> AggregateOutT;

        dim3 gridSize(2, 2);
        dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

        BlockGeometry<dim, blockEdgeSize> blockGeometry(gridSize);
        SparseGridGpu<dim, AggregateOutT, blockEdgeSize> sparseGrid(blockGeometry);

        sparseGrid.template setBackgroundValue<0>(666);
//        const unsigned int gridSizeLin = 4;
//        const unsigned int bufferPoolSize = 1024;
//        const unsigned int blockSizeInsert = 128;
//        const unsigned int gridSizeRead = gridSize + 1;
//        const unsigned int blockSizeRead = 128;

//        sparseGrid.setGPUInsertBuffer(gridSizeLin, bufferPoolSize);
        sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);

        insertValues<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel());

        mgpu::ofp_context_t ctx;
        sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

        // Get output
        openfpm::vector_gpu<AggregateOutT> output;
        output.resize(4 * 64);

        copyBlocksToOutput<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), output.toKernel());

        output.template deviceToHost<0>();
        sparseGrid.template deviceToHost<0>();

        // Compare
        bool match = true;
        for (size_t i = 0; i < output.size(); i++)
        {
//            auto expectedValue = (i < gridSize * blockSizeInsert) ? i : 666;
            auto coord = sparseGrid.getCoord(i);
            auto expectedValue = coord.get(0);

            std::cout << "sparseGrid(" << coord.get(0) << "," << coord.get(1) << ") = "
                      << sparseGrid.template get<0>(coord)
                      << " == "
                      << expectedValue
                      << " == "
                      << output.template get<0>(i) << " = output(" << i << ")"
                      << std::endl;
            match &= output.template get<0>(i) == sparseGrid.template get<0>(coord);
            match &= output.template get<0>(i) == expectedValue;
        }

        BOOST_REQUIRE_EQUAL(match, true);
    }

    BOOST_AUTO_TEST_CASE(testInsert3D)
    {
        constexpr unsigned int dim = 3;
        constexpr unsigned int blockEdgeSize = 4;
        constexpr unsigned int dataBlockSize = blockEdgeSize * blockEdgeSize;
        typedef aggregate<DataBlock<float, dataBlockSize>> AggregateSGT;
        typedef aggregate<float> AggregateOutT;

        dim3 gridSize(2, 2, 2);
        dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize, blockEdgeSize);

        BlockGeometry<dim, blockEdgeSize> blockGeometry(gridSize);
        SparseGridGpu<dim, AggregateOutT, blockEdgeSize> sparseGrid(blockGeometry);

        sparseGrid.template setBackgroundValue<0>(666);

        sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);

        insertValues<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel());

        mgpu::ofp_context_t ctx;
        sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

        // Get output
        openfpm::vector_gpu<AggregateOutT> output;
        output.resize(sparseGrid.dim3SizeToInt(gridSize) * 64);

        copyBlocksToOutput<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), output.toKernel());

        output.template deviceToHost<0>();
        sparseGrid.template deviceToHost<0>();

        // Compare
        bool match = true;
        for (size_t i = 0; i < output.size(); i++)
        {
//            auto expectedValue = (i < gridSize * blockSizeInsert) ? i : 666;
            auto coord = sparseGrid.getCoord(i);
            auto expectedValue = coord.get(0);

//            std::cout << "sparseGrid(" << coord.get(0) << "," << coord.get(1) << "," << coord.get(2) << ") = "
//                      << sparseGrid.template get<0>(coord)
//                      << " == "
//                      << expectedValue
//                      << " == "
//                      << output.template get<0>(i) << " = output(" << i << ")"
//                      << std::endl;
            match &= output.template get<0>(i) == sparseGrid.template get<0>(coord);
            match &= output.template get<0>(i) == expectedValue;
        }

        BOOST_REQUIRE_EQUAL(match, true);
    }

    BOOST_AUTO_TEST_CASE(testTagBoundaries)
    {

        constexpr unsigned int dim = 2;
        constexpr unsigned int blockEdgeSize = 8;
        typedef aggregate<float> AggregateT;

        dim3 gridSize(2, 2);
        dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

        BlockGeometry<dim, blockEdgeSize> blockGeometry(gridSize);
        SparseGridGpu<dim, AggregateT, blockEdgeSize, 64> sparseGrid(blockGeometry);

        sparseGrid.template setBackgroundValue<0>(666);
        mgpu::ofp_context_t ctx;

        sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
        dim3 pt1(0, 0, 0);
        insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), pt1, 1);
        dim3 pt2(6, 6, 0);
        insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), pt2, 1);
        dim3 pt3(7, 6, 0);
        insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), pt3, 1);
        sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

        sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
        dim3 pt4(8, 6, 0);
        insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), pt4, 1);
        sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
        /////////
        sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
        for (int y = 9; y <= 11; y++)
        {
            dim3 pt1(6, y, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), pt1, 1);
            dim3 pt2(7, y, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), pt2, 1);
        }
        sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

        sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
        for (int y = 9; y <= 11; y++)
        {
            dim3 pt1(8, y, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), pt1, 1);
            dim3 pt2(9, y, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), pt2, 1);
        }
        sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

//        sparseGrid.hostToDevice(); //just sync masks
        sparseGrid.deviceToHost(); //just sync masks
//        sparseGrid.deviceToHost<0>();

        sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
        // Now tag the boundaries
        sparseGrid.tagBoundaries();

        // Get output
        openfpm::vector_gpu<AggregateT> output;
        output.resize(4 * 64);

        copyToOutputIfPadding<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), output.toKernel());

        output.template deviceToHost<0>();
        sparseGrid.template deviceToHost<0>();

        // Compare
        bool match = true;
        for (size_t i = 0; i < output.size(); i++)
        {
            auto coord = sparseGrid.getCoord(i);
            auto expectedValue =
                     ( i == 0
                    || i == 54
                    || i == 55
                    || i == 112
                    || i == 142 || i == 143 || i == 200 || i == 201 // (6,9), (7,9), (8,9), (9,9)
                    || i == 150 || i == 209 // (6,10), (9,10)
                    || i == 158 || i == 159 || i == 216 || i == 217 // (6,11), (7,11), (8,11), (9,11)
                     ) ? 1 : 0;

            std::cout
                    << "sparseGrid(" << coord.get(0) << "," << coord.get(1) << "," << coord.get(2) << ") = "
                    << sparseGrid.template get<0>(coord) << " | "
                    << expectedValue
                    << " == "
                    << output.template get<0>(i) << " = output(" << i << ")"
                    << std::endl;
            match &= output.template get<0>(i) == expectedValue;
        }

        BOOST_REQUIRE_EQUAL(match, true);
    }

        BOOST_AUTO_TEST_CASE(testTagBoundaries2)
    {
        constexpr unsigned int dim = 2;
        constexpr unsigned int blockEdgeSize = 8;
        typedef aggregate<float> AggregateT;

        dim3 gridSize(2, 2);
        dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

        BlockGeometry<dim, blockEdgeSize> blockGeometry(gridSize);
        SparseGridGpu<dim, AggregateT, blockEdgeSize, 64> sparseGrid(blockGeometry);

        sparseGrid.template setBackgroundValue<0>(666);
        mgpu::ofp_context_t ctx;

        ///////
        {
            sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
            dim3 ptd1(6, 6, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd1, 1);
            dim3 ptd2(6, 7, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd2, 1);
            dim3 ptd3(7, 6, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd3, 1);
            dim3 ptd4(7, 7, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd4, 1);
            sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
        }
        {
            sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
            dim3 ptd1(8, 6, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd1, 1);
            dim3 ptd2(9, 6, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd2, 1);
            dim3 ptd3(8, 7, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd3, 1);
            dim3 ptd4(9, 7, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd4, 1);
            sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
        }
        {
            sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
            dim3 ptd1(6, 8, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd1, 1);
            dim3 ptd2(7, 8, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd2, 1);
            dim3 ptd3(6, 9, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd3, 1);
            dim3 ptd4(7, 9, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd4, 1);
            sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
        }
        {
            sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
            dim3 ptd1(8, 8, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd1, 1);
            dim3 ptd2(8, 9, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd2, 1);
            dim3 ptd3(9, 8, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd3, 1);
            dim3 ptd4(9, 9, 0);
            insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd4, 1);
            sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
        }
        ///////

//        sparseGrid.hostToDevice(); //just sync masks
        sparseGrid.deviceToHost(); //just sync masks
//        sparseGrid.deviceToHost<0>();

        sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
        // Now tag the boundaries
        sparseGrid.tagBoundaries();

        // Get output
        openfpm::vector_gpu<AggregateT> output;
        output.resize(4 * 64);

        copyToOutputIfPadding<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), output.toKernel());

        output.template deviceToHost<0>();
        sparseGrid.template deviceToHost<0>();

        // Compare
        bool match = true;
        for (size_t i = 0; i < output.size(); i++)
        {
            auto coord = sparseGrid.getCoord(i);
            auto expectedValue =
                     (
                             i == 54 || i == 55 || i == 62 // (6,6), (7,6), (6,7)
                          || i == 134 || i == 142 || i == 143 // (6,8), (6,9), (7,9)
                          || i == 112 || i == 113 || i == 121 // (8,6), (9,6), (9,7)
                          || i == 200 || i == 193 || i == 201 // (8,9), (9,8), (9,9)
                     ) ? 1 : 0;

            std::cout
                    << "sparseGrid(" << coord.get(0) << "," << coord.get(1) << "," << coord.get(2) << ") = "
                    << sparseGrid.template get<0>(coord) << " | "
                    << expectedValue
                    << " == "
                    << output.template get<0>(i) << " = output(" << i << ")"
                    << std::endl;
            match &= output.template get<0>(i) == expectedValue;
        }

        BOOST_REQUIRE_EQUAL(match, true);
    }

    BOOST_AUTO_TEST_CASE(testStencilHeat)
    {
        printf("\n");

        constexpr unsigned int dim = 2;
        constexpr unsigned int blockEdgeSize = 8;
        typedef aggregate<float> AggregateT;

        dim3 gridSize(2, 2);
        dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

        BlockGeometry<dim, blockEdgeSize> blockGeometry(gridSize);
        SparseGridGpu<dim, AggregateT, blockEdgeSize, 64> sparseGrid(blockGeometry);
        mgpu::ofp_context_t ctx;
        sparseGrid.template setBackgroundValue<0>(0);

        // Insert values on the grid
        sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
        insertConstantValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), 0);
        sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

        sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
        insertBoundaryValuesHeat<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel());
        sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

//        sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
//        dim3 pt2(4, 4, 0);
//        insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), pt2, 100);
//        sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

        sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!

//        // Now tag the boundaries
        sparseGrid.tagBoundaries();

        // Now apply the laplacian operator
        const unsigned int maxIter = 1000;
//        const unsigned int maxIter = 10;
        for (unsigned int iter=0; iter<maxIter; ++iter)
        {
            sparseGrid.applyStencils<HeatStencil<dim, 0>>(STENCIL_MODE_INPLACE, 0.1);
            hipDeviceSynchronize();
        }

        // Get output
        openfpm::vector_gpu<AggregateT> output;
        output.resize(4 * 64);

        copyBlocksToOutput<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), output.toKernel());

        output.template deviceToHost<0>();
        sparseGrid.template deviceToHost<0>();

        // Compare
        bool match = true;
        for (size_t i = 0; i < output.size(); i++)
        {
            auto coord = sparseGrid.getCoord(i);
            float expectedValue = 10.0 * coord.get(0) / (gridSize.x * blockEdgeSize - 1);

            std::cout
                    << "sparseGrid(" << coord.get(0) << "," << coord.get(1) << "," << coord.get(2) << ") = "
                    << sparseGrid.template get<0>(coord) << " | "
                    << expectedValue
                    << " == "
                    << output.template get<0>(i) << " = output(" << i << ")"
                    << std::endl;
            match &= fabs(output.template get<0>(i) - expectedValue) < 1e-2;

        }

        BOOST_REQUIRE_EQUAL(match, true);
//        BOOST_REQUIRE_CLOSE(output.template get<0>(255), 3.20309591e-05, 1e-6);
    }

    BOOST_AUTO_TEST_CASE(testStencilHeatInsert)
    {
        printf("\n");

        constexpr unsigned int dim = 2;
        constexpr unsigned int blockEdgeSize = 8;
        typedef aggregate<float> AggregateT;

        dim3 gridSize(2, 2);
        dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

        BlockGeometry<dim, blockEdgeSize> blockGeometry(gridSize);
        SparseGridGpu<dim, AggregateT, blockEdgeSize, 64> sparseGrid(blockGeometry);
        mgpu::ofp_context_t ctx;
        sparseGrid.template setBackgroundValue<0>(0);

        // Insert values on the grid
        sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
        insertConstantValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), 0);
        sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

        sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
        insertBoundaryValuesHeat<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel());
        sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

//        sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
//        dim3 pt2(4, 4, 0);
//        insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), pt2, 100);
//        sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

        sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!

//        // Now tag the boundaries
        sparseGrid.tagBoundaries();

        // Now apply the laplacian operator
        const unsigned int maxIter = 1000;
//        const unsigned int maxIter = 10;
        for (unsigned int iter=0; iter<maxIter; ++iter)
        {
            sparseGrid.applyStencils<HeatStencil<dim, 0>>(STENCIL_MODE_INSERT, 0.1);
//            sparseGrid.applyStencils<HeatStencil<dim, 0>>(STENCIL_MODE_INPLACE, 0.1);
            hipDeviceSynchronize();
        }

        // Get output
        openfpm::vector_gpu<AggregateT> output;
        output.resize(4 * 64);

        copyBlocksToOutput<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), output.toKernel());

        output.template deviceToHost<0>();
        sparseGrid.template deviceToHost<0>();

        // Compare
        bool match = true;
        for (size_t i = 0; i < output.size(); i++)
        {
            grid_key_dx<dim, int> coord = sparseGrid.getCoord(i);
            float expectedValue = 10.0 * coord.get(0) / (gridSize.x * blockEdgeSize - 1);

            unsigned int check = sparseGrid.getLinId(coord);

            std::cout
                    << "invLinId=" << check << ", "
//                    << "sparseGrid(" << coord.get(0) << "," << coord.get(1) << "," << coord.get(2) << ") = "
                    << "sparseGrid(" << coord.get(0) << "," << coord.get(1) << ") = "
                    << sparseGrid.template get<0>(coord) << " | "
                    << expectedValue
                    << " == "
                    << output.template get<0>(i) << " = output(" << i << ")"
                    << std::endl;
            match &= fabs(output.template get<0>(i) - expectedValue) < 1e-2;

        }

        BOOST_REQUIRE_EQUAL(match, true);
//        BOOST_REQUIRE_CLOSE(output.template get<0>(255), 3.20309591e-05, 1e-6);
    }

BOOST_AUTO_TEST_SUITE_END()
