#include "hip/hip_runtime.h"
//
// Created by tommaso on 10/06/19.
//

#define BOOST_TEST_DYN_LINK

#define DISABLE_MPI_WRITTERS

#include <boost/test/unit_test.hpp>
#include "SparseGridGpu/SparseGridGpu.hpp"
#include "SparseGridGpu/tests/utils/SparseGridGpu_testKernels.cuh"
#include "SparseGridGpu/tests/utils/SparseGridGpu_util_test.cuh"

template<unsigned int p1 , unsigned int p2, unsigned int chunksPerBlock=1, typename SparseGridType, typename ScalarT>
__global__ void insertConstantValue2(SparseGridType sparseGrid, ScalarT value)
{
    constexpr unsigned int pMask = SparseGridType::pMask;
    typedef BlockTypeOf<typename SparseGridType::AggregateType, p1> BlockT;

    sparseGrid.init();

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    auto pos = sparseGrid.getLinId(coord);
    unsigned int dataBlockId = pos / BlockT::size;
    unsigned int offset = pos % BlockT::size;

    auto encap = sparseGrid.insertBlock(dataBlockId);
    encap.template get<p1>()[offset] = value;
    encap.template get<p2>()[offset] = value;
    BlockMapGpu_ker<>::setExist(encap.template get<pMask>()[offset]);

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType>
__global__ void insertValues(SparseGridType sparseGrid)
{
    sparseGrid.init();

    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    size_t pos = sparseGrid.getLinId(coord);

    sparseGrid.template insert<p>(coord) = x;

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
    z++;
}



template<unsigned int p, typename SparseGridType, typename VectorOutType>
__global__ void copyToOutputIfPadding(SparseGridType sparseGrid, VectorOutType output)
{
    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    size_t pos = sparseGrid.getLinId(coord);


    output.template get<p>(pos) = sparseGrid.isPadding(coord) ? 1 : 0;

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType>
__global__ void insertBoundaryValuesHeat(SparseGridType sparseGrid)
{
    sparseGrid.init();

    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    float value = 0;
    if (x == 0)
    {
        value = 0;
    }
    else if (x == bDimX * gridDim.x - 1)
    {
        value = 10;
    }

    if (y == 0 || y == bDimY * gridDim.y - 1)
    {
        value = 10.0 * x / (bDimX * gridDim.x - 1);
    }

    sparseGrid.template insert<p>(coord) = value;

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int dim, unsigned int p>
struct LaplacianStencil
{
    // This is an example of a laplacian stencil to apply using the apply stencil facility of SparseGridGpu

    static constexpr unsigned int supportRadius = 1;

    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __device__ void stencil(
            SparseGridT & sparseGrid,
            grid_key_dx<dim, int> & dataBlockCoord,
            unsigned int offset,
            grid_key_dx<dim, int> & pointCoord,
            DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore)
    {
        typedef typename SparseGridT::AggregateBlockType AggregateT;
        typedef ScalarTypeOf<AggregateT, p> ScalarT;

        constexpr unsigned int enlargedBlockSize = IntPow<
                SparseGridT::getBlockEdgeSize() + 2 * supportRadius, dim>::value;

        __shared__ ScalarT enlargedBlock[enlargedBlockSize];
        sparseGrid.loadBlock<p>(dataBlockLoad, enlargedBlock);
        sparseGrid.loadGhost<p>(dataBlockCoord, enlargedBlock);
        __syncthreads();

        const auto coord = sparseGrid.getCoordInEnlargedBlock(offset);
        const auto linId = sparseGrid.getLinIdInEnlargedBlock(offset);
        ScalarT cur = enlargedBlock[linId];
        ScalarT res = -2.0*dim*cur; // The central part of the stencil
        for (int d=0; d<dim; ++d)
        {
            auto nPlusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, 1);
            auto nMinusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, -1);
            ScalarT neighbourPlus = enlargedBlock[nPlusId];
            ScalarT neighbourMinus = enlargedBlock[nMinusId];
            res += neighbourMinus + neighbourPlus;
        }
        enlargedBlock[linId] = res;

        __syncthreads();
        sparseGrid.storeBlock<p>(dataBlockStore, enlargedBlock);
    }

    template <typename SparseGridT, typename CtxT>
    static inline void __host__ flush(SparseGridT & sparseGrid, CtxT & ctx)
    {
        sparseGrid.template flush <smin_<0>> (ctx, flush_type::FLUSH_ON_DEVICE);
    }
};

//todo: Here write some sort of stencil kernel to test the client interface for stencil application

BOOST_AUTO_TEST_SUITE(SparseGridGpu_tests)

BOOST_AUTO_TEST_CASE(testInsert)
{

	std::cout << std::endl; //debug empty line

	constexpr unsigned int dim = 2;
	constexpr unsigned int blockEdgeSize = 8;
	constexpr unsigned int dataBlockSize = blockEdgeSize * blockEdgeSize;
	typedef aggregate<DataBlock<float, dataBlockSize>> AggregateSGT;
	typedef aggregate<float> AggregateOutT;

	dim3 gridSize(2, 2);
	dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateOutT, blockEdgeSize> sparseGrid(blockGeometry);

	sparseGrid.template setBackgroundValue<0>(666);
	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);

	insertValues<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel());

	mgpu::ofp_context_t ctx;
	sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	// Get output
	openfpm::vector_gpu<AggregateOutT> output;
	output.resize(4 * 64);

	copyBlocksToOutput<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	sparseGrid.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
		auto coord = sparseGrid.getCoord(i);
		auto expectedValue = coord.get(0);

		std::cout << "sparseGrid(" << coord.get(0) << "," << coord.get(1) << ") = "
				  << sparseGrid.template get<0>(coord)
				  << " == "
				  << expectedValue
				  << " == "
				  << output.template get<0>(i) << " = output(" << i << ")"
				  << std::endl;
		match &= output.template get<0>(i) == sparseGrid.template get<0>(coord);
		match &= output.template get<0>(i) == expectedValue;
	}

	BOOST_REQUIRE_EQUAL(match, true);
}

BOOST_AUTO_TEST_CASE(testInsert3D)
{
	constexpr unsigned int dim = 3;
	constexpr unsigned int blockEdgeSize = 4;
	constexpr unsigned int dataBlockSize = blockEdgeSize * blockEdgeSize;
	typedef aggregate<DataBlock<float, dataBlockSize>> AggregateSGT;
	typedef aggregate<float> AggregateOutT;

	dim3 gridSize(2, 2, 2);
	dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize, blockEdgeSize);

	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateOutT, blockEdgeSize> sparseGrid(blockGeometry);

	sparseGrid.template setBackgroundValue<0>(666);

	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);

	insertValues<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel());

	mgpu::ofp_context_t ctx;
	sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	// Get output
	openfpm::vector_gpu<AggregateOutT> output;
	output.resize(sparseGrid.dim3SizeToInt(gridSize) * 64);

	copyBlocksToOutput<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	sparseGrid.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
		auto coord = sparseGrid.getCoord(i);
		auto expectedValue = coord.get(0);

		match &= output.template get<0>(i) == sparseGrid.template get<0>(coord);
		match &= output.template get<0>(i) == expectedValue;
	}

	BOOST_REQUIRE_EQUAL(match, true);
}

BOOST_AUTO_TEST_CASE(testTagBoundaries)
{

	constexpr unsigned int dim = 2;
	constexpr unsigned int blockEdgeSize = 8;
	typedef aggregate<float> AggregateT;

	dim3 gridSize(2, 2);
	dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64> sparseGrid(blockGeometry);

	sparseGrid.template setBackgroundValue<0>(666);
	mgpu::ofp_context_t ctx;

	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
	dim3 pt1(0, 0, 0);
	CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt1, 1);
	dim3 pt2(6, 6, 0);
	CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt2, 1);
	dim3 pt3(7, 6, 0);
	CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt3, 1);
	sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
	dim3 pt4(8, 6, 0);
	CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt4, 1);
	sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	/////////
	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
	for (int y = 9; y <= 11; y++)
	{
		dim3 pt1(6, y, 0);
		CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt1, 1);
		dim3 pt2(7, y, 0);
		CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt2, 1);
	}
	sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
	for (int y = 9; y <= 11; y++)
	{
		dim3 pt1(8, y, 0);
		CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt1, 1);
		dim3 pt2(9, y, 0);
		CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt2, 1);
	}
	sparseGrid.template flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

//        sparseGrid.hostToDevice(); //just sync masks
	sparseGrid.deviceToHost(); //just sync masks
	sparseGrid.deviceToHost<0>();

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
	// Now tag the boundaries
	sparseGrid.tagBoundaries(ctx);

	// Get output
	openfpm::vector_gpu<AggregateT> output;
	output.resize(4 * 64);

	CUDA_LAUNCH_DIM3((copyToOutputIfPadding<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	sparseGrid.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
		auto coord = sparseGrid.getCoord(i);
		auto expectedValue =
				 ( i == 0
				|| i == 54
				|| i == 55
				|| i == 112
				|| i == 142 || i == 143 || i == 200 || i == 201 // (6,9), (7,9), (8,9), (9,9)
				|| i == 150 || i == 209 // (6,10), (9,10)
				|| i == 158 || i == 159 || i == 216 || i == 217 // (6,11), (7,11), (8,11), (9,11)
				 ) ? 1 : 0;

		std::cout
				<< "sparseGrid(" << coord.get(0) << "," << coord.get(1) << ") = "
				<< sparseGrid.template get<0>(coord) << " | "
				<< expectedValue
				<< " == "
				<< output.template get<0>(i) << " = output(" << i << ")"
				<< std::endl;
		match &= output.template get<0>(i) == expectedValue;
	}

	BOOST_REQUIRE_EQUAL(match, true);
}

BOOST_AUTO_TEST_CASE(testTagBoundaries2)
{
	constexpr unsigned int dim = 2;
	constexpr unsigned int blockEdgeSize = 8;
	typedef aggregate<float> AggregateT;

	dim3 gridSize(2, 2);
	dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64> sparseGrid(blockGeometry);

	sparseGrid.template setBackgroundValue<0>(666);
	mgpu::ofp_context_t ctx;

	///////
	{
		sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
		dim3 ptd1(6, 6, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd1, 1);
		dim3 ptd2(6, 7, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd2, 1);
		dim3 ptd3(7, 6, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd3, 1);
		dim3 ptd4(7, 7, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd4, 1);
		sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	}
	{
		sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
		dim3 ptd1(8, 6, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd1, 1);
		dim3 ptd2(9, 6, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd2, 1);
		dim3 ptd3(8, 7, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd3, 1);
		dim3 ptd4(9, 7, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd4, 1);
		sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	}
	{
		sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
		dim3 ptd1(6, 8, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd1, 1);
		dim3 ptd2(7, 8, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd2, 1);
		dim3 ptd3(6, 9, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd3, 1);
		dim3 ptd4(7, 9, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd4, 1);
		sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	}
	{
		sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
		dim3 ptd1(8, 8, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd1, 1);
		dim3 ptd2(8, 9, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd2, 1);
		dim3 ptd3(9, 8, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd3, 1);
		dim3 ptd4(9, 9, 0);
		insertOneValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), ptd4, 1);
		sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	}
	///////

	sparseGrid.deviceToHost(); //just sync masks

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
	// Now tag the boundaries
	sparseGrid.tagBoundaries(ctx);

	// Get output
	openfpm::vector_gpu<AggregateT> output;
	output.resize(4 * 64);

	copyToOutputIfPadding<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	sparseGrid.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
		auto coord = sparseGrid.getCoord(i);
		auto expectedValue =
				 (
						 i == 54 || i == 55 || i == 62 // (6,6), (7,6), (6,7)
					  || i == 134 || i == 142 || i == 143 // (6,8), (6,9), (7,9)
					  || i == 112 || i == 113 || i == 121 // (8,6), (9,6), (9,7)
					  || i == 200 || i == 193 || i == 201 // (8,9), (9,8), (9,9)
				 ) ? 1 : 0;

		std::cout
				<< "sparseGrid(" << coord.get(0) << "," << coord.get(1) << "," << coord.get(2) << ") = "
				<< sparseGrid.template get<0>(coord) << " | "
				<< expectedValue
				<< " == "
				<< output.template get<0>(i) << " = output(" << i << ")"
				<< std::endl;
		match &= output.template get<0>(i) == expectedValue;
	}

	BOOST_REQUIRE_EQUAL(match, true);
}

BOOST_AUTO_TEST_CASE(testStencilHeat)
{
	printf("\n");

	constexpr unsigned int dim = 2;
	constexpr unsigned int blockEdgeSize = 8;
	typedef aggregate<float,float> AggregateT;

	dim3 gridSize(2, 2);
	dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	// Insert values on the grid
	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
	insertConstantValue<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), 0);
	sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
	sparseGrid.tagBoundaries(ctx);

    hipDeviceSynchronize();

    sparseGrid.template applyBoundaryStencils<BoundaryStencilSetXRescaled<dim,0,0>>(0.0 ,gridSize.x * blockEdgeSize, 0.0, 10.0);

	hipDeviceSynchronize();

        // Now apply the laplacian operator
	const unsigned int maxIter = 1000;
//    const unsigned int maxIter = 100;
	for (unsigned int iter=0; iter<maxIter; ++iter)
	{
		sparseGrid.applyStencils<HeatStencil<dim, 0, 1>>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();
        sparseGrid.applyStencils<HeatStencil<dim, 1, 0>>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();
	}

	sparseGrid.deviceToHost<0,1>();

	// Get output
	openfpm::vector_gpu<AggregateT> output;
	output.resize(4 * 64);

	copyBlocksToOutput<0> <<< gridSize, blockSizeInsert >>> (sparseGrid.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	sparseGrid.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
		auto coord = sparseGrid.getCoord(i);
		float expectedValue = 10.0 * coord.get(0) / (gridSize.x * blockEdgeSize - 1);

		std::cout
				<< "sparseGrid(" << coord.get(0) << "," << coord.get(1) << ") = "
				<< sparseGrid.template get<0>(coord) << " | "
				<< expectedValue
				<< " == "
				<< output.template get<0>(i) << " = output(" << i << ")"
				<< std::endl;
		match &= fabs(output.template get<0>(i) - expectedValue) < 1e-2;

	}

	BOOST_REQUIRE_EQUAL(match, true);
//        BOOST_REQUIRE_CLOSE(output.template get<0>(255), 3.20309591e-05, 1e-6);
}

BOOST_AUTO_TEST_CASE(testStencilHeatInsert)
{
	printf("\n");

	constexpr unsigned int dim = 2;
	constexpr unsigned int blockEdgeSize = 8;
	typedef aggregate<float,float> AggregateT;

	dim3 gridSize(2, 2);
	dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	// Insert values on the grid
	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
	CUDA_LAUNCH_DIM3((insertConstantValue2<0,1>),gridSize, blockSizeInsert,sparseGrid.toKernel(), 0);
	sparseGrid.flush < smax_< 0 >, smax_<1>> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
	CUDA_LAUNCH_DIM3((insertBoundaryValuesHeat<0>),gridSize, blockSizeInsert,sparseGrid.toKernel());
	sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!

//        // Now tag the boundaries
	sparseGrid.tagBoundaries(ctx);

	sparseGrid.template deviceToHost<0,1>();

	// Now apply the laplacian operator
	const unsigned int maxIter = 1000;
//        const unsigned int maxIter = 10;
	for (unsigned int iter=0; iter<maxIter; ++iter)
	{
		sparseGrid.applyStencils<HeatStencil<dim, 0,0>>(STENCIL_MODE_INSERT, 0.1);
	}


	// Get output
	openfpm::vector_gpu<AggregateT> output;
	output.resize(4 * 64);

	CUDA_LAUNCH_DIM3((copyBlocksToOutput<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	sparseGrid.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
		grid_key_dx<dim, int> coord = sparseGrid.getCoord(i);
		float expectedValue = 10.0 * coord.get(0) / (gridSize.x * blockEdgeSize - 1);

		unsigned int check = sparseGrid.getLinId(coord);

		std::cout
				<< "invLinId=" << check << ", "
//                    << "sparseGrid(" << coord.get(0) << "," << coord.get(1) << "," << coord.get(2) << ") = "
				<< "sparseGrid(" << coord.get(0) << "," << coord.get(1) << ") = "
				<< sparseGrid.template get<0>(coord) << " | "
				<< expectedValue
				<< " == "
				<< output.template get<0>(i) << " = output(" << i << ")"
				<< std::endl;
		match &= fabs(output.template get<0>(i) - expectedValue) < 1e-2;

	}

	BOOST_REQUIRE_EQUAL(match, true);
//        BOOST_REQUIRE_CLOSE(output.template get<0>(255), 3.20309591e-05, 1e-6);
}

template<typename sparsegrid_type>
__global__ void sparse_grid_get_test(sparsegrid_type sparseGrid, grid_key_dx<3> key, float * data)
{
	*data = sparseGrid.template get<0>(key);
}

BOOST_AUTO_TEST_CASE(testFlushInsert)
{
	printf("\n");

	constexpr unsigned int dim = 3;
	constexpr unsigned int blockEdgeSize = 4;
	typedef aggregate<float,float> AggregateT;


	size_t sz[] = {137,100,57};

	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64> sparseGrid(sz);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	sparseGrid.insertFlush<0>(grid_key_dx<3>({3,6,7})) = 2.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({13,16,17})) = 3.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({13,46,27})) = 4.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({36,63,11})) = 5.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({37,96,47})) = 6.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({130,56,37})) = 7.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({131,76,17})) = 8.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({36,86,27})) = 9.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({34,36,7})) = 10.0;

	////// Add points in the same blocks

	sparseGrid.insertFlush<0>(grid_key_dx<3>({4,6,7})) = 2.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({12,16,17})) = 3.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({12,46,27})) = 4.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({35,63,11})) = 5.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({38,96,47})) = 6.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({131,56,37})) = 7.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({132,76,17})) = 8.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({37,86,27})) = 9.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({35,36,7})) = 10.0;

	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({3,6,7})),2.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({13,16,17})),3.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({13,46,27})),4.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({36,63,11})),5.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({37,96,47})),6.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({130,56,37})),7.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({131,76,17})),8.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({36,86,27})),9.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({34,36,7})),10.0);

	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({4,6,7})),2.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({12,16,17})),3.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({12,46,27})),4.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({35,63,11})),5.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({38,96,47})),6.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({131,56,37})),7.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({132,76,17})),8.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({37,86,27})),9.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({35,36,7})),10.0);

	sparseGrid.template hostToDevice<0>();

	// Check on device I can get information

	CudaMemory mem;

	mem.allocate(sizeof(float));

	grid_key_dx<3> key({3,6,7});

	sparse_grid_get_test<<<1,1>>>(sparseGrid.toKernel(),key,(float *)mem.getDevicePointer());

	mem.deviceToHost();

	BOOST_REQUIRE_EQUAL(*(float *)mem.getPointer(),2.0);

	grid_key_dx<3> key2({131,76,17});

	sparse_grid_get_test<<<1,1>>>(sparseGrid.toKernel(),key2,(float *)mem.getDevicePointer());

	mem.deviceToHost();

	BOOST_REQUIRE_EQUAL(*(float *)mem.getPointer(),8.0);
}

struct conv_coeff
{
	float coeff[3][3][3];
};

template<unsigned int dim, unsigned int p_src, unsigned int p_dst>
struct Conv3x3x3
{
    // This is an example of a laplacian smoothing stencil to apply using the apply stencil facility of SparseGridGpu

	typedef NNFull<dim> stencil_type;

    static constexpr unsigned int supportRadius = 1;

    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __device__ void stencil(
            SparseGridT & sparseGrid,
            const unsigned int dataBlockId,
            openfpm::sparse_index<unsigned int> dataBlockIdPos,
            unsigned int offset,
            grid_key_dx<dim, int> & pointCoord,
            DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore,
            bool applyStencilHere,
            conv_coeff & cc)
    {
        typedef typename SparseGridT::AggregateBlockType AggregateT;
        typedef ScalarTypeOf<AggregateT, p_src> ScalarT;

        constexpr unsigned int enlargedBlockSize = IntPow<
                SparseGridT::getBlockEdgeSize() + 2 * supportRadius, dim>::value;

        __shared__ ScalarT enlargedBlock[enlargedBlockSize];

        sparseGrid.template loadGhostBlock<p_src>(dataBlockLoad,dataBlockIdPos,enlargedBlock);

        __syncthreads();

        if (applyStencilHere)
        {
            const auto coord = sparseGrid.getCoordInEnlargedBlock(offset);
            const auto linId = sparseGrid.getLinIdInEnlargedBlock(offset);
            ScalarT tot = 0.0;
            for (int i = 0; i < dim; ++i)
            {
                for (int j = 0; j < dim; ++j)
                {
                    for (int k = 0; k < dim; ++k)
                    {
                    	grid_key_dx<dim,int> key;

                    	key.set_d(0,i-1);
                    	key.set_d(1,j-1);
                    	key.set_d(2,k-1);

                    	auto nPlusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, key);
                    	tot += enlargedBlock[nPlusId] * cc.coeff[i][j][k];
                    }
                }
            }

            dataBlockStore.template get<p_dst>()[offset] = tot;
        }
    }

    template <typename SparseGridT, typename CtxT>
    static inline void __host__ flush(SparseGridT & sparseGrid, CtxT & ctx)
    {
        sparseGrid.template flush <smax_<0>> (ctx, flush_type::FLUSH_ON_DEVICE);
    }
};

template<unsigned int dim, unsigned int p_src, unsigned int p_dst>
struct Conv3x3x3_noshared
{
    // This is an example of a laplacian smoothing stencil to apply using the apply stencil facility of SparseGridGpu

	typedef NNFull<dim> stencil_type;

    static constexpr unsigned int supportRadius = 1;

    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __device__ void stencil(
            SparseGridT & sparseGrid,
            const unsigned int dataBlockId,
            openfpm::sparse_index<unsigned int> dataBlockIdPos,
            unsigned int offset,
            grid_key_dx<dim, int> & pointCoord,
            DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore,
            bool applyStencilHere,
            conv_coeff & cc)
    {
        typedef typename SparseGridT::AggregateBlockType AggregateT;
        typedef ScalarTypeOf<AggregateT, p_src> ScalarT;

        __syncthreads();

        __shared__ block_offset<int> pos[BLOCK_SIZE_STENCIL];

        if (applyStencilHere)
        {
            ScalarT tot = 0.0;
            for (int i = 0; i < dim; ++i)
            {
                for (int j = 0; j < dim; ++j)
                {
                    for (int k = 0; k < dim; ++k)
                    {

                    	grid_key_dx<dim,int> key;

                    	key.set_d(0,k-1);
                    	key.set_d(1,j-1);
                    	key.set_d(2,i-1);

                    	pos[threadIdx.x] = sparseGrid.template getNNPoint<stencil_type>(dataBlockIdPos, offset, key);

                    	tot += sparseGrid.template get<p_src>(pos[threadIdx.x]) * cc.coeff[i][j][k];
                    }
                }
            }

            dataBlockStore.template get<p_dst>()[offset] = tot;
        }
    }

    template <typename SparseGridT, typename CtxT>
    static inline void __host__ flush(SparseGridT & sparseGrid, CtxT & ctx)
    {
        sparseGrid.template flush <smax_<0>> (ctx, flush_type::FLUSH_ON_DEVICE);
    }
};

template<typename SparseGridZ>
void test_convolution_3x3x3()
{
	size_t sz[] = {1000,1000,1000};

	SparseGridZ sparseGrid(sz);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	// now create 3 3D sphere

    grid_key_dx<3,int> start({256,256,256});

    dim3 gridSize(32,32,32);

    // Insert values on the grid
    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
    CUDA_LAUNCH_DIM3((insertSphere3D_radius<0>),
            gridSize, dim3(SparseGridZ::blockEdgeSize_*SparseGridZ::blockEdgeSize_*SparseGridZ::blockEdgeSize_,1,1),
            sparseGrid.toKernel(), start,64, 56, 1);

    sparseGrid.template flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.template findNeighbours<NNFull<3>>(); // Pre-compute the neighbours pos for each block!

    sparseGrid.template setNNType<NNFull<3>>();
    sparseGrid.template tagBoundaries<NNFull<3>>(ctx);

    conv_coeff cc;

    for (int i = 0 ; i < 3 ; i++)
    {
    	for (int j = 0 ; j < 3 ; j++)
    	{
    		for (int k = 0 ; k < 3 ; k++)
    		{
    			cc.coeff[k][j][i] = 1.0;
    		}
    	}
    }


    sparseGrid.template applyStencils<Conv3x3x3<3,0,1>>(STENCIL_MODE_INPLACE,cc);

    sparseGrid.template deviceToHost<0,1>();

	auto & bm = sparseGrid.private_get_blockMap();
	auto & dataVector = bm.getDataBuffer();

	bool match = true;

    for (size_t i = 0 ; i < dataVector.size() ; i++)
    {
        for (size_t j = 0 ; j < 64 ; j++)
        {
			if (dataVector.template get<2>(i)[j] == 1)
			{
				match &= dataVector.template get<0>(i)[j]*27 == dataVector.template get<1>(i)[j];
			}
        }
    }

    BOOST_REQUIRE_EQUAL(match,true);
}

template<typename SparseGridZ>
void test_convolution_3x3x3_no_shared()
{
	size_t sz[] = {1000,1000,1000};

	SparseGridZ sparseGrid(sz);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	// now create 3 3D sphere

    grid_key_dx<3,int> start({256,256,256});

    dim3 gridSize(32,32,32);

    // Insert values on the grid
    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
    CUDA_LAUNCH_DIM3((insertSphere3D_radius<0>),
            gridSize, dim3(SparseGridZ::blockEdgeSize_*SparseGridZ::blockEdgeSize_*SparseGridZ::blockEdgeSize_,1,1),
            sparseGrid.toKernel(), start,64, 56, 1);

    sparseGrid.template flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.template findNeighbours<NNFull<3>>(); // Pre-compute the neighbours pos for each block!

    sparseGrid.template setNNType<NNFull<SparseGridZ::dims>>();
    sparseGrid.template tagBoundaries<NNFull<3>>(ctx,tag_boundaries::CALCULATE_EXISTING_POINTS);

    conv_coeff cc;

    for (int i = 0 ; i < 3 ; i++)
    {
    	for (int j = 0 ; j < 3 ; j++)
    	{
    		for (int k = 0 ; k < 3 ; k++)
    		{
    			cc.coeff[k][j][i] = 1.0;
    		}
    	}
    }

    sparseGrid.template applyStencils<Conv3x3x3_noshared<SparseGridZ::dims,0,1>>(STENCIL_MODE_INPLACE_NO_SHARED,cc);

    sparseGrid.template deviceToHost<0,1>();

	auto & bm = sparseGrid.private_get_blockMap();
	auto & dataVector = bm.getDataBuffer();

	bool match = true;

    for (size_t i = 0 ; i < dataVector.size() ; i++)
    {
        for (size_t j = 0 ; j < 64 ; j++)
        {
			if (dataVector.template get<2>(i)[j] == 1)
			{
				match &= dataVector.template get<0>(i)[j]*27 == dataVector.template get<1>(i)[j];
			}
        }
    }

    BOOST_REQUIRE_EQUAL(match,true);
}

BOOST_AUTO_TEST_CASE(test3x3x3convolution_no_shared)
{
	constexpr unsigned int dim = 3;
	constexpr unsigned int blockEdgeSize = 4;
	typedef aggregate<float,float> AggregateT;

	test_convolution_3x3x3_no_shared<SparseGridGpu<dim, AggregateT, blockEdgeSize, 64, long int>>();
}

BOOST_AUTO_TEST_CASE(test3x3x3convolution_no_shared_z_morton)
{
	constexpr unsigned int dim = 3;
	constexpr unsigned int blockEdgeSize = 4;
	typedef aggregate<float,float> AggregateT;

	test_convolution_3x3x3_no_shared<SparseGridGpu_z<dim, AggregateT, blockEdgeSize, 64, long int>>();
}

BOOST_AUTO_TEST_CASE(test3x3x3convolution)
{
	constexpr unsigned int dim = 3;
	constexpr unsigned int blockEdgeSize = 4;
	typedef aggregate<float,float> AggregateT;

	test_convolution_3x3x3<SparseGridGpu<dim, AggregateT, blockEdgeSize, 64, long int>>();
}

BOOST_AUTO_TEST_CASE(test3x3x3convolution_morton_z)
{
	constexpr unsigned int dim = 3;
	constexpr unsigned int blockEdgeSize = 4;
	typedef aggregate<float,float> AggregateT;

	test_convolution_3x3x3<SparseGridGpu_z<dim, AggregateT, blockEdgeSize, 64, long int>>();
}

BOOST_AUTO_TEST_CASE(test_sparse_grid_iterator_sub_host)
{
	constexpr unsigned int dim = 3;
	constexpr unsigned int blockEdgeSize = 4;
	typedef aggregate<float, float> AggregateT;

	size_t sz[3] = {768,768,768};
	dim3 gridSize(32,32,32);

	grid_smb<dim, blockEdgeSize> blockGeometry(sz);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64, long int> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	///// Insert sparse content, a set of 3 hollow spheres /////
	// Sphere 1
	grid_key_dx<3,int> start1({256,256,256});
	sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
	CUDA_LAUNCH_DIM3((insertSphere3D<0>),
					 gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
					 sparseGrid.toKernel(), start1, 32, 0, 1);

	sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.template deviceToHost<0>();

	bool match = true;

	int count = 0;

	grid_key_dx<3> start({303,303,303});
	grid_key_dx<3> stop({337,337,337});

	auto it = sparseGrid.getIterator(start,stop);

	while (it.isNext())
	{
		auto key = it.get();

		match &= sparseGrid.template get<0>(key) == 1.0;

		sparseGrid.template get<0>(key) = 5.0;

		count++;

		++it;
	}

	BOOST_REQUIRE_EQUAL(match,true);
	BOOST_REQUIRE_EQUAL(count,42875);
}




BOOST_AUTO_TEST_CASE(test_sparse_grid_iterator_host)
{
	constexpr unsigned int dim = 3;
	constexpr unsigned int blockEdgeSize = 4;
	typedef aggregate<float, float> AggregateT;

	size_t sz[3] = {512,512,512};
	dim3 gridSize(32,32,32);

	grid_smb<dim, blockEdgeSize> blockGeometry(sz);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64, long int> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	///// Insert sparse content, a set of 3 hollow spheres /////
	// Sphere 1
	grid_key_dx<3,int> start1({256,256,256});
	sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
	CUDA_LAUNCH_DIM3((insertSphere3D<0>),
					 gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
					 sparseGrid.toKernel(), start1, 64, 32, 1);

	sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.template deviceToHost<0>();

	bool match = true;

	int count = 0;

	auto it = sparseGrid.getIterator();

	while (it.isNext())
	{
		auto key = it.get();

		match &= sparseGrid.template get<0>(key) == 1.0;
		//unsigned char bl = sparseGrid.template get<2>(key);

		count++;

		++it;
	}

	BOOST_REQUIRE_EQUAL(sparseGrid.countExistingElements(),count);
	BOOST_REQUIRE_EQUAL(match,true);
}

BOOST_AUTO_TEST_CASE(test_pack_request)
{
	size_t sz[] = {1000,1000,1000};

	constexpr int blockEdgeSize = 4;
	constexpr int dim = 3;

	typedef SparseGridGpu<dim, aggregate<float>, blockEdgeSize, 64, long int> SparseGridZ;

	SparseGridZ sparseGrid(sz);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	// now create a 3D sphere

    grid_key_dx<3,int> start({256,256,256});

    dim3 gridSize(32,32,32);

    // Insert values on the grid
    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
    CUDA_LAUNCH_DIM3((insertSphere3D_radius<0>),
            gridSize, dim3(SparseGridZ::blockEdgeSize_*SparseGridZ::blockEdgeSize_*SparseGridZ::blockEdgeSize_,1,1),
            sparseGrid.toKernel(), start,64, 56, 1);

    sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
    sparseGrid.template deviceToHost<0>();

    size_t cnt = sparseGrid.countExistingElements();

    size_t req = 0;
    sparseGrid.packRequest<0>(req,ctx);

    size_t tot = 8 +                // how many chunks
    		     sparseGrid.private_get_index_array().size()*16 + 8 +// store the scan + chunk indexes
    		     cnt*(sizeof(float) + 2); // how much data

    BOOST_REQUIRE_EQUAL(req,tot);
}

BOOST_AUTO_TEST_CASE(test_pack_request_with_iterator)
{
	size_t sz[] = {1000,1000,1000};

	constexpr int blockEdgeSize = 4;
	constexpr int dim = 3;

	typedef SparseGridGpu<dim, aggregate<float>, blockEdgeSize, 64, long int> SparseGridZ;

	SparseGridZ sparseGrid(sz);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	// now create a 3D sphere

    grid_key_dx<3,int> start({256,256,256});

    dim3 gridSize(32,32,32);

    // Insert values on the grid
    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
    CUDA_LAUNCH_DIM3((insertSphere3D_radius<0>),
            gridSize, dim3(SparseGridZ::blockEdgeSize_*SparseGridZ::blockEdgeSize_*SparseGridZ::blockEdgeSize_,1,1),
            sparseGrid.toKernel(), start,64, 56, 1);

    sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    size_t req = 0;
    sparseGrid.packReset();

    {
    grid_key_dx<3> start1({0,0,0});
    grid_key_dx<3> stop1({321,999,999});

    grid_key_dx<3> start2({322,0,0});
    grid_key_dx<3> stop2({999,999,999});

    auto it1 = sparseGrid.getIterator(start1,stop1);
    sparseGrid.template packRequest<0>(it1,req);

    auto it2 = sparseGrid.getIterator(start2,stop2);
    sparseGrid.template packRequest<0>(it2,req);

    sparseGrid.template packCalculate<0>(req,ctx);
    }

    sparseGrid.template deviceToHost<0>();


    size_t cnt = sparseGrid.countExistingElements();

    size_t tot = 8 +                // how many chunks
    		     sparseGrid.private_get_index_array().size()*16 + 8 +// store the scan + chunk indexes
    		     cnt*(sizeof(float) + 2); // how much data

    std::cout << __FILE__ << ":" << __LINE__ << "  To fix this" << std::endl;
//    BOOST_REQUIRE_EQUAL(req,tot);

    ////////////////////////////////// test something else

    req = 0;
    sparseGrid.packReset();

    {
    grid_key_dx<3> start1({0,0,0});
    grid_key_dx<3> stop1({999,999,999});

    auto it1 = sparseGrid.getIterator(start1,stop1);
    sparseGrid.template packRequest<0>(it1,req);

    auto it2 = sparseGrid.getIterator(start1,stop1);
    sparseGrid.template packRequest<0>(it2,req);

    sparseGrid.template packCalculate<0>(req,ctx);
    }


    tot = 8 +                // how many chunks
    		     sparseGrid.private_get_index_array().size()*16 + 8 + // store the scan + chunk indexes
    		     2*cnt*(sizeof(float) + 2); // how much data

//    BOOST_REQUIRE_EQUAL(req,tot);
}

BOOST_AUTO_TEST_CASE(sparsegridgpu_remove_test)
{
	size_t sz[] = {1000,1000,1000};

	constexpr int blockEdgeSize = 4;
	constexpr int dim = 3;

	typedef SparseGridGpu<dim, aggregate<float>, blockEdgeSize, 64, long int> SparseGridZ;

	SparseGridZ sparseGrid(sz);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	// now create a 3D sphere

    grid_key_dx<3,int> start({256,256,256});

    dim3 gridSize(32,32,32);

    // Insert values on the grid
    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
    CUDA_LAUNCH_DIM3((insertSphere3D_radius<0>),
            gridSize, dim3(SparseGridZ::blockEdgeSize_*SparseGridZ::blockEdgeSize_*SparseGridZ::blockEdgeSize_,1,1),
            sparseGrid.toKernel(), start,64, 56, 1);

    sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    // remove the center

    Box<3,unsigned int> remove_section1({310,0,0},{330,999,999});
    Box<3,unsigned int> remove_section2({0,310,0},{999,330,999});
    Box<3,unsigned int> remove_section3({0,0,310},{999,999,330});

    sparseGrid.remove(remove_section1);
    sparseGrid.remove(remove_section2);
    sparseGrid.remove(remove_section3);

    sparseGrid.removeAddUnpackFinalize<>(ctx);

    sparseGrid.deviceToHost<0>();

    // Check we have the sphere points with the exception of the box

    auto it = sparseGrid.getIterator();

    bool match = true;

    while (it.isNext())
    {
    	auto p = it.get();

    	Point<3,size_t> pt = p.toPoint();

    	// Calculate redius
    	float radius = sqrt((pt.get(0) - 320)*(pt.get(0) - 320) +
    						(pt.get(1) - 320)*(pt.get(1) - 320) +
    						(pt.get(2) - 320)*(pt.get(2) - 320));


    	if (radius < 55.99 || radius > 64.01)
    	{match &= false;}

    	if (remove_section1.isInside(pt) == true)
    	{match &= false;}

    	if (remove_section2.isInside(pt) == true)
    	{match &= false;}

    	if (remove_section3.isInside(pt) == true)
    	{match &= false;}

    	++it;
    }

    BOOST_REQUIRE_EQUAL(match,true);
}

template<typename SG_type>
void pack_unpack_test(SG_type & sparseGridDst, SG_type & sparseGridSrc,
		Box<3,size_t> & box1_dst,
		Box<3,size_t> & box2_dst,
		Box<3,size_t> & box3_dst,
		Box<3,size_t> & box4_dst,
		mgpu::ofp_context_t & ctx,
		bool test_pack)
{
    Box<3,size_t> box1_src({256,256,256},{273,390,390});
    Box<3,size_t> box2_src({320,256,256},{337,390,390});

    // And two vertical sections

    Box<3,size_t> box3_src({256,256,256},{273,320,390});
    Box<3,size_t> box4_src({320,256,256},{337,320,390});

    // Now we calculate the memory required to pack

    sparseGridSrc.packReset();

    size_t req = 0;
    auto sub_it = sparseGridSrc.getIterator(box1_src.getKP1(),box1_src.getKP2());
    sparseGridSrc.template packRequest<0,1>(sub_it,req);

    sub_it = sparseGridSrc.getIterator(box2_src.getKP1(),box2_src.getKP2());
    sparseGridSrc.template packRequest<0,1>(sub_it,req);

    sub_it = sparseGridSrc.getIterator(box3_src.getKP1(),box3_src.getKP2());
    sparseGridSrc.template packRequest<0,1>(sub_it,req);

    sub_it = sparseGridSrc.getIterator(box4_src.getKP1(),box4_src.getKP2());
    sparseGridSrc.template packRequest<0,1>(sub_it,req);

    sparseGridSrc.template packCalculate<0,1>(req,ctx);

    CudaMemory mem;
    mem.resize(req);

	// Create an object of preallocated memory for properties
	ExtPreAlloc<CudaMemory> & prAlloc_prp = *(new ExtPreAlloc<CudaMemory>(req,mem));

	prAlloc_prp.incRef();

	// Pack information
	Pack_stat sts;

    sub_it = sparseGridSrc.getIterator(box1_src.getKP1(),box1_src.getKP2());
    sparseGridSrc.template pack<0,1>(prAlloc_prp,sub_it,sts);

    sub_it = sparseGridSrc.getIterator(box2_src.getKP1(),box2_src.getKP2());
    sparseGridSrc.template pack<0,1>(prAlloc_prp,sub_it,sts);

    sub_it = sparseGridSrc.getIterator(box3_src.getKP1(),box3_src.getKP2());
    sparseGridSrc.template pack<0,1>(prAlloc_prp,sub_it,sts);

    sub_it = sparseGridSrc.getIterator(box4_src.getKP1(),box4_src.getKP2());
    sparseGridSrc.template pack<0,1>(prAlloc_prp,sub_it,sts);


	sparseGridSrc.template packFinalize<0,1>(prAlloc_prp,sts);

	// Now we analyze the package

	if (test_pack == true)
	{
		size_t ncnk = *(size_t *)mem.getPointer();
		BOOST_REQUIRE_EQUAL(ncnk,1107);
		size_t actual_offset = ncnk*sizeof(size_t) + sizeof(size_t) + 2*3*sizeof(int);
		mem.deviceToHost(actual_offset + ncnk*sizeof(unsigned int),actual_offset + ncnk*sizeof(unsigned int) + sizeof(unsigned int));
		unsigned int n_pnt = *(unsigned int *)((unsigned char *)mem.getPointer() + actual_offset + ncnk*sizeof(unsigned int));
		BOOST_REQUIRE_EQUAL(n_pnt,41003);

		actual_offset += align_number(sizeof(size_t),(ncnk+1)*sizeof(unsigned int));
		actual_offset += align_number(sizeof(size_t),n_pnt*(16));
		actual_offset += align_number(sizeof(size_t),n_pnt*sizeof(short int));


		ncnk = *(size_t *)((unsigned char *)mem.getPointer() + actual_offset);
		BOOST_REQUIRE_EQUAL(ncnk,1420);
		actual_offset += ncnk*sizeof(size_t) + sizeof(size_t) + 2*3*sizeof(int);
		mem.deviceToHost(actual_offset + ncnk*sizeof(unsigned int),actual_offset + ncnk*sizeof(unsigned int) + sizeof(unsigned int));
		n_pnt = *(unsigned int *)((unsigned char *)mem.getPointer() + actual_offset + ncnk*sizeof(unsigned int));
		BOOST_REQUIRE_EQUAL(n_pnt,54276);

		actual_offset += align_number(sizeof(size_t),(ncnk+1)*sizeof(unsigned int));
		actual_offset += align_number(sizeof(size_t),n_pnt*(16));
		actual_offset += align_number(sizeof(size_t),n_pnt*sizeof(short int));

		ncnk = *(size_t *)((unsigned char *)mem.getPointer() + actual_offset);
		BOOST_REQUIRE_EQUAL(ncnk,610);
		actual_offset += ncnk*sizeof(size_t) + sizeof(size_t) + 2*3*sizeof(int);
		mem.deviceToHost(actual_offset + ncnk*sizeof(unsigned int),actual_offset + ncnk*sizeof(unsigned int) + sizeof(unsigned int));
		n_pnt = *(unsigned int *)((unsigned char *)mem.getPointer() + actual_offset + ncnk*sizeof(unsigned int));
		BOOST_REQUIRE_EQUAL(n_pnt,20828);

		actual_offset += align_number(sizeof(size_t),(ncnk+1)*sizeof(unsigned int));
		actual_offset += align_number(sizeof(size_t),n_pnt*(16));
		actual_offset += align_number(sizeof(size_t),n_pnt*sizeof(short int));

		ncnk = *(size_t *)((unsigned char *)mem.getPointer() + actual_offset);
		BOOST_REQUIRE_EQUAL(ncnk,739);
		actual_offset += ncnk*sizeof(size_t) + sizeof(size_t) + 2*3*sizeof(int);
		mem.deviceToHost(actual_offset + ncnk*sizeof(unsigned int),actual_offset + ncnk*sizeof(unsigned int) + sizeof(unsigned int));
		n_pnt = *(unsigned int *)((unsigned char *)mem.getPointer() + actual_offset + ncnk*sizeof(unsigned int));
		BOOST_REQUIRE_EQUAL(n_pnt,27283);
	}

	prAlloc_prp.reset();

	Unpack_stat ps;

	sparseGridDst.removeAddUnpackReset();

	// sub-grid where to unpack
	auto sub2 = sparseGridDst.getIterator(box1_dst.getKP1(),box1_dst.getKP2());
	sparseGridDst.remove(box1_dst);
	sparseGridDst.template unpack<0,1>(prAlloc_prp,sub2,ps,ctx);

	sub2 = sparseGridDst.getIterator(box2_dst.getKP1(),box2_dst.getKP2());
	sparseGridDst.remove(box2_dst);
	sparseGridDst.template unpack<0,1>(prAlloc_prp,sub2,ps,ctx);

	sub2 = sparseGridDst.getIterator(box3_dst.getKP1(),box3_dst.getKP2());
	sparseGridDst.remove(box3_dst);
	sparseGridDst.template unpack<0,1>(prAlloc_prp,sub2,ps,ctx);

	sub2 = sparseGridDst.getIterator(box4_dst.getKP1(),box4_dst.getKP2());
	sparseGridDst.remove(box4_dst);
	sparseGridDst.template unpack<0,1>(prAlloc_prp,sub2,ps,ctx);

	sparseGridDst.template removeAddUnpackFinalize<0,1>(ctx);

	sparseGridDst.template deviceToHost<0,1>();
}

BOOST_AUTO_TEST_CASE(sparsegridgpu_pack_unpack)
{
	size_t sz[] = {1000,1000,1000};

	constexpr int blockEdgeSize = 4;
	constexpr int dim = 3;

    Box<3,size_t> box1_dst({256,256,256},{273,390,390});
    Box<3,size_t> box2_dst({274,256,256},{291,390,390});

    Box<3,size_t> box3_dst({300,256,256},{317,320,390});
    Box<3,size_t> box4_dst({320,256,256},{337,320,390});

	typedef SparseGridGpu<dim, aggregate<float,float[3]>, blockEdgeSize, 64, long int> SparseGridZ;

	SparseGridZ sparseGridSrc(sz);
	SparseGridZ sparseGridDst(sz);
	mgpu::ofp_context_t ctx;
	sparseGridSrc.template setBackgroundValue<0>(0);
	sparseGridDst.template setBackgroundValue<0>(0);

	// now create a 3D sphere

    grid_key_dx<3,int> start({256,256,256});

    dim3 gridSize(32,32,32);

    // Insert values on the grid
    sparseGridSrc.setGPUInsertBuffer(gridSize,dim3(1));
    CUDA_LAUNCH_DIM3((insertSphere3D_radiusV<0>),
            gridSize, dim3(SparseGridZ::blockEdgeSize_*SparseGridZ::blockEdgeSize_*SparseGridZ::blockEdgeSize_,1,1),
            sparseGridSrc.toKernel(), start,64, 56, 1);

    sparseGridSrc.flush < smax_< 0 >, smax_<1> > (ctx, flush_type::FLUSH_ON_DEVICE);

    // Now we pack two vertical sections

	pack_unpack_test(sparseGridDst,sparseGridSrc,
					 box1_dst,box2_dst,
					 box3_dst,box4_dst,
					ctx,true);

	sparseGridDst.template deviceToHost<0,1>();

	int cnt1 = 0;
	int cnt2 = 0;
	int cnt3 = 0;
	int cnt4 = 0;

	auto it = sparseGridDst.getIterator();

	bool match = true;

	while (it.isNext())
	{
		auto p = it.get();

		auto pt = p.toPoint();

		if (box1_dst.isInside(pt) == true)
		{
			++cnt1;

		    const long int x = (long int)pt.get(0) - (start.get(0) + gridSize.x / 2 * blockEdgeSize);
		    const long int y = (long int)pt.get(1) - (start.get(1) + gridSize.y / 2 * blockEdgeSize);
		    const long int z = (long int)pt.get(2) - (start.get(2) + gridSize.z / 2 * blockEdgeSize);

		    float radius = sqrt((float) (x*x + y*y + z*z));

		    bool is_active = radius < 64 && radius > 56;

		    if (is_active == true)
		    {match &= true;}
		    else
		    {match &= false;}
		}
		else if (box2_dst.isInside(pt) == true)
		{
			++cnt2;

		    const long int x = (long int)pt.get(0) - (start.get(0) - 46 + gridSize.x / 2 * blockEdgeSize);
		    const long int y = (long int)pt.get(1) - (start.get(1) + gridSize.y / 2 * blockEdgeSize);
		    const long int z = (long int)pt.get(2) - (start.get(2) + gridSize.z / 2 * blockEdgeSize);

		    float radius = sqrt((float) (x*x + y*y + z*z));

		    bool is_active = radius < 64 && radius > 56;

		    if (is_active == true)
		    {match &= true;}
		    else
		    {match &= false;}
		}
		else if (box3_dst.isInside(pt) == true)
		{
			++cnt3;

		    const long int x = (long int)pt.get(0) - (start.get(0) + 44 + gridSize.x / 2 * blockEdgeSize);
		    const long int y = (long int)pt.get(1) - (start.get(1) + gridSize.y / 2 * blockEdgeSize);
		    const long int z = (long int)pt.get(2) - (start.get(2) + gridSize.z / 2 * blockEdgeSize);

		    float radius = sqrt((float) (x*x + y*y + z*z));

		    bool is_active = radius < 64 && radius > 56;

		    if (is_active == true)
		    {match &= true;}
		    else
		    {match &= false;}
		}
		else if (box4_dst.isInside(pt) == true)
		{
			++cnt4;

		    const long int x = (long int)pt.get(0) - (start.get(0) + gridSize.x / 2 * blockEdgeSize);
		    const long int y = (long int)pt.get(1) - (start.get(1) + gridSize.y / 2 * blockEdgeSize);
		    const long int z = (long int)pt.get(2) - (start.get(2) + gridSize.z / 2 * blockEdgeSize);

		    float radius = sqrt((float) (x*x + y*y + z*z));

		    bool is_active = radius < 64 && radius > 56;

		    if (is_active == true)
		    {match &= true;}
		    else
		    {match &= false;}
		}

		++it;
	}

	BOOST_REQUIRE_EQUAL(match,true);
	BOOST_REQUIRE_EQUAL(cnt1,41003);
	BOOST_REQUIRE_EQUAL(cnt2,54276);
	BOOST_REQUIRE_EQUAL(cnt3,20828);
	BOOST_REQUIRE_EQUAL(cnt4,27283);

	// Now we remove even points

    // Insert values on the grid
    sparseGridSrc.setGPUInsertBuffer(gridSize,dim3(1));
    CUDA_LAUNCH_DIM3((removeSphere3D_even_radiusV<0>),
            gridSize, dim3(SparseGridZ::blockEdgeSize_*SparseGridZ::blockEdgeSize_*SparseGridZ::blockEdgeSize_,1,1),
            sparseGridSrc.toKernel(), start,64, 56, 1);

    pack_unpack_test(sparseGridDst,sparseGridSrc,
			 	 	 box1_dst,box2_dst,
			 	 	 box3_dst,box4_dst,
    				ctx,false);

	sparseGridDst.template deviceToHost<0,1>();

	cnt1 = 0;
	cnt2 = 0;
	cnt3 = 0;
	cnt4 = 0;

	auto it2 = sparseGridDst.getIterator();

	match = true;

	while (it2.isNext())
	{
		auto p = it2.get();

		auto pt = p.toPoint();

		if (box1_dst.isInside(pt) == true)
		{
			++cnt1;

			const long int x = (long int)pt.get(0) - (start.get(0) + gridSize.x / 2 * blockEdgeSize);
			const long int y = (long int)pt.get(1) - (start.get(1) + gridSize.y / 2 * blockEdgeSize);
			const long int z = (long int)pt.get(2) - (start.get(2) + gridSize.z / 2 * blockEdgeSize);

			float radius = sqrt((float) (x*x + y*y + z*z));

			bool is_active = radius < 64 && radius > 56;

			if (is_active == true)
			{match &= true;}
			else
			{match &= false;}
		}
		else if (box2_dst.isInside(pt) == true)
		{
			++cnt2;

			const long int x = (long int)pt.get(0) - (start.get(0) - 46 + gridSize.x / 2 * blockEdgeSize);
			const long int y = (long int)pt.get(1) - (start.get(1) + gridSize.y / 2 * blockEdgeSize);
			const long int z = (long int)pt.get(2) - (start.get(2) + gridSize.z / 2 * blockEdgeSize);

			float radius = sqrt((float) (x*x + y*y + z*z));

			bool is_active = radius < 64 && radius > 56;

			if (is_active == true)
			{match &= true;}
			else
			{match &= false;}
		}
		else if (box3_dst.isInside(pt) == true)
		{
			++cnt3;

			const long int x = (long int)pt.get(0) - (start.get(0) + 44 + gridSize.x / 2 * blockEdgeSize);
			const long int y = (long int)pt.get(1) - (start.get(1) + gridSize.y / 2 * blockEdgeSize);
			const long int z = (long int)pt.get(2) - (start.get(2) + gridSize.z / 2 * blockEdgeSize);

			float radius = sqrt((float) (x*x + y*y + z*z));

			bool is_active = radius < 64 && radius > 56;

			if (is_active == true)
			{match &= true;}
			else
			{match &= false;}
		}
		else if (box4_dst.isInside(pt) == true)
		{
			++cnt4;

			const long int x = (long int)pt.get(0) - (start.get(0) + gridSize.x / 2 * blockEdgeSize);
			const long int y = (long int)pt.get(1) - (start.get(1) + gridSize.y / 2 * blockEdgeSize);
			const long int z = (long int)pt.get(2) - (start.get(2) + gridSize.z / 2 * blockEdgeSize);

			float radius = sqrt((float) (x*x + y*y + z*z));

			bool is_active = radius < 64 && radius > 56;

			if (is_active == true)
			{match &= true;}
			else
			{match &= false;}
		}

		++it2;
	}

	BOOST_REQUIRE_EQUAL(match,true);
	BOOST_REQUIRE_EQUAL(cnt1,20520);
	BOOST_REQUIRE_EQUAL(cnt2,27152);
	BOOST_REQUIRE_EQUAL(cnt3,10423);
	BOOST_REQUIRE_EQUAL(cnt4,13649);
}

#if defined(OPENFPM_DATA_ENABLE_IO_MODULE) || defined(PERFORMANCE_TEST)

BOOST_AUTO_TEST_CASE(testSparseGridGpuOutput3DHeatStencil)
{
	constexpr unsigned int dim = 3;
	constexpr unsigned int blockEdgeSize = 4;
	typedef aggregate<float, float> AggregateT;

	size_t sz[3] = {512,512,512};
//        dim3 gridSize(128,128,128);
	dim3 gridSize(32,32,32);

	grid_smb<dim, blockEdgeSize> blockGeometry(sz);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64, long int> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	///// Insert sparse content, a set of 3 hollow spheres /////
	// Sphere 1
	grid_key_dx<3,int> start1({256,256,256});
	sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
	CUDA_LAUNCH_DIM3((insertSphere3D<0>),
					 gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
					 sparseGrid.toKernel(), start1, 64, 32, 1);
	hipDeviceSynchronize();
	sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	hipDeviceSynchronize();

	// Sphere 2
	grid_key_dx<3,int> start2({192,192,192});
	sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
	CUDA_LAUNCH_DIM3((insertSphere3D<0>),
					 gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
					 sparseGrid.toKernel(), start2, 64, 44, 1);
	hipDeviceSynchronize();
	sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	hipDeviceSynchronize();

	// Sphere 3
	grid_key_dx<3,int> start3({340,192,192});
	sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
	CUDA_LAUNCH_DIM3((insertSphere3D<0>),
					 gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
					 sparseGrid.toKernel(), start3, 20, 15, 1);
	hipDeviceSynchronize();
	sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	hipDeviceSynchronize();
	///// /////

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
	sparseGrid.tagBoundaries(ctx);

	// Now apply some boundary conditions
	sparseGrid.template applyBoundaryStencils<BoundaryStencilSetXRescaled<dim,0,0>>(
			192, 384,
			0.0, 10.0);
	hipDeviceSynchronize();

	// Now apply the laplacian operator
//        const unsigned int maxIter = 1000;
	const unsigned int maxIter = 100;
	for (unsigned int iter=0; iter<maxIter; ++iter)
	{
		for (int innerIter=0; innerIter<10; ++innerIter)
		{
			sparseGrid.applyStencils<HeatStencil<dim, 0, 1>>(STENCIL_MODE_INPLACE, 0.1);
			hipDeviceSynchronize();
			sparseGrid.applyStencils<HeatStencil<dim, 1, 0>>(STENCIL_MODE_INPLACE, 0.1);
			hipDeviceSynchronize();
		}
	}

	sparseGrid.deviceToHost<0,1>();
	sparseGrid.write("SparseGridGPU_output3DHeatStencil.vtk");
}

BOOST_AUTO_TEST_CASE(testSparseGridGpuOutput)
{
	constexpr unsigned int dim = 2;
	constexpr unsigned int blockEdgeSize = 8;
	typedef aggregate<float> AggregateT;

	size_t sz[2] = {1000000,1000000};
	dim3 gridSize(128,128);

	grid_smb<dim, blockEdgeSize> blockGeometry(sz);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64, long int> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	grid_key_dx<2,int> start({500000,500000});

	// Insert values on the grid
	sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
	CUDA_LAUNCH_DIM3((insertSphere<0>),gridSize, dim3(blockEdgeSize*blockEdgeSize,1),sparseGrid.toKernel(), start, 512, 256, 1);
	sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
	sparseGrid.tagBoundaries(ctx);

	sparseGrid.template deviceToHost<0>();

	sparseGrid.write("SparseGridGPU_output.vtk");
}

BOOST_AUTO_TEST_CASE(testSparseGridGpuOutput3D)
{
	constexpr unsigned int dim = 3;
	constexpr unsigned int blockEdgeSize = 4;
	typedef aggregate<float> AggregateT;

	size_t sz[3] = {512,512,512};
//        dim3 gridSize(128,128,128);
	dim3 gridSize(32,32,32);

	grid_smb<dim, blockEdgeSize> blockGeometry(sz);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64, long int> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	grid_key_dx<3,int> start({256,256,256});

	// Insert values on the grid
	sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
	CUDA_LAUNCH_DIM3((insertSphere3D<0>),
			gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
			sparseGrid.toKernel(), start, 64, 56, 1);
	sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	hipDeviceSynchronize();

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
	sparseGrid.tagBoundaries(ctx);

	sparseGrid.template applyBoundaryStencils<BoundaryStencilSetX<dim,0,0>>();

	hipDeviceSynchronize();

	sparseGrid.template deviceToHost<0>();

	sparseGrid.write("SparseGridGPU_output3D.vtk");

	bool test = compare("SparseGridGPU_output3D.vtk","test_data/SparseGridGPU_output3D_test.vtk");
	BOOST_REQUIRE_EQUAL(true,test);
}


#endif

BOOST_AUTO_TEST_SUITE_END()
