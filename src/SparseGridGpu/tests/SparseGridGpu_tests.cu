#include "hip/hip_runtime.h"
//
// Created by tommaso on 10/06/19.
//

#define BOOST_TEST_DYN_LINK

#define DISABLE_MPI_WRITTERS
//#define OPENFPM_DATA_ENABLE_IO_MODULE

#include <boost/test/unit_test.hpp>
#include "SparseGridGpu/SparseGridGpu.hpp"
#include "SparseGridGpu/tests/utils/SparseGridGpu_testKernels.cuh"

template<unsigned int p1 , unsigned int p2, unsigned int chunksPerBlock=1, typename SparseGridType, typename ScalarT>
__global__ void insertConstantValue2(SparseGridType sparseGrid, ScalarT value)
{
    constexpr unsigned int pMask = SparseGridType::pMask;
    typedef BlockTypeOf<typename SparseGridType::AggregateType, p1> BlockT;

    sparseGrid.init();

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    auto pos = sparseGrid.getLinId(coord);
    unsigned int dataBlockId = pos / BlockT::size;
    unsigned int offset = pos % BlockT::size;

    auto encap = sparseGrid.insertBlock(dataBlockId);
    encap.template get<p1>()[offset] = value;
    encap.template get<p2>()[offset] = value;
    BlockMapGpu_ker<>::setExist(encap.template get<pMask>()[offset]);

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType>
__global__ void insertValues(SparseGridType sparseGrid)
{
    sparseGrid.init();

    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    size_t pos = sparseGrid.getLinId(coord);

    sparseGrid.template insert<p>(coord) = x;

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType, typename ScalarT>
__global__ void insertConstantValue(SparseGridType sparseGrid, ScalarT value)
{
    sparseGrid.init();

    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    sparseGrid.template insert<p>(coord) = value;

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType, typename ValueT>
__global__ void insertOneValue(SparseGridType sparseGrid, dim3 pt, ValueT value)
{
    sparseGrid.init();

    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    dim3 thCoord(x, y, z);
    if (thCoord.x == pt.x && thCoord.y == pt.y && thCoord.z == pt.z)
    {
        grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});
        sparseGrid.template insert<p>(coord) = value;
    }
    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType, typename VectorOutType>
__global__ void copyBlocksToOutput(SparseGridType sparseGrid, VectorOutType output)
{
    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    size_t pos = sparseGrid.getLinId(coord);

    auto value = sparseGrid.template get<p>(coord);

//    printf("copyBlocksToOutput: bDim=(%d,%d), bId=(%d,%d), tId=(%d,%d) : "
//           "pos=%ld, coord={%d,%d}, value=%d\n",
//           bDimX, bDimY,
//           bIdX, bIdY,
//           tIdX, tIdY,
//           pos,
//           x, y,
//           static_cast<int>(value)); //debug

    output.template get<p>(pos) = value;

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType, typename VectorOutType>
__global__ void copyToOutputIfPadding(SparseGridType sparseGrid, VectorOutType output)
{
    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    size_t pos = sparseGrid.getLinId(coord);


    output.template get<p>(pos) = sparseGrid.isPadding(coord) ? 1 : 0;

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType>
__global__ void insertBoundaryValuesHeat(SparseGridType sparseGrid)
{
    sparseGrid.init();

    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    float value = 0;
    if (x == 0)
    {
        value = 0;
    }
    else if (x == bDimX * gridDim.x - 1)
    {
        value = 10;
    }

    if (y == 0 || y == bDimY * gridDim.y - 1)
    {
        value = 10.0 * x / (bDimX * gridDim.x - 1);
    }

    sparseGrid.template insert<p>(coord) = value;

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int dim, unsigned int p>
struct LaplacianStencil
{
    // This is an example of a laplacian stencil to apply using the apply stencil facility of SparseGridGpu

    static constexpr unsigned int supportRadius = 1;

    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __device__ void stencil(
            SparseGridT & sparseGrid,
            grid_key_dx<dim, int> & dataBlockCoord,
            unsigned int offset,
            grid_key_dx<dim, int> & pointCoord,
            DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore)
    {
        typedef typename SparseGridT::AggregateBlockType AggregateT;
        typedef ScalarTypeOf<AggregateT, p> ScalarT;

        constexpr unsigned int enlargedBlockSize = IntPow<
                SparseGridT::getBlockEdgeSize() + 2 * supportRadius, dim>::value;

        __shared__ ScalarT enlargedBlock[enlargedBlockSize];
        sparseGrid.loadBlock<p>(dataBlockLoad, enlargedBlock);
        sparseGrid.loadGhost<p>(dataBlockCoord, enlargedBlock);
        __syncthreads();

        const auto coord = sparseGrid.getCoordInEnlargedBlock(offset);
        const auto linId = sparseGrid.getLinIdInEnlargedBlock(offset);
        ScalarT cur = enlargedBlock[linId];
        ScalarT res = -2.0*dim*cur; // The central part of the stencil
        for (int d=0; d<dim; ++d)
        {
            auto nPlusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, 1);
            auto nMinusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, -1);
            ScalarT neighbourPlus = enlargedBlock[nPlusId];
            ScalarT neighbourMinus = enlargedBlock[nMinusId];
            res += neighbourMinus + neighbourPlus;
        }
        enlargedBlock[linId] = res;

        __syncthreads();
        sparseGrid.storeBlock<p>(dataBlockStore, enlargedBlock);
    }

    template <typename SparseGridT, typename CtxT>
    static inline void __host__ flush(SparseGridT & sparseGrid, CtxT & ctx)
    {
        sparseGrid.template flush <smin_<0>> (ctx, flush_type::FLUSH_ON_DEVICE);
    }
};

template<unsigned int dim, unsigned int p_src, unsigned int p_dst>
struct HeatStencil
{
    // This is an example of a laplacian smoothing stencil to apply using the apply stencil facility of SparseGridGpu

	typedef NNStar<dim> stencil_type;

    static constexpr unsigned int supportRadius = 1;

    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __device__ void stencil(
            SparseGridT & sparseGrid,
            const unsigned int dataBlockId,
            openfpm::sparse_index<unsigned int> dataBlockIdPos,
            unsigned int offset,
            grid_key_dx<dim, int> & pointCoord,
            DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore,
            bool applyStencilHere,
            float dt)
    {
        typedef typename SparseGridT::AggregateBlockType AggregateT;
        typedef ScalarTypeOf<AggregateT, p_src> ScalarT;

        constexpr unsigned int enlargedBlockSize = IntPow<
                SparseGridT::getBlockEdgeSize() + 2 * supportRadius, dim>::value;

        __shared__ ScalarT enlargedBlock[enlargedBlockSize];
//        sparseGrid.loadBlock<p>(dataBlockLoad, enlargedBlock);
//        sparseGrid.loadGhost<p>(dataBlockId, enlargedBlock);

        sparseGrid.loadGhostBlock<p_src>(dataBlockLoad,dataBlockIdPos,enlargedBlock);

//        sparseGrid.loadGhost<p>(dataBlockId, nullptr, enlargedBlock);
        __syncthreads();

        if (applyStencilHere)
        {
            const auto coord = sparseGrid.getCoordInEnlargedBlock(offset);
            const auto linId = sparseGrid.getLinIdInEnlargedBlock(offset);
            ScalarT cur = enlargedBlock[linId];
            ScalarT laplacian = -2.0 * dim * cur; // The central part of the stencil
            for (int d = 0; d < dim; ++d)
            {
                auto nPlusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, 1);
                auto nMinusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, -1);
                ScalarT neighbourPlus = enlargedBlock[nPlusId];
                ScalarT neighbourMinus = enlargedBlock[nMinusId];
                laplacian += neighbourMinus + neighbourPlus;
            }
            enlargedBlock[linId] = cur + dt * laplacian;
        }

        __syncthreads();
        sparseGrid.storeBlock<p_dst>(dataBlockStore, enlargedBlock);
    }

    template <typename SparseGridT, typename CtxT>
    static inline void __host__ flush(SparseGridT & sparseGrid, CtxT & ctx)
    {
        sparseGrid.template flush <smax_<0>> (ctx, flush_type::FLUSH_ON_DEVICE);
    }
};

template<unsigned int dim, unsigned int p>
struct HeatStencil2
{
    // This is an example of a laplacian smoothing stencil to apply using the apply stencil facility of SparseGridGpu

    static constexpr unsigned int flops = 3 + 2*dim;

    static constexpr unsigned int supportRadius = 1;

    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __device__ void stencil(
            SparseGridT & sparseGrid,
            grid_key_dx<dim, int> & dataBlockCoord,
            unsigned int offset,
            grid_key_dx<dim, int> & pointCoord,
            DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore,
            float dt, unsigned int maxIter=1000)
    {
        typedef typename SparseGridT::AggregateBlockType AggregateT;
        typedef ScalarTypeOf<AggregateT, p> ScalarT;
        typedef BlockTypeOf<AggregateT, p> BlockT;
        constexpr unsigned int blockSize = BlockT::size;

        constexpr unsigned int enlargedBlockSize = IntPow<
                SparseGridT::getBlockEdgeSize() + 2 * supportRadius, dim>::value;

        const auto coord = sparseGrid.getCoordInEnlargedBlock(offset);
        const auto linId = sparseGrid.getLinIdInEnlargedBlock(offset);
        char boundaryDirection[dim];
        bool isBoundary = sparseGrid.getIfBoundaryElementInEnlargedBlock(coord, boundaryDirection);

        unsigned int nPlusId[dim], nMinusId[dim];
        for (int d=0; d<dim; ++d)
        {
            nPlusId[d] = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, 1);
            nMinusId[d] = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, -1);
        }

        __shared__ ScalarT enlargedBlock[enlargedBlockSize];

        ScalarT * nPlus[dim];
        ScalarT * nMinus[dim];
        for (int d=0; d<dim; ++d)
        {
            const auto boundaryDir = boundaryDirection[d];
            const auto nCoord = sparseGrid.getNeighbour(pointCoord, d, boundaryDir);
            const auto nOffset = sparseGrid.getLinId(nCoord) % blockSize;
            nPlus[d] = &(enlargedBlock[nPlusId[d]]);
            nMinus[d] = &(enlargedBlock[nMinusId[d]]);
            if (boundaryDir==1)
            {
                nPlus[d] = sparseGrid.getBlock(nCoord).template get<p>().block + nOffset;
            }
            else if (boundaryDir==-1)
            {
                nMinus[d] = sparseGrid.getBlock(nCoord).template get<p>().block + nOffset;
            }
        }

        sparseGrid.loadBlock<p>(dataBlockLoad, enlargedBlock);
        __syncthreads();
        for (unsigned int iter=0; iter<maxIter; ++iter)
        {
//            sparseGrid.loadGhost<p>(dataBlockCoord, enlargedBlock);
//            __syncthreads();

//todo: capisci come mai questa load non va mentre con la load ghost si!

            ScalarT cur = enlargedBlock[linId];
            ScalarT laplacian = -2.0 * dim * cur; // The central part of the stencil
            for (int d = 0; d < dim; ++d)
            {
//                const auto boundary = boundaryDirection[d];
//                ScalarT neighbourPlus = enlargedBlock[nPlusId[d]];
//                ScalarT neighbourMinus = enlargedBlock[nMinusId[d]];
//                if (boundary == 1)
//                {
//                    neighbourPlus = *(nPlus[d]);
//                }
//                else if (boundary == -1)
//                {
//                    neighbourMinus = *(nMinus[d]);
//                }
//                laplacian += neighbourMinus + neighbourPlus;
                laplacian += *(nMinus[d]) + *(nPlus[d]);
            }
            enlargedBlock[linId] = cur + dt * laplacian;

            __syncthreads();
//            sparseGrid.storeBlock<p>(dataBlockLoad, enlargedBlock);
            if (isBoundary)
            {
                dataBlockLoad.template get<p>()[offset] = enlargedBlock[linId];
            }
            __syncthreads();
        }
        __syncthreads();
        sparseGrid.storeBlock<p>(dataBlockStore, enlargedBlock);
        __syncthreads();
    }

    template <typename SparseGridT, typename CtxT>
    static inline void __host__ flush(SparseGridT & sparseGrid, CtxT & ctx)
    {
        sparseGrid.template flush <smin_<0>> (ctx, flush_type::FLUSH_ON_DEVICE);
    }
};

//todo: Here write some sort of stencil kernel to test the client interface for stencil application

BOOST_AUTO_TEST_SUITE(SparseGridGpu_tests)

BOOST_AUTO_TEST_CASE(testInsert)
{

	std::cout << std::endl; //debug empty line

	constexpr unsigned int dim = 2;
	constexpr unsigned int blockEdgeSize = 8;
	constexpr unsigned int dataBlockSize = blockEdgeSize * blockEdgeSize;
	typedef aggregate<DataBlock<float, dataBlockSize>> AggregateSGT;
	typedef aggregate<float> AggregateOutT;

	dim3 gridSize(2, 2);
	dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateOutT, blockEdgeSize> sparseGrid(blockGeometry);

	sparseGrid.template setBackgroundValue<0>(666);
	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);

	insertValues<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel());

	mgpu::ofp_context_t ctx;
	sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	// Get output
	openfpm::vector_gpu<AggregateOutT> output;
	output.resize(4 * 64);

	copyBlocksToOutput<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	sparseGrid.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
//            auto expectedValue = (i < gridSize * blockSizeInsert) ? i : 666;
		auto coord = sparseGrid.getCoord(i);
		auto expectedValue = coord.get(0);

		std::cout << "sparseGrid(" << coord.get(0) << "," << coord.get(1) << ") = "
				  << sparseGrid.template get<0>(coord)
				  << " == "
				  << expectedValue
				  << " == "
				  << output.template get<0>(i) << " = output(" << i << ")"
				  << std::endl;
		match &= output.template get<0>(i) == sparseGrid.template get<0>(coord);
		match &= output.template get<0>(i) == expectedValue;
	}

	BOOST_REQUIRE_EQUAL(match, true);
}

BOOST_AUTO_TEST_CASE(testInsert3D)
{
	constexpr unsigned int dim = 3;
	constexpr unsigned int blockEdgeSize = 4;
	constexpr unsigned int dataBlockSize = blockEdgeSize * blockEdgeSize;
	typedef aggregate<DataBlock<float, dataBlockSize>> AggregateSGT;
	typedef aggregate<float> AggregateOutT;

	dim3 gridSize(2, 2, 2);
	dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize, blockEdgeSize);

	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateOutT, blockEdgeSize> sparseGrid(blockGeometry);

	sparseGrid.template setBackgroundValue<0>(666);

	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);

	insertValues<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel());

	mgpu::ofp_context_t ctx;
	sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	// Get output
	openfpm::vector_gpu<AggregateOutT> output;
	output.resize(sparseGrid.dim3SizeToInt(gridSize) * 64);

	copyBlocksToOutput<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	sparseGrid.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
//            auto expectedValue = (i < gridSize * blockSizeInsert) ? i : 666;
		auto coord = sparseGrid.getCoord(i);
		auto expectedValue = coord.get(0);

//            std::cout << "sparseGrid(" << coord.get(0) << "," << coord.get(1) << "," << coord.get(2) << ") = "
//                      << sparseGrid.template get<0>(coord)
//                      << " == "
//                      << expectedValue
//                      << " == "
//                      << output.template get<0>(i) << " = output(" << i << ")"
//                      << std::endl;
		match &= output.template get<0>(i) == sparseGrid.template get<0>(coord);
		match &= output.template get<0>(i) == expectedValue;
	}

	BOOST_REQUIRE_EQUAL(match, true);
}

BOOST_AUTO_TEST_CASE(testTagBoundaries)
{

	constexpr unsigned int dim = 2;
	constexpr unsigned int blockEdgeSize = 8;
	typedef aggregate<float> AggregateT;

	dim3 gridSize(2, 2);
	dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64> sparseGrid(blockGeometry);

	sparseGrid.template setBackgroundValue<0>(666);
	mgpu::ofp_context_t ctx;

	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
	dim3 pt1(0, 0, 0);
	CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt1, 1);
	dim3 pt2(6, 6, 0);
	CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt2, 1);
	dim3 pt3(7, 6, 0);
	CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt3, 1);
	sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
	dim3 pt4(8, 6, 0);
	CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt4, 1);
	sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	/////////
	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
	for (int y = 9; y <= 11; y++)
	{
		dim3 pt1(6, y, 0);
		CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt1, 1);
		dim3 pt2(7, y, 0);
		CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt2, 1);
	}
	sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
	for (int y = 9; y <= 11; y++)
	{
		dim3 pt1(8, y, 0);
		CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt1, 1);
		dim3 pt2(9, y, 0);
		CUDA_LAUNCH_DIM3((insertOneValue<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), pt2, 1);
	}
	sparseGrid.template flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

//        sparseGrid.hostToDevice(); //just sync masks
	sparseGrid.deviceToHost(); //just sync masks
	sparseGrid.deviceToHost<0>();

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
	// Now tag the boundaries
	sparseGrid.tagBoundaries();

	// Get output
	openfpm::vector_gpu<AggregateT> output;
	output.resize(4 * 64);

	CUDA_LAUNCH_DIM3((copyToOutputIfPadding<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	sparseGrid.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
		auto coord = sparseGrid.getCoord(i);
		auto expectedValue =
				 ( i == 0
				|| i == 54
				|| i == 55
				|| i == 112
				|| i == 142 || i == 143 || i == 200 || i == 201 // (6,9), (7,9), (8,9), (9,9)
				|| i == 150 || i == 209 // (6,10), (9,10)
				|| i == 158 || i == 159 || i == 216 || i == 217 // (6,11), (7,11), (8,11), (9,11)
				 ) ? 1 : 0;

		std::cout
				<< "sparseGrid(" << coord.get(0) << "," << coord.get(1) << ") = "
				<< sparseGrid.template get<0>(coord) << " | "
				<< expectedValue
				<< " == "
				<< output.template get<0>(i) << " = output(" << i << ")"
				<< std::endl;
		match &= output.template get<0>(i) == expectedValue;
	}

	BOOST_REQUIRE_EQUAL(match, true);
}

BOOST_AUTO_TEST_CASE(testTagBoundaries2)
{
	constexpr unsigned int dim = 2;
	constexpr unsigned int blockEdgeSize = 8;
	typedef aggregate<float> AggregateT;

	dim3 gridSize(2, 2);
	dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64> sparseGrid(blockGeometry);

	sparseGrid.template setBackgroundValue<0>(666);
	mgpu::ofp_context_t ctx;

	///////
	{
		sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
		dim3 ptd1(6, 6, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd1, 1);
		dim3 ptd2(6, 7, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd2, 1);
		dim3 ptd3(7, 6, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd3, 1);
		dim3 ptd4(7, 7, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd4, 1);
		sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	}
	{
		sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
		dim3 ptd1(8, 6, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd1, 1);
		dim3 ptd2(9, 6, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd2, 1);
		dim3 ptd3(8, 7, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd3, 1);
		dim3 ptd4(9, 7, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd4, 1);
		sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	}
	{
		sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
		dim3 ptd1(6, 8, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd1, 1);
		dim3 ptd2(7, 8, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd2, 1);
		dim3 ptd3(6, 9, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd3, 1);
		dim3 ptd4(7, 9, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd4, 1);
		sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	}
	{
		sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
		dim3 ptd1(8, 8, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd1, 1);
		dim3 ptd2(8, 9, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd2, 1);
		dim3 ptd3(9, 8, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd3, 1);
		dim3 ptd4(9, 9, 0);
		insertOneValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), ptd4, 1);
		sparseGrid.flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	}
	///////

//        sparseGrid.hostToDevice(); //just sync masks
	sparseGrid.deviceToHost(); //just sync masks
//        sparseGrid.deviceToHost<0>();

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
	// Now tag the boundaries
	sparseGrid.tagBoundaries();

	// Get output
	openfpm::vector_gpu<AggregateT> output;
	output.resize(4 * 64);

	copyToOutputIfPadding<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	sparseGrid.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
		auto coord = sparseGrid.getCoord(i);
		auto expectedValue =
				 (
						 i == 54 || i == 55 || i == 62 // (6,6), (7,6), (6,7)
					  || i == 134 || i == 142 || i == 143 // (6,8), (6,9), (7,9)
					  || i == 112 || i == 113 || i == 121 // (8,6), (9,6), (9,7)
					  || i == 200 || i == 193 || i == 201 // (8,9), (9,8), (9,9)
				 ) ? 1 : 0;

		std::cout
				<< "sparseGrid(" << coord.get(0) << "," << coord.get(1) << "," << coord.get(2) << ") = "
				<< sparseGrid.template get<0>(coord) << " | "
				<< expectedValue
				<< " == "
				<< output.template get<0>(i) << " = output(" << i << ")"
				<< std::endl;
		match &= output.template get<0>(i) == expectedValue;
	}

	BOOST_REQUIRE_EQUAL(match, true);
}

BOOST_AUTO_TEST_CASE(testStencilHeat)
{
	printf("\n");

	constexpr unsigned int dim = 2;
	constexpr unsigned int blockEdgeSize = 8;
	typedef aggregate<float,float> AggregateT;

	dim3 gridSize(2, 2);
	dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	// Insert values on the grid
	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
	insertConstantValue<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), 0);
	sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

//	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
//	insertBoundaryValuesHeat<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel());
//	sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
	sparseGrid.tagBoundaries();

    hipDeviceSynchronize();

    sparseGrid.template applyBoundaryStencils<BoundaryStencilSetXRescaled<dim,0,0>>(0.0 ,gridSize.x * blockEdgeSize, 0.0, 10.0);

	hipDeviceSynchronize();

        // Now apply the laplacian operator
	const unsigned int maxIter = 1000;
//    const unsigned int maxIter = 100;
	for (unsigned int iter=0; iter<maxIter; ++iter)
	{
		sparseGrid.applyStencils<HeatStencil<dim, 0, 1>>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();
        sparseGrid.applyStencils<HeatStencil<dim, 1, 0>>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();
//        // DEBUG //
//        sparseGrid.deviceToHost<0,1>();
//        sparseGrid.write("test_heat_stencil"+std::to_string(iter)+".vtk");
//        ////
	}

	sparseGrid.deviceToHost<0,1>();

	// Get output
	openfpm::vector_gpu<AggregateT> output;
	output.resize(4 * 64);

	copyBlocksToOutput<0> << < gridSize, blockSizeInsert >> > (sparseGrid.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	sparseGrid.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
		auto coord = sparseGrid.getCoord(i);
		float expectedValue = 10.0 * coord.get(0) / (gridSize.x * blockEdgeSize - 1);

		std::cout
				<< "sparseGrid(" << coord.get(0) << "," << coord.get(1) << ") = "
				<< sparseGrid.template get<0>(coord) << " | "
				<< expectedValue
				<< " == "
				<< output.template get<0>(i) << " = output(" << i << ")"
				<< std::endl;
		match &= fabs(output.template get<0>(i) - expectedValue) < 1e-2;

	}

	BOOST_REQUIRE_EQUAL(match, true);
//        BOOST_REQUIRE_CLOSE(output.template get<0>(255), 3.20309591e-05, 1e-6);
}

BOOST_AUTO_TEST_CASE(testStencilHeatInsert)
{
	printf("\n");

	constexpr unsigned int dim = 2;
	constexpr unsigned int blockEdgeSize = 8;
	typedef aggregate<float,float> AggregateT;

	dim3 gridSize(2, 2);
	dim3 blockSizeInsert(blockEdgeSize, blockEdgeSize);

	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	// Insert values on the grid
	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
	CUDA_LAUNCH_DIM3((insertConstantValue2<0,1>),gridSize, blockSizeInsert,sparseGrid.toKernel(), 0);
	sparseGrid.flush < smax_< 0 >, smax_<1>> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
	CUDA_LAUNCH_DIM3((insertBoundaryValuesHeat<0>),gridSize, blockSizeInsert,sparseGrid.toKernel());
	sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

//	sparseGrid.setGPUInsertBuffer(gridSize, blockSizeInsert);
//	CUDA_LAUNCH_DIM3((insertBoundaryValuesHeat<1>),gridSize, blockSizeInsert,sparseGrid.toKernel());
//	sparseGrid.flush < smax_< 1 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!

//        // Now tag the boundaries
	sparseGrid.tagBoundaries();

	sparseGrid.template deviceToHost<0,1>();

	// Now apply the laplacian operator
	const unsigned int maxIter = 1000;
//        const unsigned int maxIter = 10;
	for (unsigned int iter=0; iter<maxIter; ++iter)
	{
		sparseGrid.applyStencils<HeatStencil<dim, 0,0>>(STENCIL_MODE_INSERT, 0.1);
//		sparseGrid.applyStencils<HeatStencil<dim, 1,0>>(STENCIL_MODE_INSERT, 0.1);
	}


	// Get output
	openfpm::vector_gpu<AggregateT> output;
	output.resize(4 * 64);

	CUDA_LAUNCH_DIM3((copyBlocksToOutput<0>),gridSize, blockSizeInsert,sparseGrid.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	sparseGrid.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
		grid_key_dx<dim, int> coord = sparseGrid.getCoord(i);
		float expectedValue = 10.0 * coord.get(0) / (gridSize.x * blockEdgeSize - 1);

		unsigned int check = sparseGrid.getLinId(coord);

		std::cout
				<< "invLinId=" << check << ", "
//                    << "sparseGrid(" << coord.get(0) << "," << coord.get(1) << "," << coord.get(2) << ") = "
				<< "sparseGrid(" << coord.get(0) << "," << coord.get(1) << ") = "
				<< sparseGrid.template get<0>(coord) << " | "
				<< expectedValue
				<< " == "
				<< output.template get<0>(i) << " = output(" << i << ")"
				<< std::endl;
		match &= fabs(output.template get<0>(i) - expectedValue) < 1e-2;

	}

	BOOST_REQUIRE_EQUAL(match, true);
//        BOOST_REQUIRE_CLOSE(output.template get<0>(255), 3.20309591e-05, 1e-6);
}

template<typename sparsegrid_type>
__global__ void sparse_grid_get_test(sparsegrid_type sparseGrid, grid_key_dx<3> key, float * data)
{
	*data = sparseGrid.template get<0>(key);
}

BOOST_AUTO_TEST_CASE(testFlushInsert)
{
	printf("\n");

	constexpr unsigned int dim = 3;
	constexpr unsigned int blockEdgeSize = 4;
	typedef aggregate<float,float> AggregateT;


	size_t sz[] = {137,100,57};

	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64> sparseGrid(sz);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	sparseGrid.insertFlush<0>(grid_key_dx<3>({3,6,7})) = 2.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({13,16,17})) = 3.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({13,46,27})) = 4.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({36,63,11})) = 5.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({37,96,47})) = 6.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({130,56,37})) = 7.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({131,76,17})) = 8.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({36,86,27})) = 9.0;
	sparseGrid.insertFlush<0>(grid_key_dx<3>({34,36,7})) = 10.0;

	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({3,6,7})),2.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({13,16,17})),3.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({13,46,27})),4.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({36,63,11})),5.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({37,96,47})),6.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({130,56,37})),7.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({131,76,17})),8.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({36,86,27})),9.0);
	BOOST_REQUIRE_EQUAL(sparseGrid.get<0>(grid_key_dx<3>({34,36,7})),10.0);

	sparseGrid.template hostToDevice<0>();

	// Check on device I can get information

	CudaMemory mem;

	mem.allocate(sizeof(float));

	grid_key_dx<3> key({3,6,7});

	sparse_grid_get_test<<<1,1>>>(sparseGrid.toKernel(),key,(float *)mem.getDevicePointer());

	mem.deviceToHost();

	BOOST_REQUIRE_EQUAL(*(float *)mem.getPointer(),2.0);

	grid_key_dx<3> key2({131,76,17});

	sparse_grid_get_test<<<1,1>>>(sparseGrid.toKernel(),key2,(float *)mem.getDevicePointer());

	mem.deviceToHost();

	BOOST_REQUIRE_EQUAL(*(float *)mem.getPointer(),8.0);
}

struct conv_coeff
{
	float coeff[3][3][3];
};

template<unsigned int dim, unsigned int p_src, unsigned int p_dst>
struct Conv3x3x3
{
    // This is an example of a laplacian smoothing stencil to apply using the apply stencil facility of SparseGridGpu

	typedef NNFull<dim> stencil_type;

    static constexpr unsigned int supportRadius = 1;

    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __device__ void stencil(
            SparseGridT & sparseGrid,
            const unsigned int dataBlockId,
            openfpm::sparse_index<unsigned int> dataBlockIdPos,
            unsigned int offset,
            grid_key_dx<dim, int> & pointCoord,
            DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore,
            bool applyStencilHere,
            conv_coeff & cc)
    {
        typedef typename SparseGridT::AggregateBlockType AggregateT;
        typedef ScalarTypeOf<AggregateT, p_src> ScalarT;

        constexpr unsigned int enlargedBlockSize = IntPow<
                SparseGridT::getBlockEdgeSize() + 2 * supportRadius, dim>::value;

        __shared__ ScalarT enlargedBlock[enlargedBlockSize];

        sparseGrid.loadGhostBlock<p_src>(dataBlockLoad,dataBlockIdPos,enlargedBlock);

        __syncthreads();

        if (applyStencilHere)
        {
            const auto coord = sparseGrid.getCoordInEnlargedBlock(offset);
            const auto linId = sparseGrid.getLinIdInEnlargedBlock(offset);
            ScalarT tot = 0.0;
            for (int i = 0; i < dim; ++i)
            {
                for (int j = 0; j < dim; ++j)
                {
                    for (int k = 0; k < dim; ++k)
                    {
                    	grid_key_dx<dim,int> key;

                    	key.set_d(0,i-1);
                    	key.set_d(1,j-1);
                    	key.set_d(2,k-1);

                    	auto nPlusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, key);
                    	tot += enlargedBlock[nPlusId] * cc.coeff[k][j][i];
                    }
                }
            }

            dataBlockStore.template get<p_dst>()[offset] = tot;
        }
    }

    template <typename SparseGridT, typename CtxT>
    static inline void __host__ flush(SparseGridT & sparseGrid, CtxT & ctx)
    {
        sparseGrid.template flush <smax_<0>> (ctx, flush_type::FLUSH_ON_DEVICE);
    }
};



BOOST_AUTO_TEST_CASE(test3x3x3convolution)
{
	constexpr unsigned int dim = 3;
	constexpr unsigned int blockEdgeSize = 4;
	typedef aggregate<float,float> AggregateT;


	size_t sz[] = {1000,1000,1000};

	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64, long int> sparseGrid(sz);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	// now create 3 3D sphere

    grid_key_dx<3,int> start({256,256,256});

    dim3 gridSize(32,32,32);

    // Insert values on the grid
    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
    CUDA_LAUNCH_DIM3((insertSphere3D_radius<0>),
            gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
            sparseGrid.toKernel(), start,64, 56, 1);

    sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.template findNeighbours<NNFull<3>>(); // Pre-compute the neighbours pos for each block!

    sparseGrid.template setNNType<NNFull<dim>>();
    sparseGrid.template tagBoundaries<NNFull<3>>();

    conv_coeff cc;

    for (int i = 0 ; i < 3 ; i++)
    {
    	for (int j = 0 ; j < 3 ; j++)
    	{
    		for (int k = 0 ; k < 3 ; k++)
    		{
    			cc.coeff[k][j][i] = 1.0;
    		}
    	}
    }


    sparseGrid.template applyStencils<Conv3x3x3<dim,0,1>>(STENCIL_MODE_INPLACE,cc);

    sparseGrid.template deviceToHost<0,1>();

	auto & bm = sparseGrid.private_get_blockMap();
	auto & dataVector = bm.getDataBuffer();

	bool match = true;

    for (size_t i = 0 ; i < dataVector.size() ; i++)
    {
        for (size_t j = 0 ; j < 64 ; j++)
        {
			if (dataVector.template get<2>(i)[j] == 1)
			{
				match &= dataVector.template get<0>(i)[j]*27 == dataVector.template get<1>(i)[j];
			}
        }
    }

    BOOST_REQUIRE_EQUAL(match,true);
}

#if defined(OPENFPM_DATA_ENABLE_IO_MODULE) || defined(PERFORMANCE_TEST)


BOOST_AUTO_TEST_CASE(testSparseGridGpuOutput)
{
	constexpr unsigned int dim = 2;
	constexpr unsigned int blockEdgeSize = 8;
	typedef aggregate<float> AggregateT;

	size_t sz[2] = {1000000,1000000};
	dim3 gridSize(128,128);

	grid_smb<dim, blockEdgeSize> blockGeometry(sz);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, 64, long int> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	grid_key_dx<2,int> start({500000,500000});

	// Insert values on the grid
	sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
	CUDA_LAUNCH_DIM3((insertSphere<0>),gridSize, dim3(blockEdgeSize*blockEdgeSize,1),sparseGrid.toKernel(), start, 512, 256, 1);
	sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
	sparseGrid.tagBoundaries();

	sparseGrid.template deviceToHost<0>();

	sparseGrid.write("SparseGridGPU_output.vtk");
}

    BOOST_AUTO_TEST_CASE(testSparseGridGpuOutput3D)
    {
        constexpr unsigned int dim = 3;
        constexpr unsigned int blockEdgeSize = 4;
        typedef aggregate<float> AggregateT;

        size_t sz[3] = {512,512,512};
//        dim3 gridSize(128,128,128);
        dim3 gridSize(32,32,32);

        grid_smb<dim, blockEdgeSize> blockGeometry(sz);
        SparseGridGpu<dim, AggregateT, blockEdgeSize, 64, long int> sparseGrid(blockGeometry);
        mgpu::ofp_context_t ctx;
        sparseGrid.template setBackgroundValue<0>(0);

        grid_key_dx<3,int> start({256,256,256});

        // Insert values on the grid
        sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
        CUDA_LAUNCH_DIM3((insertSphere3D<0>),
                gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
                sparseGrid.toKernel(), start, 64, 56, 1);
        sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

        hipDeviceSynchronize();

        sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
        sparseGrid.tagBoundaries();

        sparseGrid.template applyBoundaryStencils<BoundaryStencilSetX<dim,0,0>>();

        hipDeviceSynchronize();

        sparseGrid.template deviceToHost<0>();

        sparseGrid.write("SparseGridGPU_output3D.vtk");
    }

    BOOST_AUTO_TEST_CASE(testSparseGridGpuOutput3DHeatStencil)
    {
        constexpr unsigned int dim = 3;
        constexpr unsigned int blockEdgeSize = 4;
        typedef aggregate<float, float> AggregateT;

        size_t sz[3] = {512,512,512};
//        dim3 gridSize(128,128,128);
        dim3 gridSize(32,32,32);

        grid_smb<dim, blockEdgeSize> blockGeometry(sz);
        SparseGridGpu<dim, AggregateT, blockEdgeSize, 64, long int> sparseGrid(blockGeometry);
        mgpu::ofp_context_t ctx;
        sparseGrid.template setBackgroundValue<0>(0);

        ///// Insert sparse content, a set of 3 hollow spheres /////
        // Sphere 1
        grid_key_dx<3,int> start1({256,256,256});
        sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
        CUDA_LAUNCH_DIM3((insertSphere3D<0>),
                         gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
                         sparseGrid.toKernel(), start1, 64, 32, 1);
        hipDeviceSynchronize();
        sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
        hipDeviceSynchronize();

        // Sphere 2
        grid_key_dx<3,int> start2({192,192,192});
        sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
        CUDA_LAUNCH_DIM3((insertSphere3D<0>),
                         gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
                         sparseGrid.toKernel(), start2, 64, 44, 1);
        hipDeviceSynchronize();
        sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
        hipDeviceSynchronize();

        // Sphere 3
        grid_key_dx<3,int> start3({340,192,192});
        sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
        CUDA_LAUNCH_DIM3((insertSphere3D<0>),
                         gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
                         sparseGrid.toKernel(), start3, 20, 15, 1);
        hipDeviceSynchronize();
        sparseGrid.flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
        hipDeviceSynchronize();
        ///// /////

        sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
        sparseGrid.tagBoundaries();

        // Now apply some boundary conditions
        sparseGrid.template applyBoundaryStencils<BoundaryStencilSetXRescaled<dim,0,0>>(
                192, 384,
                0.0, 10.0);
        hipDeviceSynchronize();

        // Now apply the laplacian operator
//        const unsigned int maxIter = 1000;
        const unsigned int maxIter = 100;
        for (unsigned int iter=0; iter<maxIter; ++iter)
        {
            for (int innerIter=0; innerIter<10; ++innerIter)
            {
                sparseGrid.applyStencils<HeatStencil<dim, 0, 1>>(STENCIL_MODE_INPLACE, 0.1);
                hipDeviceSynchronize();
                sparseGrid.applyStencils<HeatStencil<dim, 1, 0>>(STENCIL_MODE_INPLACE, 0.1);
                hipDeviceSynchronize();
            }
            // DEBUG //
            sparseGrid.deviceToHost<0,1>();
            sparseGrid.write("SparseGridGPU_output3DHeatStencil_"+std::to_string(iter)+".vtk");
            ////
        }

        sparseGrid.template deviceToHost<0,1>();

        sparseGrid.write("SparseGridGPU_output3DHeatStencil.vtk");
    }

#endif

BOOST_AUTO_TEST_SUITE_END()
