#include "hip/hip_runtime.h"
//
// Created by tommaso on 14/05/19.
//

#define BOOST_TEST_DYN_LINK

#include <hip/hip_runtime.h>
#include "config.h"
#include <boost/test/unit_test.hpp>
#include "SparseGridGpu/BlockMapGpu.hpp"
#include "SparseGridGpu/BlockMapGpu_ker.cuh"
#include "SparseGridGpu/BlockMapGpu_kernels.cuh"
#include "SparseGridGpu/DataBlock.cuh"

#include <limits>

template<unsigned int p, typename SparseGridType, typename VectorOutType>
__global__ void copyBlocksToOutput(SparseGridType sparseGrid, VectorOutType output)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    output.template get<p>(pos) = sparseGrid.template get<p>(pos);
}

template<unsigned int p, typename SparseGridType>
__global__ void insertValues(SparseGridType sparseGrid)
{
    sparseGrid.init();

    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    sparseGrid.template insert<p>(pos) = pos;

    __syncthreads();

    sparseGrid.flush_block_insert();
}

template<unsigned int p, unsigned int chunksPerBlock, typename SparseGridType>
__global__ void insertValuesBlocked(SparseGridType sparseGrid)
{
    constexpr unsigned int pMask = SparseGridType::pMask;
    typedef BlockTypeOf<typename SparseGridType::AggregateType, p> BlockT;
    typedef BlockTypeOf<typename SparseGridType::AggregateType, pMask> MaskBlockT;

    sparseGrid.init();


    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int dataBlockId = pos / BlockT::size;
    unsigned int offset = pos % BlockT::size;

    auto encap = sparseGrid.template insertBlock<chunksPerBlock>(dataBlockId,BlockT::size);


    encap.template get<p>()[offset] = pos;
    BlockMapGpu_ker<>::setExist(encap.template get<pMask>()[offset]);

    __syncthreads();

    sparseGrid.flush_block_insert();
}

template<unsigned int p, typename SparseGridType>
__global__ void insertValuesHalfBlock(SparseGridType sparseGrid)
{
    sparseGrid.init();

    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    constexpr unsigned int dataChunkSize = BlockTypeOf<typename SparseGridType::AggregateType, p>::size;
    if (threadIdx.x % dataChunkSize < dataChunkSize/ 2)
    {
        sparseGrid.template insert<p>(pos) = pos;
    }

    __syncthreads();

    sparseGrid.flush_block_insert();
}

BOOST_AUTO_TEST_SUITE(BlockMapGpu_tests)

BOOST_AUTO_TEST_CASE(testBitwiseOps)
{
	BOOST_REQUIRE(BlockMapGpu_ker<>::getBit(1,0));
	BOOST_REQUIRE(!BlockMapGpu_ker<>::getBit(2,0));
	BOOST_REQUIRE(BlockMapGpu_ker<>::getBit(2,1));
	BOOST_REQUIRE(BlockMapGpu_ker<>::getBit(3,0));
	BOOST_REQUIRE(BlockMapGpu_ker<>::getBit(3,1));
	unsigned int m = 0;
	BOOST_REQUIRE(!BlockMapGpu_ker<>::getBit(m,0));
	BlockMapGpu_ker<>::setBit(m, 0);
	BOOST_REQUIRE(BlockMapGpu_ker<>::getBit(m,0));
	BlockMapGpu_ker<>::unsetBit(m, 0);
	BOOST_REQUIRE(!BlockMapGpu_ker<>::getBit(m,0));
}

BOOST_AUTO_TEST_CASE(testBackground)
{
	typedef aggregate<DataBlock<float, 64>> AggregateSGT;
	typedef aggregate<float> AggregateOutT;
	BlockMapGpu<AggregateSGT> sparseGrid;
	sparseGrid.template setBackgroundValue<0>(666);

	const unsigned int gridSize = 10;
	const unsigned int blockSize = 128;

	// Get output
	openfpm::vector_gpu<AggregateOutT> output;
	output.resize(gridSize * blockSize);
	hipLaunchKernelGGL(HIP_KERNEL_NAME(copyBlocksToOutput<0>), dim3(gridSize), dim3(blockSize), 0, 0, sparseGrid.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	sparseGrid.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
//            std::cout << "output(" << i << ") = " << output.template get<0>(i) << std::endl;
		match &= output.template get<0>(i) == sparseGrid.template get<0>(i);
	}

	BOOST_REQUIRE_EQUAL(match, true);
}

BOOST_AUTO_TEST_CASE(testInsert)
{
	typedef aggregate<DataBlock<float, 64>> AggregateT;
	typedef aggregate<float> AggregateOutT;
	BlockMapGpu<AggregateT, 128> blockMap;
	blockMap.template setBackgroundValue<0>(666);

	const unsigned int gridSize = 3;
	const unsigned int bufferPoolSize = 128; // Should be multiple of BlockT::size
	const unsigned int blockSizeInsert = 128;
	const unsigned int gridSizeRead = gridSize + 1;
	const unsigned int blockSizeRead = 128;

	// Prealloc insert buffer
	blockMap.setGPUInsertBuffer(gridSize, bufferPoolSize);

	// Insert values
	hipLaunchKernelGGL(HIP_KERNEL_NAME(insertValues<0>), dim3(gridSize), dim3(blockSizeInsert), 0, 0, blockMap.toKernel());

	// Flush inserts
	mgpu::ofp_context_t ctx;
	blockMap.flush<smax_<0>>(ctx, flush_type::FLUSH_ON_DEVICE);

	// Get output
	openfpm::vector_gpu<AggregateOutT> output;
	output.resize(gridSizeRead * blockSizeRead);

	hipLaunchKernelGGL(HIP_KERNEL_NAME(copyBlocksToOutput<0>), dim3(gridSizeRead), dim3(blockSizeRead), 0, 0, blockMap.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	blockMap.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
		auto expectedValue = (i < gridSize * blockSizeInsert) ? i : 666;
		std::cout << "blockMap(" << i << ") = " << blockMap.template get<0>(i)
				<< " == "
				<< expectedValue
				<< " == "
				<< output.template get<0>(i) << " = output(" << i << ")"
						<< std::endl;
		match &= output.template get<0>(i) == blockMap.template get<0>(i);
		match &= output.template get<0>(i) == expectedValue;
	}

	BOOST_REQUIRE_EQUAL(match, true);
}

BOOST_AUTO_TEST_CASE(testInsert_halfBlock) //todo
{
	typedef aggregate<DataBlock<float, 64>> AggregateT;
	typedef aggregate<float> AggregateOutT;
	BlockMapGpu<AggregateT, 128> blockMap;
	blockMap.template setBackgroundValue<0>(666);

	const unsigned int gridSize = 3;
	const unsigned int bufferPoolSize = 128; // Should be multiple of BlockT::size
	const unsigned int blockSizeInsert = 128;
	const unsigned int gridSizeRead = gridSize + 1;
	const unsigned int blockSizeRead = 128;

	// Prealloc insert buffer
	blockMap.setGPUInsertBuffer(gridSize, bufferPoolSize);

	// Insert values
	hipLaunchKernelGGL(HIP_KERNEL_NAME(insertValuesHalfBlock<0>), dim3(gridSize), dim3(blockSizeInsert), 0, 0, blockMap.toKernel());

	// Flush inserts
	mgpu::ofp_context_t ctx;
	blockMap.flush<smax_<0>>(ctx, flush_type::FLUSH_ON_DEVICE);

	// Get output
	openfpm::vector_gpu<AggregateOutT> output;
	output.resize(gridSizeRead * blockSizeRead);

	hipLaunchKernelGGL(HIP_KERNEL_NAME(copyBlocksToOutput<0>), dim3(gridSizeRead), dim3(blockSizeRead), 0, 0, blockMap.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	blockMap.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
		auto expectedValue = (i < gridSize * blockSizeInsert) ? i : 666;
		constexpr unsigned int dataChunkSize = BlockTypeOf<AggregateT, 0>::size;
		int offset = i % dataChunkSize;
		if (! (offset < dataChunkSize / 2))
		{
			expectedValue = 666; // Just the first half of each block was inserted
		}
		std::cout << "blockMap(" << i << ") = " << blockMap.template get<0>(i)
				  << " == "
				  << expectedValue
				  << " == "
				  << output.template get<0>(i) << " = output(" << i << ")"
				  << std::endl;
		match &= output.template get<0>(i) == blockMap.template get<0>(i);
		match &= output.template get<0>(i) == expectedValue;
	}

	BOOST_REQUIRE_EQUAL(match, true);
}

BOOST_AUTO_TEST_CASE(testInsert_blocked)
{
	typedef aggregate<DataBlock<float, 64>> AggregateT;
	typedef aggregate<float> AggregateOutT;
	BlockMapGpu<AggregateT, 128> sparseGrid;
	sparseGrid.template setBackgroundValue<0>(666);

	const unsigned int gridSize = 3;
	const unsigned int bufferPoolSize = 4; // Should be multiple of BlockT::size
	const unsigned int blockSizeInsert = 128;
	const unsigned int gridSizeRead = gridSize + 1;
	const unsigned int blockSizeRead = 128;

////////// DEBUG
//        const unsigned int gridSize = 2;
//        const unsigned int bufferPoolSize = 128; // Should be multiple of BlockT::size
//        const unsigned int blockSizeInsert = 64;
//        const unsigned int gridSizeRead = gridSize + 1;
//        const unsigned int blockSizeRead = 64;
//////////

	// Prealloc insert buffer
	sparseGrid.setGPUInsertBuffer(gridSize, bufferPoolSize);

	// Insert values
	hipLaunchKernelGGL(HIP_KERNEL_NAME(insertValuesBlocked<0, 2>), dim3(gridSize), dim3(blockSizeInsert), 0, 0, sparseGrid.toKernel());

	// Flush inserts
	mgpu::ofp_context_t ctx;
	sparseGrid.flush<smax_<0>>(ctx, flush_type::FLUSH_ON_DEVICE);

	// Get output
	openfpm::vector_gpu<AggregateOutT> output;
	output.resize(gridSizeRead * blockSizeRead);

	hipLaunchKernelGGL(HIP_KERNEL_NAME(copyBlocksToOutput<0>), dim3(gridSizeRead), dim3(blockSizeRead), 0, 0, sparseGrid.toKernel(), output.toKernel());

	output.template deviceToHost<0>();
	sparseGrid.template deviceToHost<0>();

	// Compare
	bool match = true;
	for (size_t i = 0; i < output.size(); i++)
	{
		auto expectedValue = (i < gridSize * blockSizeInsert) ? i : 666;
		std::cout << "sparseGrid(" << i << ") = " << sparseGrid.template get<0>(i)
				  << " == "
				  << expectedValue
				  << " == "
				  << output.template get<0>(i) << " = output(" << i << ")"
				  << std::endl;
		match &= output.template get<0>(i) == sparseGrid.template get<0>(i);
		match &= output.template get<0>(i) == expectedValue;
	}

	BOOST_REQUIRE_EQUAL(match, true);
}

BOOST_AUTO_TEST_SUITE_END()

