#include "hip/hip_runtime.h"
//
// Created by tommaso on 4/07/19.
//

#define BOOST_TEST_DYN_LINK
#define OPENFPM_DATA_ENABLE_IO_MODULE
#define DISABLE_MPI_WRITTERS

#include <boost/test/unit_test.hpp>
#include "SparseGridGpu/SparseGridGpu.hpp"
#include "cuda_macro.h"
#include <boost/property_tree/ptree.hpp>
#include <boost/property_tree/xml_parser.hpp>
#include "util/stat/common_statistics.hpp"
#include "Plot/GoogleChart.hpp"
#include "util/performance/performance_util.hpp"
#include "SparseGridGpu/tests/utils/SparseGridGpu_testKernels.cuh"

extern char * test_dir;

// Property tree
struct report_sparse_grid_tests
{
	boost::property_tree::ptree graphs;
};

report_sparse_grid_tests report_sparsegrid_funcs;
std::string suiteURI = "performance.SparseGridGpu";

void write_test_report()
{
    const char *perfResultsXmlFile = "SparseGridGpu_performance.xml";

    unsigned int plotCounter = 0;

    // Dense 2D
    {
        std::string dim = "2";
        std::string linMode = "N";
        std::string base = "performance.SparseGridGpu.device.stencil.dense."+linMode+"."+dim+"D";

        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").type", "line");
        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").interpolation", "none");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").title",
                                           "SparseGridGPU stencil "+linMode+" "+dim+"D"+" grid scaling performance");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.title", "GridEdgeSize");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.title", "GFlops");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).source",
                                           base + ".gridScaling(#).GFlops.mean");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.data(0).source",
                                           base + ".gridScaling(#).gridSize.x");
//    report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").options.log_x", true);
        int bes = static_cast<int>( report_sparsegrid_funcs.graphs.template get<double>(
                base + ".gridScaling(0).blockSize"));
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).title",
                                           "blockEdge=" + std::to_string(bes));
        ++plotCounter;

        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").type", "line");
        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").interpolation", "none");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").title",
                                           "SparseGridGPU stencil "+linMode+" "+dim+"D"+" block scaling performance");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.title",
                                           "BlockEdgeSize");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.title", "GFlops");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).source",
                                           base + ".blockScaling(#).GFlops.mean");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.data(0).source",
                                           base + ".blockScaling(#).blockSize");
//    report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").options.log_x",true);
        int ges = static_cast<int>( report_sparsegrid_funcs.graphs.template get<double>(
                base + ".blockScaling(0).gridSize.x"));
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).title",
                                           "gridEdge=" + std::to_string(ges));
        ++plotCounter;
    }

    // Dense 2D Z-morton
    {
        std::string dim = "2";
        std::string linMode = "Z";
        std::string base = "performance.SparseGridGpu.device.stencil.dense."+linMode+"."+dim+"D";

        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").type", "line");
        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").interpolation", "none");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").title",
                                           "SparseGridGPU stencil "+linMode+" "+dim+"D"+" grid scaling performance");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.title", "GridEdgeSize");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.title", "GFlops");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).source",
                                           base + ".gridScaling(#).GFlops.mean");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.data(0).source",
                                           base + ".gridScaling(#).gridSize.x");
//    report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").options.log_x", true);
        int bes = static_cast<int>( report_sparsegrid_funcs.graphs.template get<double>(
                base + ".gridScaling(0).blockSize"));
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).title",
                                           "blockEdge=" + std::to_string(bes));
        ++plotCounter;

        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").type", "line");
        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").interpolation", "none");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").title",
                                           "SparseGridGPU stencil "+linMode+" "+dim+"D"+" block scaling performance");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.title",
                                           "BlockEdgeSize");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.title", "GFlops");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).source",
                                           base + ".blockScaling(#).GFlops.mean");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.data(0).source",
                                           base + ".blockScaling(#).blockSize");
//    report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").options.log_x",true);
        int ges = static_cast<int>( report_sparsegrid_funcs.graphs.template get<double>(
                base + ".blockScaling(0).gridSize.x"));
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).title",
                                           "gridEdge=" + std::to_string(ges));
        ++plotCounter;
    }

    // Dense 3D
    {
        std::string dim = "3";
        std::string linMode = "N";
        std::string base = "performance.SparseGridGpu.device.stencil.dense."+linMode+"."+dim+"D";

        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").type", "line");
        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").interpolation", "none");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").title",
                                           "SparseGridGPU stencil "+linMode+" "+dim+"D"+" grid scaling performance");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.title", "GridEdgeSize");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.title", "GFlops");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).source",
                                           base + ".gridScaling(#).GFlops.mean");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.data(0).source",
                                           base + ".gridScaling(#).gridSize.x");
//    report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").options.log_x", true);
        int bes = static_cast<int>( report_sparsegrid_funcs.graphs.template get<double>(
                base + ".gridScaling(0).blockSize"));
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).title",
                                           "blockEdge=" + std::to_string(bes));
        ++plotCounter;

        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").type", "line");
        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").interpolation", "none");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").title",
                                           "SparseGridGPU stencil "+linMode+" "+dim+"D"+" block scaling performance");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.title",
                                           "BlockEdgeSize");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.title", "GFlops");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).source",
                                           base + ".blockScaling(#).GFlops.mean");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.data(0).source",
                                           base + ".blockScaling(#).blockSize");
//    report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").options.log_x",true);
        int ges = static_cast<int>( report_sparsegrid_funcs.graphs.template get<double>(
                base + ".blockScaling(0).gridSize.x"));
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).title",
                                           "gridEdge=" + std::to_string(ges));
        ++plotCounter;
    }

    // Sparse 2D
    {
        std::string dim = "2";
        std::string pattern = "sparse";
        std::string linMode = "N";
        std::string base = "performance.SparseGridGpu.device.stencil."+pattern+"."+linMode+"."+dim+"D";

        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").type", "line");
        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").interpolation", "none");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").title",
                                           "SparseGridGPU stencil "+pattern+" "+linMode+" "+dim+"D"
                                           +" grid scaling performance");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.title", "GridEdgeSize");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.title", "GFlops");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).source",
                                           base + ".gridScaling(#).GFlops.mean");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.data(0).source",
                                           base + ".gridScaling(#).gridSize.x");
//    report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").options.log_x", true);
        int bes = static_cast<int>( report_sparsegrid_funcs.graphs.template get<double>(
                base + ".gridScaling(0).blockSize"));
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).title",
                                           "blockEdge=" + std::to_string(bes));
        ++plotCounter;

        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").type", "line");
        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").interpolation", "none");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").title",
                                           "SparseGridGPU stencil "+pattern+" "+linMode+" "+dim+"D"
                                           +" block scaling performance");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.title",
                                           "BlockEdgeSize");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.title", "GFlops");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).source",
                                           base + ".blockScaling(#).GFlops.mean");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.data(0).source",
                                           base + ".blockScaling(#).blockSize");
//    report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").options.log_x",true);
        int ges = static_cast<int>( report_sparsegrid_funcs.graphs.template get<double>(
                base + ".blockScaling(0).gridSize.x"));
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).title",
                                           "gridEdge=" + std::to_string(ges));
        ++plotCounter;
    }

    // Sparse 3D
    {
        std::string dim = "3";
        std::string pattern = "sparse";
        std::string linMode = "N";
        std::string base = "performance.SparseGridGpu.device.stencil."+pattern+"."+linMode+"."+dim+"D";

        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").type", "line");
        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").interpolation", "none");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").title",
                                           "SparseGridGPU stencil "+pattern+" "+linMode+" "+dim+"D"
                                           +" grid scaling performance");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.title", "GridEdgeSize");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.title", "GFlops");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).source",
                                           base + ".gridScaling(#).GFlops.mean");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.data(0).source",
                                           base + ".gridScaling(#).gridSize.x");
//    report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").options.log_x", true);
        int bes = static_cast<int>( report_sparsegrid_funcs.graphs.template get<double>(
                base + ".gridScaling(0).blockSize"));
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).title",
                                           "blockEdge=" + std::to_string(bes));
        ++plotCounter;

        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").type", "line");
        report_sparsegrid_funcs.graphs.put("graphs.graph(" + std::to_string(plotCounter) + ").interpolation", "none");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").title",
                                           "SparseGridGPU stencil "+pattern+" "+linMode+" "+dim+"D"
                                           +" block scaling performance");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.title",
                                           "BlockEdgeSize");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.title", "GFlops");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).source",
                                           base + ".blockScaling(#).GFlops.mean");
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").x.data(0).source",
                                           base + ".blockScaling(#).blockSize");
//    report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").options.log_x",true);
        int ges = static_cast<int>( report_sparsegrid_funcs.graphs.template get<double>(
                base + ".blockScaling(0).gridSize.x"));
        report_sparsegrid_funcs.graphs.add("graphs.graph(" + std::to_string(plotCounter) + ").y.data(0).title",
                                           "gridEdge=" + std::to_string(ges));
        ++plotCounter;
    }

//	report_sparsegrid_funcs.graphs.put("graphs.graph(1).type","line");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(1).title","SparseGridGPU insert blocked performance");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(1).x.title","size");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(1).y.title","Milion inserts");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(1).y.data(0).source","performance.SparseGridGpu(#).insert.Minsert.mean");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(1).x.data(0).source","performance.SparseGridGpu(#).insert.gridSize.x");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(1).y.data(0).title","line");
//
//	report_sparsegrid_funcs.graphs.put("graphs.graph(2).type","line");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(2).title","SparseGridGPU insert single performance");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(2).x.title","size");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(2).y.title","Milion inserts");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(2).y.data(0).source","performance.SparseGridGpu(#).insertSingle.Minsert.mean");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(2).x.data(0).source","performance.SparseGridGpu(#).insertSingle.gridSize.x");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(2).y.data(0).title","line");
//
//	report_sparsegrid_funcs.graphs.put("graphs.graph(3).type","line");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(3).title","SparseGridGPU insert single performance");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(3).x.title","size");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(3).y.title","Milion inserts");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(3).y.data(0).source","performance.SparseGridGpu(#).insertStencil.GElems.mean");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(3).x.data(0).source","performance.SparseGridGpu(#).insertStencil.gridSize.x");
//	report_sparsegrid_funcs.graphs.add("graphs.graph(3).y.data(0).title","line");

    std::string file_xml_results(test_dir);
    file_xml_results += std::string("/") + std::string(perfResultsXmlFile);

    boost::property_tree::xml_writer_settings<std::string> settings(' ', 4);
    boost::property_tree::write_xml(file_xml_results, report_sparsegrid_funcs.graphs, std::locale(), settings);

    std::string file_xml_ref(test_dir);
//	file_xml_ref += std::string("/openfpm_pdata/SparseGridGpu_performance_ref.xml"); // This the normal setup
    file_xml_ref += std::string("/") + std::string(perfResultsXmlFile); // This is our setup to get the stdDev on plots

    GoogleChart cg;

    StandardXMLPerformanceGraph(file_xml_results, file_xml_ref, cg, 1);

    addUpdtateTime(cg,1);
    cg.write("SparseGridGpu_performance.html");
}

struct Fixture
{
    Fixture()
    {
        BOOST_TEST_MESSAGE( "Setup fixture" );
    }

    ~Fixture()
    {
        BOOST_TEST_MESSAGE( "Teardown fixture" );
        write_test_report();
    }
};


template<unsigned int p, typename SparseGridType>
__global__ void insertValues2D(SparseGridType sparseGrid, const int offsetX=0, const int offsetY=0)
{
    sparseGrid.init();

    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    int x = bIdX * bDimX + tIdX + offsetX;
    int y = bIdY * bDimY + tIdY + offsetY;
    grid_key_dx<SparseGridType::d, int> coord({x, y});

    sparseGrid.template insert<p>(coord) = x*x*y*y; // some function...

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
}

template<unsigned int p, unsigned int chunksPerBlock, unsigned int blockEdgeSize, typename SparseGridType>
__global__ void insertValues2DBlocked(SparseGridType sparseGrid, const int sOffsetX=0, const int sOffsetY=0)
{
    constexpr unsigned int pMask = SparseGridType::pMask;
    typedef BlockTypeOf<typename SparseGridType::AggregateType, p> BlockT;
    typedef BlockTypeOf<typename SparseGridType::AggregateType, pMask> MaskBlockT;

    sparseGrid.init();

    __shared__ BlockT *blocks[chunksPerBlock];
    __shared__ MaskBlockT *masks[chunksPerBlock];

    int posX = blockIdx.x * blockDim.x + threadIdx.x + sOffsetX;
    int posY = blockIdx.y * blockDim.y + threadIdx.y + sOffsetY;
    const unsigned int offsetX = posX % blockEdgeSize;
    const unsigned int offsetY = posY % blockEdgeSize;

    const unsigned int blockDimX = blockDim.x / blockEdgeSize;
    const unsigned int blockOffsetX = threadIdx.x / blockEdgeSize;
    const unsigned int blockOffsetY = threadIdx.y / blockEdgeSize;

    const unsigned int dataBlockNum = blockOffsetY*blockDimX + blockOffsetX;
    const unsigned int offset = offsetY * blockEdgeSize + offsetX;

//    if (offset == 0) // Just one thread per data block
//    {
        grid_key_dx<SparseGridType::d, int> blockCoord({posX / blockEdgeSize, posY / blockEdgeSize});
        auto encap = sparseGrid.insertBlockNew(sparseGrid.getBlockLinId(blockCoord));
        blocks[dataBlockNum] = &(encap.template get<p>());
        masks[dataBlockNum] = &(encap.template get<pMask>());
//    }

    __syncthreads();

    blocks[dataBlockNum]->block[offset] = posX*posX * posY*posY;
    BlockMapGpu_ker<>::setExist(masks[dataBlockNum]->block[offset]);

    __syncthreads();

    sparseGrid.flush_block_insert();
}



template<unsigned int p, unsigned int chunksPerBlock=1, typename SparseGridType, typename ScalarT>
__global__ void insertConstantValue(SparseGridType sparseGrid, ScalarT value)
{
    constexpr unsigned int pMask = SparseGridType::pMask;
    typedef BlockTypeOf<typename SparseGridType::AggregateType, p> BlockT;
    typedef BlockTypeOf<typename SparseGridType::AggregateType, pMask> MaskBlockT;

    sparseGrid.init();

    __shared__ BlockT *blocks[chunksPerBlock];
    __shared__ MaskBlockT *masks[chunksPerBlock];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    auto pos = sparseGrid.getLinId(coord);
    unsigned int dataBlockId = pos / BlockT::size;
    unsigned int offset = pos % BlockT::size;
    unsigned int dataBlockNum = dataBlockId % chunksPerBlock;

//    if (offset == 0) // Just one thread per data block
//    {
        auto encap = sparseGrid.insertBlockNew(dataBlockId);
        blocks[dataBlockNum] = &(encap.template get<p>());
        masks[dataBlockNum] = &(encap.template get<pMask>());
//    }

    __syncthreads();
    blocks[dataBlockNum]->block[offset] = value;
    BlockMapGpu_ker<>::setExist(masks[dataBlockNum]->block[offset]);

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
    z++;
}


template<unsigned int p, typename SparseGridType, typename ValueT>
__global__ void insertOneValue(SparseGridType sparseGrid, dim3 pt, ValueT value)
{
    sparseGrid.init();

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    dim3 thCoord(x, y, z);
    if (thCoord.x == pt.x && thCoord.y == pt.y && thCoord.z == pt.z)
    {
        grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});
        sparseGrid.template insert<p>(coord) = value;
    }
    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType, typename VectorOutType>
__global__ void copyBlocksToOutput(SparseGridType sparseGrid, VectorOutType output)
{
    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    size_t pos = sparseGrid.getLinId(coord);

    auto value = sparseGrid.template get<p>(coord);

    output.template get<p>(pos) = value;

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int dim, unsigned int p_src, unsigned int p_dst>
struct HeatStencil
{
	typedef NNStar stencil_type;

    // This is an example of a laplacian smoothing stencil to apply using the apply stencil facility of SparseGridGpu

    static constexpr unsigned int flops = 3 + 2*dim;

    static constexpr unsigned int supportRadius = 1;

    /*! \brief Stencil function
     *
     * \param sparseGrid This is the sparse grid data-structure
     * \param dataBlockId The id of the block
     * \param offset index in local coordinate of the point where we are working
	 * \param dataBlockLoad dataBlock from where we read
	 * \param dataBlockStore dataBlock from where we write
	 * \param isActive the point is active if exist and is not padding
	 * \param dt delta t
     *
     *
     */
    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __device__ void stencil(
            SparseGridT & sparseGrid,
            const unsigned int dataBlockId,
            const openfpm::sparse_index<unsigned int> dataBlockIdPos,
            const unsigned int offset,
            const grid_key_dx<dim, int> & pointCoord,
            const DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore,
            bool isActive,
            float dt)
    {
        typedef typename SparseGridT::AggregateBlockType AggregateT;
        typedef ScalarTypeOf<AggregateT, p_src> ScalarT;

        constexpr unsigned int enlargedBlockSize = IntPow<
                SparseGridT::getBlockEdgeSize() + 2 * supportRadius, dim>::value;

        __shared__ ScalarT enlargedBlock[enlargedBlockSize];

        sparseGrid.loadGhostBlock<p_src>(dataBlockLoad, dataBlockIdPos, enlargedBlock);

        __syncthreads();

        if (isActive)
        {
            const auto coord = sparseGrid.getCoordInEnlargedBlock(offset);
            const auto linId = sparseGrid.getLinIdInEnlargedBlock(offset);
            ScalarT cur = enlargedBlock[linId];
            ScalarT laplacian = -2.0 * dim * cur; // The central part of the stencil

            for (int d = 0; d < dim; ++d)
            {
                auto nPlusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, 1);
                auto nMinusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, -1);
                ScalarT neighbourPlus = enlargedBlock[nPlusId];
                ScalarT neighbourMinus = enlargedBlock[nMinusId];
                laplacian += neighbourMinus + neighbourPlus;
            }
            enlargedBlock[linId] = cur + dt * laplacian;
        }
    }

    /*! \brief Stencil Host function
    *
    * \param sparseGrid This is the sparse grid data-structure
    * \param dataBlockId The id of the block
    * \param offset index in local coordinate of the point where we are working
    * \param dataBlockLoad dataBlock from where we read
    * \param dataBlockStore dataBlock from where we write
    * \param isActive the point is active if exist and is not padding
    * \param dt delta t
    *
    *
    */
    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __host__ void stencilHost(
            SparseGridT & sparseGrid,
            const unsigned int dataBlockId,
            const openfpm::sparse_index<unsigned int> dataBlockIdPos,
            const unsigned int offset,
            const grid_key_dx<dim, int> & pointCoord,
            const DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore,
            bool isActive,
            float dt)
    {
        constexpr unsigned int blockEdgeSize = SparseGridT::getBlockEdgeSize();

        if (isActive)
        {
            auto cur = dataBlockLoad.template get<p_src>()[offset];
            auto laplacian = -2.0 * dim * cur; // The central part of the stencil

            auto neighbourCoord = pointCoord;
            auto counter = offset;
            unsigned int dimStride = 1;
            for (int d = 0; d < dim; ++d)
            {
                const auto localOffset = counter % blockEdgeSize;

                if (localOffset == 0) // This means we are at the lower boundary for this dimension
                {
                    neighbourCoord.set_d(d, neighbourCoord.get(d) - 1);
                    laplacian += sparseGrid.template get<p_src>(neighbourCoord);
                    neighbourCoord.set_d(d, neighbourCoord.get(d) + 1);
                }
                else
                {
                    laplacian += dataBlockLoad.template get<p_src>()[offset - dimStride];
                }
                if (localOffset == blockEdgeSize - 1) // This means we are at the lower boundary for this dimension
                {
                neighbourCoord.set_d(d, neighbourCoord.get(d) + 1);
                laplacian += sparseGrid.template get<p_src>(neighbourCoord);
                neighbourCoord.set_d(d, neighbourCoord.get(d) - 1);
                }
                else
                {
                    laplacian += dataBlockLoad.template get<p_src>()[offset + dimStride];
                }
                //
                counter /= blockEdgeSize;
                dimStride *= blockEdgeSize;
            }
            dataBlockStore.template get<p_dst>()[offset] = cur + dt * laplacian;
        }
    }

    template <typename SparseGridT, typename CtxT>
    static inline void __host__ flush(SparseGridT & sparseGrid, CtxT & ctx)
    {
        sparseGrid.template flush <sRight_<0>> (ctx, flush_type::FLUSH_ON_DEVICE);
    }
};

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
void testStencilHeat_perf(unsigned int i, std::string base)
{
    auto testName = "In-place stencil";
    typedef HeatStencil<SparseGridZ::dims,0,1> StencilT;

    report_sparsegrid_funcs.graphs.put(base + ".dim",2);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*SparseGridZ::blockEdgeSize_);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*SparseGridZ::blockEdgeSize_);

    unsigned int iterations = 100;

    openfpm::vector<double> measures_gf;
    openfpm::vector<double> measures_tm;

	dim3 gridSize(gridEdgeSize, gridEdgeSize);
	dim3 blockSize(SparseGridZ::blockEdgeSize_,SparseGridZ::blockEdgeSize_);
	typename SparseGridZ::grid_info blockGeometry(gridSize);
	SparseGridZ sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

    unsigned long long numElements = gridEdgeSize*SparseGridZ::blockEdgeSize_*gridEdgeSize*SparseGridZ::blockEdgeSize_;

	// Initialize the grid
	sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
	CUDA_LAUNCH_DIM3((insertConstantValue<0>),gridSize, blockSize,sparseGrid.toKernel(), 0);
	sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
	dim3 sourcePt(gridSize.x * SparseGridZ::blockEdgeSize_ / 2, gridSize.y * SparseGridZ::blockEdgeSize_ / 2, 0);
	insertOneValue<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), sourcePt, 100);
	sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!

	for (unsigned int iter=0; iter<iterations; ++iter)
	{
		hipDeviceSynchronize();

		timer ts;
		ts.start();

		sparseGrid.template applyStencils<StencilT>(STENCIL_MODE_INPLACE, 0.1);

		hipDeviceSynchronize();
		ts.stop();

		measures_tm.add(ts.getwct());

	    float gElemS = numElements / (1e9 * ts.getwct());
	    float gFlopsS = gElemS * StencilT::flops;

		measures_gf.add(gFlopsS);
	}

	double mean_tm = 0;
	double deviation_tm = 0;
	standard_deviation(measures_tm,mean_tm,deviation_tm);

	double mean_gf = 0;
	double deviation_gf = 0;
	standard_deviation(measures_gf,mean_gf,deviation_gf);

    // All times above are in ms

    float gElemS = numElements / (1e9 * mean_tm);
    float gFlopsS = gElemS * StencilT::flops;
    std::cout << "Test: " << testName << std::endl;
    std::cout << "Block: " << SparseGridZ::blockEdgeSize_ << "x" << SparseGridZ::blockEdgeSize_ << std::endl;
    std::cout << "Grid: " << gridEdgeSize*SparseGridZ::blockEdgeSize_ << "x" << gridEdgeSize*SparseGridZ::blockEdgeSize_ << std::endl;
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;

    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeat_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilN");

    testStencilHeat_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize>>(i, base);
    hipDeviceSynchronize();
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeatZ_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilZ");

    testStencilHeat_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu_z<dim, AggregateT, blockEdgeSize, chunkSize>>(i, base);
    hipDeviceSynchronize();
}

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
void testStencilHeat3D_perf(unsigned int i, std::string base)
{
    auto testName = "In-place 3D stencil";
//    unsigned int gridEdgeSize = 128;
//    unsigned int gridEdgeSize = 64;
    typedef HeatStencil<SparseGridZ::dims,0,1> StencilT;

    report_sparsegrid_funcs.graphs.put(base + ".dim",3);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*SparseGridZ::blockEdgeSize_);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*SparseGridZ::blockEdgeSize_);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.z",gridEdgeSize*SparseGridZ::blockEdgeSize_);

    unsigned int iterations = 100;

    openfpm::vector<double> measures_gf;
    openfpm::vector<double> measures_tm;

    dim3 gridSize(gridEdgeSize, gridEdgeSize, gridEdgeSize);
    dim3 blockSize(SparseGridZ::blockEdgeSize_, SparseGridZ::blockEdgeSize_, SparseGridZ::blockEdgeSize_);

    typename SparseGridZ::grid_info blockGeometry(gridSize);
    SparseGridZ sparseGrid(blockGeometry);
    mgpu::ofp_context_t ctx;
    sparseGrid.template setBackgroundValue<0>(0);

    unsigned long long numElements = gridEdgeSize*SparseGridZ::blockEdgeSize_
            *gridEdgeSize*SparseGridZ::blockEdgeSize_
            *gridEdgeSize*SparseGridZ::blockEdgeSize_;

    // Initialize the grid
    sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
    CUDA_LAUNCH_DIM3((insertConstantValue<0>),gridSize, blockSize,sparseGrid.toKernel(), 0);
    sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
    dim3 sourcePt(gridSize.x * SparseGridZ::blockEdgeSize_ / 2,
            gridSize.y * SparseGridZ::blockEdgeSize_ / 2,
            gridSize.z * SparseGridZ::blockEdgeSize_ / 2);
    insertOneValue<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), sourcePt, 100);
    sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!

    for (unsigned int iter=0; iter<iterations; ++iter)
    {
        hipDeviceSynchronize();

        timer ts;
        ts.start();

        sparseGrid.template applyStencils<StencilT>(STENCIL_MODE_INPLACE, 0.1);

        hipDeviceSynchronize();
        ts.stop();

        measures_tm.add(ts.getwct());

        float gElemS = numElements / (1e9 * ts.getwct());
        float gFlopsS = gElemS * StencilT::flops;

        measures_gf.add(gFlopsS);
    }

    double mean_tm = 0;
    double deviation_tm = 0;
    standard_deviation(measures_tm,mean_tm,deviation_tm);

    double mean_gf = 0;
    double deviation_gf = 0;
    standard_deviation(measures_gf,mean_gf,deviation_gf);

    // All times above are in ms

    float gElemS = numElements / (1e9 * mean_tm);
    float gFlopsS = gElemS * StencilT::flops;
    std::cout << "Test: " << testName << std::endl;
    std::cout << "Block: " << SparseGridZ::blockEdgeSize_
              << "x" << SparseGridZ::blockEdgeSize_
              << "x" << SparseGridZ::blockEdgeSize_
              << std::endl;
    std::cout << "Grid: " << gridEdgeSize*SparseGridZ::blockEdgeSize_
        << "x" << gridEdgeSize*SparseGridZ::blockEdgeSize_
        << "x" << gridEdgeSize*SparseGridZ::blockEdgeSize_
        << std::endl;
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;

    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeat3D_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 3;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilN3D");

    testStencilHeat3D_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize>>(i, base);
    hipDeviceSynchronize();
}

//template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
//void testStencilHeatSparse_perf(unsigned int i, std::string base)
//{
//    auto testName = "In-place sparse stencil";
////    unsigned int gridEdgeSize = 128;
//    constexpr unsigned int dim = SparseGridZ::dims;
////    const unsigned int blockEdgeSize = SparseGridZ::blockEdgeSize_;
//
//    typedef HeatStencil<dim, 0, 1> Stencil01T;
//    typedef HeatStencil<dim, 1, 0> Stencil10T;
//
////    std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").stencil");
//
//    report_sparsegrid_funcs.graphs.put(base + ".dim",2);
//    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
//    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*blockEdgeSize);
//    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*blockEdgeSize);
//
//    unsigned int iterations = 100;
//
//    openfpm::vector<double> measures_gf;
//    openfpm::vector<double> measures_tm;
//
//    dim3 gridSize(gridEdgeSize, gridEdgeSize);
//    dim3 blockSize(blockEdgeSize,blockEdgeSize);
//    size_t sz[2] = {1000000,1000000};
//    typename SparseGridZ::grid_info blockGeometry(sz);
//    SparseGridZ sparseGrid(blockGeometry);
//    mgpu::ofp_context_t ctx;
//    sparseGrid.template setBackgroundValue<0>(0);
//
//    ///// Insert sparse content, a set of 3 hollow spheres /////
//    constexpr unsigned int rBig = gridEdgeSize * blockEdgeSize / 2;
//    constexpr unsigned int rSmall = rBig/2;
//    constexpr unsigned int rBig2 = rBig;
//    constexpr unsigned int rSmall2 = rBig2 - (rBig2/16);
//    constexpr unsigned int rBig3 = rBig/8;
//    constexpr unsigned int rSmall3 = rBig3 - (rBig3/10);
//    // Sphere 1
//    grid_key_dx<2,int> start1({500000,500000});
//    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
//    CUDA_LAUNCH_DIM3((insertSphere<0>),
//                     gridSize, dim3(blockEdgeSize*blockEdgeSize,1,1),
//                     sparseGrid.toKernel(), start1, rBig, rSmall, 1);
//    hipDeviceSynchronize();
//    sparseGrid.template flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
//    hipDeviceSynchronize();
//
//    // Sphere 2
//    grid_key_dx<2,int> start2({500000+rBig,500000+rBig});
//    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
//    CUDA_LAUNCH_DIM3((insertSphere<0>),
//                     gridSize, dim3(blockEdgeSize*blockEdgeSize,1,1),
//                     sparseGrid.toKernel(), start2, rBig2, rSmall2, 1);
//    hipDeviceSynchronize();
//    sparseGrid.template flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
//    hipDeviceSynchronize();
//
//    // Sphere 3
//    grid_key_dx<2,int> start3({500000+rBig,500000});
//    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
//    CUDA_LAUNCH_DIM3((insertSphere<0>),
//                     gridSize, dim3(blockEdgeSize*blockEdgeSize,1,1),
//                     sparseGrid.toKernel(), start3, rBig3, rSmall3, 1);
//    hipDeviceSynchronize();
//    sparseGrid.template flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
//    hipDeviceSynchronize();
//    ///// /////
//
//    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
//    sparseGrid.tagBoundaries();
//
//    sparseGrid.template deviceToHost<0>(); // NECESSARY as count takes place on Host!
//    auto existingElements = sparseGrid.countExistingElements();
//    auto boundaryElements = sparseGrid.countBoundaryElements();
//    unsigned long long numElements = existingElements - boundaryElements;
//
//    // Now apply some boundary conditions
//    sparseGrid.template applyBoundaryStencils<BoundaryStencilSetXRescaled<dim,0,0>>(
//            500000, 500000+(2*rBig),
//            0.0, 10.0);
//    hipDeviceSynchronize();
//
//    for (unsigned int iter=0; iter<iterations; ++iter)
//    {
//        hipDeviceSynchronize();
//
//        timer ts;
//        ts.start();
//
//        sparseGrid.template applyStencils<Stencil01T>(STENCIL_MODE_INPLACE, 0.1);
//        hipDeviceSynchronize();
//        sparseGrid.template applyStencils<Stencil10T>(STENCIL_MODE_INPLACE, 0.1);
//        hipDeviceSynchronize();
//
//        ts.stop();
//
//        measures_tm.add(ts.getwct());
//
//        float gElemS = numElements / (1e9 * ts.getwct());
//        float gFlopsS = gElemS * Stencil01T::flops;
//
//        measures_gf.add(gFlopsS);
//    }
//
//    double mean_tm = 0;
//    double deviation_tm = 0;
//    standard_deviation(measures_tm,mean_tm,deviation_tm);
//
//    double mean_gf = 0;
//    double deviation_gf = 0;
//    standard_deviation(measures_gf,mean_gf,deviation_gf);
//
//    // All times above are in ms
//
//    float gElemS = numElements / (1e9 * mean_tm);
//    float gFlopsS = gElemS * Stencil01T::flops;
//    std::cout << "Test: " << testName << std::endl;
//    std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << std::endl;
//    std::cout << "Grid: " << gridEdgeSize*blockEdgeSize << "x" << gridEdgeSize*blockEdgeSize << std::endl;
//    double dataOccupancyMean, dataOccupancyDev;
//    sparseGrid.deviceToHost();
//    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
//    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
//    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
//    std::cout << "Iterations: " << iterations << std::endl;
//    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
//    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;
//
//    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
//    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
//    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
//    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);
//
////    // DEBUG
////    sparseGrid.template deviceToHost<0,1>();
////    sparseGrid.write("SparseGridGPU_testStencilHeatSparse_perf_DEBUG.vtk");
//}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
void testStencilHeatSparse_perf(unsigned int i, std::string base)
{
    auto testName = "In-place sparse stencil";
//    unsigned int gridEdgeSize = 128;
    constexpr unsigned int dim = SparseGridZ::dims;
//    const unsigned int blockEdgeSize = SparseGridZ::blockEdgeSize_;

    typedef HeatStencil<dim, 0, 1> Stencil01T;
    typedef HeatStencil<dim, 1, 0> Stencil10T;

//    std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").stencil");

    report_sparsegrid_funcs.graphs.put(base + ".dim",2);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*blockEdgeSize);

    unsigned int iterations = 100;

    openfpm::vector<double> measures_gf;
    openfpm::vector<double> measures_tm;

    dim3 gridSize(gridEdgeSize, gridEdgeSize);
    dim3 blockSize(blockEdgeSize,blockEdgeSize);
    unsigned int spatialEdgeSize = 1000000;
    size_t sz[2] = {spatialEdgeSize, spatialEdgeSize};
    typename SparseGridZ::grid_info blockGeometry(sz);
    SparseGridZ sparseGrid(blockGeometry);
    mgpu::ofp_context_t ctx;
    sparseGrid.template setBackgroundValue<0>(0);

    ///// Insert sparse content, a set of concentric spheres /////
    const unsigned int numSpheres = gridEdgeSize / 4;
//    const unsigned int numSpheres = 1;
    unsigned int centerPoint = spatialEdgeSize / 2;

    for (int i = 1; i <= numSpheres; ++i)
    {
        unsigned int rBig = 2*i * blockEdgeSize;
        unsigned int rSmall = (2*i-1) * blockEdgeSize;
        // Sphere i-th
        grid_key_dx<dim, int> start1({centerPoint, centerPoint});
        sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
        CUDA_LAUNCH_DIM3((insertSphere<0>),
                         gridSize, dim3(blockEdgeSize * blockEdgeSize, 1, 1),
                         sparseGrid.toKernel(), start1, rBig, rSmall, 5);
        hipDeviceSynchronize();
        sparseGrid.template flush<smax_<0 >>(ctx, flush_type::FLUSH_ON_DEVICE);
        hipDeviceSynchronize();
    }
    ///// /////

    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
    sparseGrid.tagBoundaries();

    sparseGrid.template deviceToHost<0>(); // NECESSARY as count takes place on Host!
    auto existingElements = sparseGrid.countExistingElements();
    auto boundaryElements = sparseGrid.countBoundaryElements();
    unsigned long long numElements = existingElements - boundaryElements;

    // Now apply some boundary conditions
    sparseGrid.template applyBoundaryStencils<BoundaryStencilSetXRescaled<dim,0,0>>(
            centerPoint, centerPoint + 2*blockEdgeSize*gridEdgeSize,
            0.0, 10.0);
    hipDeviceSynchronize();

    for (unsigned int iter=0; iter<iterations; ++iter)
    {
        hipDeviceSynchronize();

        timer ts;
        ts.start();

        sparseGrid.template applyStencils<Stencil01T>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();
        sparseGrid.template applyStencils<Stencil10T>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();

        ts.stop();

        measures_tm.add(ts.getwct());

        float gElemS = numElements / (1e9 * ts.getwct());
        float gFlopsS = gElemS * Stencil01T::flops;

        measures_gf.add(gFlopsS);
    }

    double mean_tm = 0;
    double deviation_tm = 0;
    standard_deviation(measures_tm,mean_tm,deviation_tm);

    double mean_gf = 0;
    double deviation_gf = 0;
    standard_deviation(measures_gf,mean_gf,deviation_gf);

    // All times above are in ms

    float gElemS = numElements / (1e9 * mean_tm);
    float gFlopsS = gElemS * Stencil01T::flops;
    std::cout << "Test: " << testName << std::endl;
    std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << std::endl;
    std::cout << "Grid: " << gridEdgeSize*blockEdgeSize << "x" << gridEdgeSize*blockEdgeSize << std::endl;
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;

    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);

//    // DEBUG
//    sparseGrid.template deviceToHost<0,1>();
//    sparseGrid.write("SparseGridGPU_testStencilHeatSparse_perf_DEBUG.vtk");
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeatSparse_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilNSparse");

    testStencilHeatSparse_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize, long int>>(i, base);
    hipDeviceSynchronize();
}

//template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
//void testStencilHeat3DSparse_perf(unsigned int i, std::string base)
//{
//    auto testName = "In-place 3D sparse stencil";
////    unsigned int gridEdgeSize = 32;
//    constexpr unsigned int dim = SparseGridZ::dims;
////    const unsigned int blockEdgeSize = SparseGridZ::blockEdgeSize_;
//
//    typedef HeatStencil<dim, 0, 1> Stencil01T;
//    typedef HeatStencil<dim, 1, 0> Stencil10T;
//
////    std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").stencil");
//
//    report_sparsegrid_funcs.graphs.put(base + ".dim",dim);
//    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
//    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x", gridEdgeSize * blockEdgeSize);
//    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y", gridEdgeSize * blockEdgeSize);
//    report_sparsegrid_funcs.graphs.put(base + ".gridSize.z", gridEdgeSize * blockEdgeSize);
//
//    unsigned int iterations = 100;
//
//    openfpm::vector<double> measures_gf;
//    openfpm::vector<double> measures_tm;
//
//    dim3 gridSize(gridEdgeSize, gridEdgeSize, gridEdgeSize);
//    dim3 blockSize(blockEdgeSize, blockEdgeSize, blockEdgeSize);
//    unsigned int spatialEdgeSize = 10000;
//    size_t sz[3] = {spatialEdgeSize, spatialEdgeSize, spatialEdgeSize};
//    typename SparseGridZ::grid_info blockGeometry(sz);
//    SparseGridZ sparseGrid(blockGeometry);
//    mgpu::ofp_context_t ctx;
//    sparseGrid.template setBackgroundValue<0>(0);
//
//    ///// Insert sparse content, a set of 3 hollow spheres /////
//    constexpr unsigned int rBig = gridEdgeSize * blockEdgeSize / 2;
//    constexpr unsigned int rSmall = rBig/2;
//    constexpr unsigned int rBig2 = rBig;
//    constexpr unsigned int rSmall2 = rBig2 - (rBig2/3);
//    constexpr unsigned int rBig3 = rBig/4;
//    constexpr unsigned int rSmall3 = rBig3 - (rBig3/4);
//    // Sphere 1
//    unsigned int centerPoint = spatialEdgeSize/2;
//    grid_key_dx<dim,int> start1({centerPoint, centerPoint, centerPoint});
//    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
//    CUDA_LAUNCH_DIM3((insertSphere3D<0>),
//                     gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
//                     sparseGrid.toKernel(), start1, rBig, rSmall, 1);
//    hipDeviceSynchronize();
//    sparseGrid.template flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
//    hipDeviceSynchronize();
//
//    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
//    sparseGrid.tagBoundaries();
//
//    // Sphere 2
//    grid_key_dx<dim,int> start2({centerPoint - rBig, centerPoint - rBig, centerPoint - rBig});
//    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
//    CUDA_LAUNCH_DIM3((insertSphere3D<0>),
//                     gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
//                     sparseGrid.toKernel(), start2, rBig2, rSmall2, 1);
//    hipDeviceSynchronize();
//    sparseGrid.template flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
//    hipDeviceSynchronize();
//
//    // Sphere 3
//    grid_key_dx<dim,int> start3({centerPoint + rBig, centerPoint - rBig, centerPoint - rBig});
//    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
//    CUDA_LAUNCH_DIM3((insertSphere3D<0>),
//                     gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
//                     sparseGrid.toKernel(), start3, rBig3, rSmall3, 1);
//    hipDeviceSynchronize();
//    sparseGrid.template flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
//    hipDeviceSynchronize();
//    ///// /////
//
//    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
//    sparseGrid.tagBoundaries();
//
//    sparseGrid.template deviceToHost<0>(); // NECESSARY as count takes place on Host!
//    auto existingElements = sparseGrid.countExistingElements();
//    auto boundaryElements = sparseGrid.countBoundaryElements();
//    unsigned long long numElements = existingElements - boundaryElements;
//
//    // Now apply some boundary conditions
//    sparseGrid.template applyBoundaryStencils<BoundaryStencilSetXRescaled<dim,0,0>>(
//            centerPoint - rBig, centerPoint + (2 * rBig),
//            0.0, 10.0);
//    hipDeviceSynchronize();
//
//    for (unsigned int iter=0; iter<iterations; ++iter)
//    {
//        hipDeviceSynchronize();
//
//        timer ts;
//        ts.start();
//
//        sparseGrid.template applyStencils<Stencil01T>(STENCIL_MODE_INPLACE, 0.1);
//        hipDeviceSynchronize();
//        sparseGrid.template applyStencils<Stencil10T>(STENCIL_MODE_INPLACE, 0.1);
//        hipDeviceSynchronize();
//
//        ts.stop();
//
//        measures_tm.add(ts.getwct());
//
//        float gElemS = numElements / (1e9 * ts.getwct());
//        float gFlopsS = gElemS * Stencil01T::flops;
//
//        measures_gf.add(gFlopsS);
//    }
//
//    double mean_tm = 0;
//    double deviation_tm = 0;
//    standard_deviation(measures_tm,mean_tm,deviation_tm);
//
//    double mean_gf = 0;
//    double deviation_gf = 0;
//    standard_deviation(measures_gf,mean_gf,deviation_gf);
//
//    // All times above are in ms
//
//    float gElemS = numElements / (1e9 * mean_tm);
//    float gFlopsS = gElemS * Stencil01T::flops;
//    std::cout << "Test: " << testName << std::endl;
//    std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << "x" << blockEdgeSize << std::endl;
//    std::cout << "Grid: " << gridEdgeSize * blockEdgeSize
//              << "x" << gridEdgeSize * blockEdgeSize
//              << "x" << gridEdgeSize * blockEdgeSize
//              << std::endl;
//    double dataOccupancyMean, dataOccupancyDev;
//    sparseGrid.deviceToHost();
//    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
//    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
//    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
//    std::cout << "Iterations: " << iterations << std::endl;
//    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
//    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;
//
//    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
//    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
//    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
//    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);
//
////    // DEBUG
////    sparseGrid.template deviceToHost<0,1>();
////    sparseGrid.write("SparseGridGPU_testStencilHeat3DSparse_perf_DEBUG.vtk");
//}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
void testStencilHeat3DSparse_perf(unsigned int i, std::string base)
{
    auto testName = "In-place 3D sparse stencil";
//    unsigned int gridEdgeSize = 32;
    constexpr unsigned int dim = SparseGridZ::dims;
//    const unsigned int blockEdgeSize = SparseGridZ::blockEdgeSize_;

    typedef HeatStencil<dim, 0, 1> Stencil01T;
    typedef HeatStencil<dim, 1, 0> Stencil10T;

//    std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").stencil");

    report_sparsegrid_funcs.graphs.put(base + ".dim",dim);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x", gridEdgeSize * blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y", gridEdgeSize * blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.z", gridEdgeSize * blockEdgeSize);

    unsigned int iterations = 100;

    openfpm::vector<double> measures_gf;
    openfpm::vector<double> measures_tm;

    dim3 gridSize(gridEdgeSize, gridEdgeSize, gridEdgeSize);
    dim3 blockSize(blockEdgeSize, blockEdgeSize, blockEdgeSize);
    unsigned int spatialEdgeSize = 10000;
    size_t sz[3] = {spatialEdgeSize, spatialEdgeSize, spatialEdgeSize};
    typename SparseGridZ::grid_info blockGeometry(sz);
    SparseGridZ sparseGrid(blockGeometry);
    mgpu::ofp_context_t ctx;
    sparseGrid.template setBackgroundValue<0>(0);

    ///// Insert sparse content, a set of concentric spheres /////
    const unsigned int numSpheres = gridEdgeSize / 4;
    unsigned int centerPoint = spatialEdgeSize / 2;

    for (int i = 1; i <= numSpheres; ++i)
    {
        unsigned int rBig = 2*i * blockEdgeSize;
        unsigned int rSmall = (2*i-1) * blockEdgeSize;
        // Sphere i-th
        grid_key_dx<dim, int> start1({centerPoint, centerPoint, centerPoint});
        sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
        CUDA_LAUNCH_DIM3((insertSphere3D<0>),
                         gridSize, dim3(blockEdgeSize * blockEdgeSize * blockEdgeSize, 1, 1),
                         sparseGrid.toKernel(), start1, rBig, rSmall, 1);
        hipDeviceSynchronize();
        sparseGrid.template flush<smax_<0 >>(ctx, flush_type::FLUSH_ON_DEVICE);
        hipDeviceSynchronize();
    }
    ///// /////

    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
    sparseGrid.tagBoundaries();

    sparseGrid.template deviceToHost<0>(); // NECESSARY as count takes place on Host!
    auto existingElements = sparseGrid.countExistingElements();
    auto boundaryElements = sparseGrid.countBoundaryElements();
    unsigned long long numElements = existingElements - boundaryElements;

    // Now apply some boundary conditions
    sparseGrid.template applyBoundaryStencils<BoundaryStencilSetXRescaled<dim,0,0>>(
            centerPoint, centerPoint + 2*blockEdgeSize*gridEdgeSize,
            0.0, 10.0);
    hipDeviceSynchronize();

    for (unsigned int iter=0; iter<iterations; ++iter)
    {
        hipDeviceSynchronize();

        timer ts;
        ts.start();

        sparseGrid.template applyStencils<Stencil01T>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();
        sparseGrid.template applyStencils<Stencil10T>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();

        ts.stop();

        measures_tm.add(ts.getwct());

        float gElemS = numElements / (1e9 * ts.getwct());
        float gFlopsS = gElemS * Stencil01T::flops;

        measures_gf.add(gFlopsS);
    }

    double mean_tm = 0;
    double deviation_tm = 0;
    standard_deviation(measures_tm,mean_tm,deviation_tm);

    double mean_gf = 0;
    double deviation_gf = 0;
    standard_deviation(measures_gf,mean_gf,deviation_gf);

    // All times above are in ms

    float gElemS = numElements / (1e9 * mean_tm);
    float gFlopsS = gElemS * Stencil01T::flops;
    std::cout << "Test: " << testName << std::endl;
    std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << "x" << blockEdgeSize << std::endl;
    std::cout << "Grid: " << gridEdgeSize * blockEdgeSize
              << "x" << gridEdgeSize * blockEdgeSize
              << "x" << gridEdgeSize * blockEdgeSize
              << std::endl;
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;

    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);

//    // DEBUG
//    sparseGrid.template deviceToHost<0,1>();
//    sparseGrid.write("SparseGridGPU_testStencilHeat3DSparse_perf_DEBUG.vtk");
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeat3DSparse_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 3;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilN3DSparse");

    testStencilHeat3DSparse_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize, long int>>(i, base);
    hipDeviceSynchronize();
}

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
void testStencilHeatHost_perf(unsigned int i, std::string base)
{
    // todo: Make sure to reimplement the host stencil application function to pre-load to a block of memory both content and ghost
    // this way we can avoid binary searches...
    auto testName = "In-place stencil HOST";
    typedef HeatStencil<SparseGridZ::dims,0,1> StencilT;

    constexpr unsigned int dim = 2;

//    std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").stencil");

    report_sparsegrid_funcs.graphs.put(base + ".dim",dim);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*SparseGridZ::blockEdgeSize_);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*SparseGridZ::blockEdgeSize_);

//    unsigned int iterations = 100;
    unsigned int iterations = 10;
//    unsigned int iterations = 2;
//    unsigned int iterations = 1; // Debug

    openfpm::vector<double> measures_gf;
    openfpm::vector<double> measures_tm;

    dim3 gridSize(gridEdgeSize, gridEdgeSize);
    dim3 blockSize(SparseGridZ::blockEdgeSize_,SparseGridZ::blockEdgeSize_);
    typename SparseGridZ::grid_info blockGeometry(gridSize);
    SparseGridZ sparseGrid(blockGeometry);
    mgpu::ofp_context_t ctx;
    sparseGrid.template setBackgroundValue<0>(0);

    unsigned long long numElements = gridEdgeSize*SparseGridZ::blockEdgeSize_*gridEdgeSize*SparseGridZ::blockEdgeSize_;

    // Initialize the grid
    sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
    CUDA_LAUNCH_DIM3((insertConstantValue<0>),gridSize, blockSize,sparseGrid.toKernel(), 0);
    sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
    dim3 sourcePt(gridSize.x * SparseGridZ::blockEdgeSize_ / 2, gridSize.y * SparseGridZ::blockEdgeSize_ / 2, 0);
    insertOneValue<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), sourcePt, 100);
    sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
    hipDeviceSynchronize();

    sparseGrid.template deviceToHost<0>();

    for (unsigned int iter=0; iter<iterations; ++iter)
    {
        hipDeviceSynchronize();

        timer ts;
        ts.start();

        sparseGrid.template applyStencilsHost<StencilT>(STENCIL_MODE_INPLACE, 0.1);

        hipDeviceSynchronize();
        ts.stop();

        measures_tm.add(ts.getwct());

        float gElemS = numElements / (1e9 * ts.getwct());
        float gFlopsS = gElemS * StencilT::flops;

        measures_gf.add(gFlopsS);
    }

    double mean_tm = 0;
    double deviation_tm = 0;
    standard_deviation(measures_tm,mean_tm,deviation_tm);

    double mean_gf = 0;
    double deviation_gf = 0;
    standard_deviation(measures_gf,mean_gf,deviation_gf);

    // All times above are in ms

    float gElemS = numElements / (1e9 * mean_tm);
    float gFlopsS = gElemS * StencilT::flops;

    std::cout << "Test: " << testName << std::endl;
    std::cout << "Host: " << SparseGridZ::blockEdgeSize_ << "x" << SparseGridZ::blockEdgeSize_ << std::endl;
    std::cout << "Grid: " << gridEdgeSize*SparseGridZ::blockEdgeSize_ << "x" << gridEdgeSize*SparseGridZ::blockEdgeSize_ << std::endl;
    double dataOccupancyMean=0, dataOccupancyDev=0;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl
                << "\t " << gFlopsS << " GFlops/s" << std::endl;

    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeatHost_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilN_Host");

    testStencilHeatHost_perf<blockEdgeSize, gridEdgeSize,
        SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize>>(i, base);
}

BOOST_AUTO_TEST_SUITE(performance, *boost::unit_test::fixture<Fixture>())

BOOST_AUTO_TEST_SUITE(SparseGridGpu_test)

//BOOST_AUTO_TEST_CASE(testStencilHeatHost_gridScaling)
//{
//    std::string testURI = suiteURI + ".host.stencil.dense.N.2D.gridScaling";
//    unsigned int counter = 0;
//    launch_testStencilHeatHost_perf<8, 128>(testURI, counter++);
//    launch_testStencilHeatHost_perf<8, 256>(testURI, counter++);
//    launch_testStencilHeatHost_perf<8, 512>(testURI, counter++);
//    launch_testStencilHeatHost_perf<8, 1024>(testURI, counter++);
////    launch_testStencilHeatHost_perf<8, 2048>(testURI, counter++);
//}
//
//BOOST_AUTO_TEST_CASE(testStencilHeatHost_blockScaling)
//{
//    std::string testURI = suiteURI + ".host.stencil.dense.N.2D.blockScaling";
//    unsigned int counter = 0;
//    launch_testStencilHeatHost_perf<4, 2048>(testURI, counter++);
//    launch_testStencilHeatHost_perf<8, 1024>(testURI, counter++);
//    launch_testStencilHeatHost_perf<16, 512>(testURI, counter++);
//    launch_testStencilHeatHost_perf<32, 256>(testURI, counter++);
//}

BOOST_AUTO_TEST_CASE(testStencilHeat_gridScaling)
{
    std::string testURI = suiteURI + ".device.stencil.dense.N.2D.gridScaling";
    unsigned int counter = 0;
    constexpr unsigned int blockEdgeSize = 16;
    launch_testStencilHeat_perf<blockEdgeSize, 128>(testURI, counter++);
    launch_testStencilHeat_perf<blockEdgeSize, 256>(testURI, counter++);
    launch_testStencilHeat_perf<blockEdgeSize, 512>(testURI, counter++);
    launch_testStencilHeat_perf<blockEdgeSize, 1024>(testURI, counter++);
//    launch_testStencilHeat_perf<blockEdgeSize, 2048>(testURI, counter++);
}

BOOST_AUTO_TEST_CASE(testStencilHeat_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencil.dense.N.2D.blockScaling";
    unsigned int counter = 0;
    // Note - blockEdgeSize == 2 doesn't work
    launch_testStencilHeat_perf<4, 2048>(testURI, counter++);
    launch_testStencilHeat_perf<8, 1024>(testURI, counter++);
    launch_testStencilHeat_perf<16, 512>(testURI, counter++);
    launch_testStencilHeat_perf<32, 256>(testURI, counter++);
}

BOOST_AUTO_TEST_CASE(testStencilHeatZ_gridScaling)
{
    std::string testURI = suiteURI + ".device.stencil.dense.Z.2D.gridScaling";
    unsigned int counter = 0;
    constexpr unsigned int blockEdgeSize = 16;
    launch_testStencilHeatZ_perf<blockEdgeSize, 128>(testURI, counter++);
    launch_testStencilHeatZ_perf<blockEdgeSize, 256>(testURI, counter++);
    launch_testStencilHeatZ_perf<blockEdgeSize, 512>(testURI, counter++);
    launch_testStencilHeatZ_perf<blockEdgeSize, 1024>(testURI, counter++);
//    launch_testStencilHeatZ_perf<blockEdgeSize, 2048>(testURI, counter++);
}

BOOST_AUTO_TEST_CASE(testStencilHeatZ_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencil.dense.Z.2D.blockScaling";
    unsigned int counter = 0;
    // Note - blockEdgeSize == 2 doesn't work
    launch_testStencilHeatZ_perf<4, 2048>(testURI, counter++);
    launch_testStencilHeatZ_perf<8, 1024>(testURI, counter++);
    launch_testStencilHeatZ_perf<16, 512>(testURI, counter++);
    launch_testStencilHeatZ_perf<32, 256>(testURI, counter++);
}

BOOST_AUTO_TEST_CASE(testStencilHeat3D_gridScaling)
{
    std::string testURI = suiteURI + ".device.stencil.dense.N.3D.gridScaling";
    unsigned int counter = 0;
    constexpr unsigned int blockEdgeSize = 8;
    launch_testStencilHeat3D_perf<blockEdgeSize, 8>(testURI, counter++);
    launch_testStencilHeat3D_perf<blockEdgeSize, 16>(testURI, counter++);
    launch_testStencilHeat3D_perf<blockEdgeSize, 32>(testURI, counter++);
    launch_testStencilHeat3D_perf<blockEdgeSize, 64>(testURI, counter++);
//    launch_testStencilHeat3D_perf<blockEdgeSize, 128>(testURI, counter++);
}

BOOST_AUTO_TEST_CASE(testStencilHeat3D_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencil.dense.N.3D.blockScaling";
    unsigned int counter = 0;
    launch_testStencilHeat3D_perf<2, 128>(testURI, counter++);
    launch_testStencilHeat3D_perf<4, 64>(testURI, counter++);
    launch_testStencilHeat3D_perf<8, 32>(testURI, counter++);
//    launch_testStencilHeat3D_perf<16, 16>(testURI, counter++); // Too big, it doesn't work
}

//BOOST_AUTO_TEST_CASE(testStencilHeatZ3D)
//{
//    constexpr unsigned int dim = 3;
//    constexpr unsigned int blockEdgeSize = 4;
//
//    typedef aggregate<float,float> AggregateT;
//    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;
//
//    report_sparsegrid_funcs.graphs.put("performance.SparseGridGpu(1).stencil.test.name","StencilZ3D");
//
//    testStencilHeat3D_perf<SparseGridGpu_z<dim, AggregateT, blockEdgeSize, chunkSize>>(1);
//}

BOOST_AUTO_TEST_CASE(testStencilHeatSparse_gridScaling)
{
    std::string testURI = suiteURI + ".device.stencil.sparse.N.2D.gridScaling";
    unsigned int counter = 0;
    constexpr unsigned int blockEdgeSize = 16;
    launch_testStencilHeatSparse_perf<blockEdgeSize, 128>(testURI, counter++);
    launch_testStencilHeatSparse_perf<blockEdgeSize, 256>(testURI, counter++);
    launch_testStencilHeatSparse_perf<blockEdgeSize, 512>(testURI, counter++);
    launch_testStencilHeatSparse_perf<blockEdgeSize, 1024>(testURI, counter++);
//    launch_testStencilHeatSparse_perf<blockEdgeSize, 2048>(testURI, counter++);
}

BOOST_AUTO_TEST_CASE(testStencilHeatSparse_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencil.sparse.N.2D.blockScaling";
    unsigned int counter = 0;
    // Note - blockEdgeSize == 2 doesn't work
    launch_testStencilHeatSparse_perf<4, 1024>(testURI, counter++);
    launch_testStencilHeatSparse_perf<8, 512>(testURI, counter++);
    launch_testStencilHeatSparse_perf<16, 256>(testURI, counter++);
    launch_testStencilHeatSparse_perf<32, 128>(testURI, counter++);
}

BOOST_AUTO_TEST_CASE(testStencilHeat3DSparse_gridScaling)
{
    std::string testURI = suiteURI + ".device.stencil.sparse.N.3D.gridScaling";
    unsigned int counter = 0;
    constexpr unsigned int blockEdgeSize = 8;
    launch_testStencilHeat3DSparse_perf<blockEdgeSize, 8>(testURI, counter++);
    launch_testStencilHeat3DSparse_perf<blockEdgeSize, 16>(testURI, counter++);
    launch_testStencilHeat3DSparse_perf<blockEdgeSize, 32>(testURI, counter++);
    launch_testStencilHeat3DSparse_perf<blockEdgeSize, 64>(testURI, counter++);
}

BOOST_AUTO_TEST_CASE(testStencilHeat3DSparse_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencil.sparse.N.3D.blockScaling";
    unsigned int counter = 0;
    launch_testStencilHeat3DSparse_perf<2, 128>(testURI, counter++);
    launch_testStencilHeat3DSparse_perf<4, 64>(testURI, counter++);
    launch_testStencilHeat3DSparse_perf<8, 32>(testURI, counter++);
//    launch_testStencilHeat3DSparse_perf<16, 16>(testURI, counter++); // Too big, it doesn't work
}

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void testInsertStencil(std::string testURI, unsigned int i)
{
	auto testName = "Insert stencil";
	constexpr unsigned int dim = 2;
//	constexpr unsigned int blockEdgeSize = 8;
	constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;
	typedef aggregate<float,float> AggregateT;
	typedef HeatStencil<dim,0,1> StencilT;

	unsigned int iterations = 10;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilInsertN");

    report_sparsegrid_funcs.graphs.put(base + ".dim",2);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*blockEdgeSize);

	dim3 gridSize(gridEdgeSize, gridEdgeSize);
	dim3 blockSize(blockEdgeSize, blockEdgeSize);
	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	// Initialize the grid
	sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
	CUDA_LAUNCH_DIM3((insertConstantValue<0>),gridSize, blockSize,sparseGrid.toKernel(), 0);
	sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
	dim3 sourcePt(gridSize.x * blockEdgeSize / 2, gridSize.y * blockEdgeSize / 2, 0);
	insertOneValue<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), sourcePt, 100);
	sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!

	unsigned long long numElements = gridEdgeSize*blockEdgeSize*gridEdgeSize*blockEdgeSize;

	for (unsigned int iter=0; iter<5; ++iter)
	{
		sparseGrid.template applyStencils<StencilT>(STENCIL_MODE_INSERT, 0.1);
		sparseGrid.template flush<smax_<0>>(ctx, flush_type::FLUSH_ON_DEVICE);
	}

    openfpm::vector<double> gElemSMeasures;
    openfpm::vector<double> gFlopsSMeasures;

    for (unsigned int iter=0; iter<iterations; ++iter)
	{
		hipDeviceSynchronize();

		timer ts;
		ts.start();

		sparseGrid.template applyStencils<StencilT>(STENCIL_MODE_INSERT, 0.1);
		sparseGrid.template flush<smax_<0>>(ctx, flush_type::FLUSH_ON_DEVICE);

		hipDeviceSynchronize();

		ts.stop();

		float gElemS = numElements / (1e9 * ts.getwct());
		float gFlopsS = gElemS * StencilT::flops;

		gElemSMeasures.add(gElemS);
		gFlopsSMeasures.add(gFlopsS);
	}


	double elemMean=0, elemDeviation=0;
	standard_deviation(gElemSMeasures, elemMean, elemDeviation);
    report_sparsegrid_funcs.graphs.put(base + ".GElems.mean",elemMean);
    report_sparsegrid_funcs.graphs.put(base +".GElems.dev",elemDeviation);
    double flopsMean=0, flopsDeviation=0;
    standard_deviation(gFlopsSMeasures, flopsMean, flopsDeviation);
    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",flopsMean);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",flopsDeviation);

	std::cout << "Test: " << testName << "\n";
	std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << "\n";
	std::cout << "Grid: " << gridEdgeSize*blockEdgeSize << "x" << gridEdgeSize*blockEdgeSize << "\n";
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << "\n";
	std::cout << "Throughput:\n\t" << elemMean << " GElem/s dev: " << elemDeviation << " GElem/s" << std::endl
	            << "\t" << flopsMean << " GFlops/s dev: " << flopsDeviation << " GFlops/s" << std::endl;
}

BOOST_AUTO_TEST_CASE(testStencilHeatInsert_gridScaling)
{
    std::string testURI = suiteURI + ".device.stencilInsert.dense.N.2D.gridScaling";
    unsigned int counter = 0;
    testInsertStencil<8, 64>(testURI, counter++);
	testInsertStencil<8, 128>(testURI, counter++);
	testInsertStencil<8, 256>(testURI, counter++);
	testInsertStencil<8, 512>(testURI, counter++);
	testInsertStencil<8, 1024>(testURI, counter++);
}

BOOST_AUTO_TEST_CASE(testStencilHeatInsert_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencilInsert.dense.N.2D.blockScaling";
    unsigned int counter = 0;
    testInsertStencil<4, 1024>(testURI, counter++);
    testInsertStencil<8, 512>(testURI, counter++);
    testInsertStencil<16, 256>(testURI, counter++);
    testInsertStencil<32, 128>(testURI, counter++);
}

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void testInsertSingle(std::string testURI, unsigned int i)
{
	auto testName = "Insert single (one chunk per element)";
	constexpr unsigned int dim = 2;
//	constexpr unsigned int blockEdgeSize = 8;
	constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;
	typedef aggregate<float> AggregateT;

	unsigned int iterations = 10;
	bool prePopulateGrid = true;

//    std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").insertSingle");
    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","InsertSingle");

    report_sparsegrid_funcs.graphs.put(base + ".dim",dim);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*blockEdgeSize);

	dim3 gridSize(gridEdgeSize, gridEdgeSize);
	dim3 blockSize(blockEdgeSize, blockEdgeSize);
	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	if (prePopulateGrid)
	{
		// Pre-populate grid
		sparseGrid.setGPUInsertBuffer(gridSize, blockSize);
		insertValues2D<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), 0, 0);
		sparseGrid.template flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
		hipDeviceSynchronize();
		///
	}

	for (unsigned int iter=0; iter<5; ++iter)
	{
		auto offset = 0;
		sparseGrid.setGPUInsertBuffer(gridSize, blockSize);
		insertValues2D<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), offset, offset);
		sparseGrid.template flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
		hipDeviceSynchronize();
	}

	unsigned long long numElements = gridEdgeSize*blockEdgeSize*gridEdgeSize*blockEdgeSize;
	openfpm::vector<double> measures;

	for (unsigned int iter=0; iter<iterations; ++iter)
	{
		auto offset = 0;

		hipDeviceSynchronize();

		timer ts;
		ts.start();

		sparseGrid.setGPUInsertBuffer(gridSize, blockSize);
		insertValues2D<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), offset, offset);
		sparseGrid.template flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
		hipDeviceSynchronize();

		ts.stop();

		float mElemS = numElements / (1e6 * ts.getwct());
		measures.add(mElemS);
	}

	double mean = 0;
	double deviation = 0;
	standard_deviation(measures,mean,deviation);

    report_sparsegrid_funcs.graphs.put(base + ".Minsert.mean",mean);
    report_sparsegrid_funcs.graphs.put(base +".Minsert.dev",deviation);

	// All times above are in ms

	std::cout << "Test: " << testName << "\n";
    std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << "\n";
    std::cout << "Grid: " << gridEdgeSize*blockEdgeSize << "x" << gridEdgeSize*blockEdgeSize << "\n";
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << "\n";
	std::cout << "Throughput:\n\t" << mean << "M/s" << "\n";
}

BOOST_AUTO_TEST_CASE(testInsert_gridScaling)
{
    std::string testURI = suiteURI + ".device.insert.dense.single.2D.gridScaling";
    unsigned int counter = 0;
    testInsertSingle<8, 64>(testURI, counter++);
    testInsertSingle<8, 128>(testURI, counter++);
    testInsertSingle<8, 256>(testURI, counter++);
//    testInsertSingle<8, 512>(testURI, counter++);
//    testInsertSingle<8, 1024>(testURI, counter++);
}

BOOST_AUTO_TEST_CASE(testInsert_blockScaling)
{
    std::string testURI = suiteURI + ".device.insert.dense.single.2D.blockScaling";
    unsigned int counter = 0;
    testInsertSingle<2, 1024>(testURI, counter++);
    testInsertSingle<4, 512>(testURI, counter++);
    testInsertSingle<8, 256>(testURI, counter++);
//    testInsertSingle<16, 128>(testURI, counter++);
//    testInsertSingle<32, 64>(testURI, counter++);
}

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void test_insert_block(std::string testURI, unsigned int i)
{
	auto testName = "Insert (one chunk per block)";
	constexpr unsigned int dim = 2;
//	constexpr unsigned int blockEdgeSize = 8;
	constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;
	typedef aggregate<float> AggregateT;

//	std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").insert");
    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","InsertBlock");

	report_sparsegrid_funcs.graphs.put(base + ".name","Block insert");
    report_sparsegrid_funcs.graphs.put(base + ".dim",dim);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*blockEdgeSize);

	unsigned int iterations = 10;

	openfpm::vector<double> measures;

	unsigned long long numElements = gridEdgeSize*blockEdgeSize*gridEdgeSize*blockEdgeSize;
	dim3 gridSize(gridEdgeSize, gridEdgeSize);
	dim3 blockSize(blockEdgeSize, blockEdgeSize);
	dim3 blockSizeBlockedInsert(1, 1);
	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	// Warmup
	for (unsigned int iter=0; iter<5; ++iter)
	{
		auto offset = 0;
		sparseGrid.setGPUInsertBuffer(gridSize, blockSizeBlockedInsert);
		insertValues2DBlocked<0, 1, blockEdgeSize> << < gridSize, blockSize >> >
				(sparseGrid.toKernel(), offset, offset);
		sparseGrid.template flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	}


	hipDeviceSynchronize();


	for (unsigned int iter=0; iter<iterations; ++iter)
	{
		auto offset = 0;

		hipDeviceSynchronize();

		timer ts;
		ts.start();

		sparseGrid.setGPUInsertBuffer(gridSize, blockSizeBlockedInsert);
		insertValues2DBlocked<0, 1, blockEdgeSize> << < gridSize, blockSize >> >
				(sparseGrid.toKernel(), offset, offset);
		sparseGrid.template flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

		hipDeviceSynchronize();

		ts.stop();

		float mElemS = numElements / (1e6 * ts.getwct());
		measures.add(mElemS);
	}

	double mean = 0;
	double deviation = 0;
	standard_deviation(measures,mean,deviation);

    report_sparsegrid_funcs.graphs.put(base + ".Minsert.mean",mean);
    report_sparsegrid_funcs.graphs.put(base +".Minsert.dev",deviation);

	// All times above are in ms

	std::cout << "Test: " << testName << "\n";
    std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << "\n";
    std::cout << "Grid: " << gridEdgeSize*blockEdgeSize << "x" << gridEdgeSize*blockEdgeSize << "\n";
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << "\n";
	std::cout << "\tInsert: " << mean << " dev: " << deviation << " s" << std::endl;
	std::cout << "Throughput:\n\t" << mean << " MElem/s\n";
}

BOOST_AUTO_TEST_CASE(testInsertBlocked_gridScaling)
{
    std::string testURI = suiteURI + ".device.insert.dense.block.2D.gridScaling";
    unsigned int counter = 0;
    test_insert_block<8,64>(testURI, counter++);
    test_insert_block<8,128>(testURI, counter++);
    test_insert_block<8,256>(testURI, counter++);
    test_insert_block<8,512>(testURI, counter++);
    test_insert_block<8,1024>(testURI, counter++);
//    test_insert_block<8,2048>(testURI, counter++); // Out of memory
}

BOOST_AUTO_TEST_CASE(testInsertBlocked_blockScaling)
{
    std::string testURI = suiteURI + ".device.insert.dense.block.2D.blockScaling";
    unsigned int counter = 0;
    test_insert_block<2,2048>(testURI, counter++);
    test_insert_block<4,1024>(testURI, counter++);
    test_insert_block<8,512>(testURI, counter++);
    test_insert_block<16,256>(testURI, counter++);
    test_insert_block<32,128>(testURI, counter++);
}

BOOST_AUTO_TEST_CASE(write_teport)
{
    write_test_report();
}

BOOST_AUTO_TEST_SUITE_END()

BOOST_AUTO_TEST_SUITE_END()
