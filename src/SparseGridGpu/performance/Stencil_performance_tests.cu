#include "hip/hip_runtime.h"
//
// Created by tommaso on 4/07/19.
//

#define BOOST_TEST_DYN_LINK
#define OPENFPM_DATA_ENABLE_IO_MODULE
#define DISABLE_MPI_WRITTERS

#include <boost/test/unit_test.hpp>
#include "SparseGridGpu/SparseGridGpu.hpp"
#include "cuda_macro.h"
#include <boost/property_tree/ptree.hpp>
#include <boost/property_tree/xml_parser.hpp>
#include "util/stat/common_statistics.hpp"
#include "Plot/GoogleChart.hpp"
#include "util/performance/performance_util.hpp"
#include "SparseGridGpu/tests/utils/SparseGridGpu_testKernels.cuh"
#include <set>
#include "performancePlots.hpp"

extern char * test_dir;

// Property tree

report_sparse_grid_tests report_sparsegrid_funcs;
std::string suiteURI = "performance.SparseGridGpu";
std::set<std::string> testSet;

struct Fixture
{
    Fixture()
    {
        BOOST_TEST_MESSAGE( "Setup fixture" );
    }

    ~Fixture()
    {
        BOOST_TEST_MESSAGE( "Teardown fixture" );
        write_test_report(report_sparsegrid_funcs, testSet);
    }
};


template<unsigned int p, typename SparseGridType>
__global__ void insertValues2D(SparseGridType sparseGrid, const int offsetX=0, const int offsetY=0)
{
    sparseGrid.init();

    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    int x = bIdX * bDimX + tIdX + offsetX;
    int y = bIdY * bDimY + tIdY + offsetY;
    grid_key_dx<SparseGridType::d, int> coord({x, y});

    sparseGrid.template insert<p>(coord) = x*x*y*y; // some function...

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
}

template<unsigned int p, unsigned int chunksPerBlock, unsigned int blockEdgeSize, typename SparseGridType>
__global__ void insertValues2DBlocked(SparseGridType sparseGrid, const int sOffsetX=0, const int sOffsetY=0)
{
    constexpr unsigned int pMask = SparseGridType::pMask;
    typedef BlockTypeOf<typename SparseGridType::AggregateType, p> BlockT;
    typedef BlockTypeOf<typename SparseGridType::AggregateType, pMask> MaskBlockT;

    sparseGrid.init();

    __shared__ BlockT *blocks[chunksPerBlock];
    __shared__ MaskBlockT *masks[chunksPerBlock];

    int posX = blockIdx.x * blockDim.x + threadIdx.x + sOffsetX;
    int posY = blockIdx.y * blockDim.y + threadIdx.y + sOffsetY;
    const unsigned int offsetX = posX % blockEdgeSize;
    const unsigned int offsetY = posY % blockEdgeSize;

    const unsigned int blockDimX = blockDim.x / blockEdgeSize;
    const unsigned int blockOffsetX = threadIdx.x / blockEdgeSize;
    const unsigned int blockOffsetY = threadIdx.y / blockEdgeSize;

    const unsigned int dataBlockNum = blockOffsetY*blockDimX + blockOffsetX;
    const unsigned int offset = offsetY * blockEdgeSize + offsetX;

//    if (offset == 0) // Just one thread per data block
//    {
        grid_key_dx<SparseGridType::d, int> blockCoord({posX / blockEdgeSize, posY / blockEdgeSize});
        auto encap = sparseGrid.insertBlockNew(sparseGrid.getBlockLinId(blockCoord));
        blocks[dataBlockNum] = &(encap.template get<p>());
        masks[dataBlockNum] = &(encap.template get<pMask>());
//    }

    __syncthreads();

    blocks[dataBlockNum]->block[offset] = posX*posX * posY*posY;
    BlockMapGpu_ker<>::setExist(masks[dataBlockNum]->block[offset]);

    __syncthreads();

    sparseGrid.flush_block_insert();
}



template<unsigned int p, unsigned int chunksPerBlock=1, typename SparseGridType, typename ScalarT>
__global__ void insertConstantValue(SparseGridType sparseGrid, ScalarT value)
{
    constexpr unsigned int pMask = SparseGridType::pMask;
    typedef BlockTypeOf<typename SparseGridType::AggregateType, p> BlockT;
    typedef BlockTypeOf<typename SparseGridType::AggregateType, pMask> MaskBlockT;

    sparseGrid.init();

    __shared__ BlockT *blocks[chunksPerBlock];
    __shared__ MaskBlockT *masks[chunksPerBlock];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    auto pos = sparseGrid.getLinId(coord);
    unsigned int dataBlockId = pos / BlockT::size;
    unsigned int offset = pos % BlockT::size;
    unsigned int dataBlockNum = dataBlockId % chunksPerBlock;

//    if (offset == 0) // Just one thread per data block
//    {
        auto encap = sparseGrid.insertBlockNew(dataBlockId);
        blocks[dataBlockNum] = &(encap.template get<p>());
        masks[dataBlockNum] = &(encap.template get<pMask>());
//    }

    __syncthreads();
    blocks[dataBlockNum]->block[offset] = value;
    BlockMapGpu_ker<>::setExist(masks[dataBlockNum]->block[offset]);

    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    x++;
    y++;
    z++;
}


template<unsigned int p, typename SparseGridType, typename ValueT>
__global__ void insertOneValue(SparseGridType sparseGrid, dim3 pt, ValueT value)
{
    sparseGrid.init();

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    dim3 thCoord(x, y, z);
    if (thCoord.x == pt.x && thCoord.y == pt.y && thCoord.z == pt.z)
    {
        grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});
        sparseGrid.template insert<p>(coord) = value;
    }
    __syncthreads();

    sparseGrid.flush_block_insert();

    // Compiler avoid warning
    y++;
    z++;
}

template<unsigned int p, typename SparseGridType, typename VectorOutType>
__global__ void copyBlocksToOutput(SparseGridType sparseGrid, VectorOutType output)
{
    const auto bDimX = blockDim.x;
    const auto bDimY = blockDim.y;
    const auto bDimZ = blockDim.z;
    const auto bIdX = blockIdx.x;
    const auto bIdY = blockIdx.y;
    const auto bIdZ = blockIdx.z;
    const auto tIdX = threadIdx.x;
    const auto tIdY = threadIdx.y;
    const auto tIdZ = threadIdx.z;
    int x = bIdX * bDimX + tIdX;
    int y = bIdY * bDimY + tIdY;
    int z = bIdZ * bDimZ + tIdZ;
    grid_key_dx<SparseGridType::d, size_t> coord({x, y, z});

    size_t pos = sparseGrid.getLinId(coord);

    auto value = sparseGrid.template get<p>(coord);

    output.template get<p>(pos) = value;

    // Compiler avoid warning
    x++;
    y++;
    z++;
}

template<unsigned int dim, unsigned int p_src, unsigned int p_dst>
struct HeatStencil
{
	typedef NNStar stencil_type;

    // This is an example of a laplacian smoothing stencil to apply using the apply stencil facility of SparseGridGpu

    static constexpr unsigned int flops = 3 + 2*dim;

    static constexpr unsigned int supportRadius = 1;

    /*! \brief Stencil function
     *
     * \param sparseGrid This is the sparse grid data-structure
     * \param dataBlockId The id of the block
     * \param offset index in local coordinate of the point where we are working
	 * \param dataBlockLoad dataBlock from where we read
	 * \param dataBlockStore dataBlock from where we write
	 * \param isActive the point is active if exist and is not padding
	 * \param dt delta t
     *
     *
     */
    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __device__ void stencil(
            SparseGridT & sparseGrid,
            const unsigned int dataBlockId,
            const openfpm::sparse_index<unsigned int> dataBlockIdPos,
            const unsigned int offset,
            const grid_key_dx<dim, int> & pointCoord,
            const DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore,
            bool isActive,
            float dt)
    {
        typedef typename SparseGridT::AggregateBlockType AggregateT;
        typedef ScalarTypeOf<AggregateT, p_src> ScalarT;

        constexpr unsigned int enlargedBlockSize = IntPow<
                SparseGridT::getBlockEdgeSize() + 2 * supportRadius, dim>::value;

        __shared__ ScalarT enlargedBlock[enlargedBlockSize];

        sparseGrid.loadGhostBlock<p_src>(dataBlockLoad, dataBlockIdPos, enlargedBlock);

        __syncthreads();

        if (isActive)
        {
            const auto coord = sparseGrid.getCoordInEnlargedBlock(offset);
            const auto linId = sparseGrid.getLinIdInEnlargedBlock(offset);
            ScalarT cur = enlargedBlock[linId];
            ScalarT laplacian = -2.0 * dim * cur; // The central part of the stencil

            for (int d = 0; d < dim; ++d)
            {
                auto nPlusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, 1);
                auto nMinusId = sparseGrid.getNeighbourLinIdInEnlargedBlock(coord, d, -1);
                ScalarT neighbourPlus = enlargedBlock[nPlusId];
                ScalarT neighbourMinus = enlargedBlock[nMinusId];
                laplacian += neighbourMinus + neighbourPlus;
            }
            enlargedBlock[linId] = cur + dt * laplacian;
        }
    }

    /*! \brief Stencil Host function
    *
    * \param sparseGrid This is the sparse grid data-structure
    * \param dataBlockId The id of the block
    * \param offset index in local coordinate of the point where we are working
    * \param dataBlockLoad dataBlock from where we read
    * \param dataBlockStore dataBlock from where we write
    * \param isActive the point is active if exist and is not padding
    * \param dt delta t
    *
    *
    */
    template<typename SparseGridT, typename DataBlockWrapperT>
    static inline __host__ void stencilHost(
            SparseGridT & sparseGrid,
            const unsigned int dataBlockId,
            const openfpm::sparse_index<unsigned int> dataBlockIdPos,
            const unsigned int offset,
            const grid_key_dx<dim, int> & pointCoord,
            const DataBlockWrapperT & dataBlockLoad,
            DataBlockWrapperT & dataBlockStore,
            bool isActive,
            float dt)
    {
        constexpr unsigned int blockEdgeSize = SparseGridT::getBlockEdgeSize();

        if (isActive)
        {
            auto cur = dataBlockLoad.template get<p_src>()[offset];
            auto laplacian = -2.0 * dim * cur; // The central part of the stencil

            auto neighbourCoord = pointCoord;
            auto counter = offset;
            unsigned int dimStride = 1;
            for (int d = 0; d < dim; ++d)
            {
                const auto localOffset = counter % blockEdgeSize;

                if (localOffset == 0) // This means we are at the lower boundary for this dimension
                {
                    neighbourCoord.set_d(d, neighbourCoord.get(d) - 1);
                    laplacian += sparseGrid.template get<p_src>(neighbourCoord);
                    neighbourCoord.set_d(d, neighbourCoord.get(d) + 1);
                }
                else
                {
                    laplacian += dataBlockLoad.template get<p_src>()[offset - dimStride];
                }
                if (localOffset == blockEdgeSize - 1) // This means we are at the lower boundary for this dimension
                {
                neighbourCoord.set_d(d, neighbourCoord.get(d) + 1);
                laplacian += sparseGrid.template get<p_src>(neighbourCoord);
                neighbourCoord.set_d(d, neighbourCoord.get(d) - 1);
                }
                else
                {
                    laplacian += dataBlockLoad.template get<p_src>()[offset + dimStride];
                }
                //
                counter /= blockEdgeSize;
                dimStride *= blockEdgeSize;
            }
            dataBlockStore.template get<p_dst>()[offset] = cur + dt * laplacian;
        }
    }

    template <typename SparseGridT, typename CtxT>
    static inline void __host__ flush(SparseGridT & sparseGrid, CtxT & ctx)
    {
        sparseGrid.template flush <sRight_<0>> (ctx, flush_type::FLUSH_ON_DEVICE);
    }
};

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
void testStencilHeat_perf(unsigned int i, std::string base)
{
    auto testName = "In-place stencil";
    typedef HeatStencil<SparseGridZ::dims,0,1> StencilT;

    report_sparsegrid_funcs.graphs.put(base + ".dim",2);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*SparseGridZ::blockEdgeSize_);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*SparseGridZ::blockEdgeSize_);

    unsigned int iterations = 100;

    openfpm::vector<double> measures_gf;
    openfpm::vector<double> measures_tm;

	dim3 gridSize(gridEdgeSize, gridEdgeSize);
	dim3 blockSize(SparseGridZ::blockEdgeSize_,SparseGridZ::blockEdgeSize_);
	typename SparseGridZ::grid_info blockGeometry(gridSize);
	SparseGridZ sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

    unsigned long long numElements = gridEdgeSize*SparseGridZ::blockEdgeSize_*gridEdgeSize*SparseGridZ::blockEdgeSize_;

	// Initialize the grid
	sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
	CUDA_LAUNCH_DIM3((insertConstantValue<0>),gridSize, blockSize,sparseGrid.toKernel(), 0);
	sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
	dim3 sourcePt(gridSize.x * SparseGridZ::blockEdgeSize_ / 2, gridSize.y * SparseGridZ::blockEdgeSize_ / 2, 0);
	insertOneValue<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), sourcePt, 100);
	sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!

	for (unsigned int iter=0; iter<iterations; ++iter)
	{
		hipDeviceSynchronize();

		timer ts;
		ts.start();

		sparseGrid.template applyStencils<StencilT>(STENCIL_MODE_INPLACE, 0.1);

		hipDeviceSynchronize();
		ts.stop();

		measures_tm.add(ts.getwct());

	    float gElemS = numElements / (1e9 * ts.getwct());
	    float gFlopsS = gElemS * StencilT::flops;

		measures_gf.add(gFlopsS);
	}

	double mean_tm = 0;
	double deviation_tm = 0;
	standard_deviation(measures_tm,mean_tm,deviation_tm);

	double mean_gf = 0;
	double deviation_gf = 0;
	standard_deviation(measures_gf,mean_gf,deviation_gf);

    // All times above are in ms

    float gElemS = numElements / (1e9 * mean_tm);
    float gFlopsS = gElemS * StencilT::flops;
    std::cout << "Test: " << testName << std::endl;
    std::cout << "Block: " << SparseGridZ::blockEdgeSize_ << "x" << SparseGridZ::blockEdgeSize_ << std::endl;
    std::cout << "Grid: " << gridEdgeSize*SparseGridZ::blockEdgeSize_ << "x" << gridEdgeSize*SparseGridZ::blockEdgeSize_ << std::endl;
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;

    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeat_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilN");

    testStencilHeat_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize>>(i, base);
    hipDeviceSynchronize();
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeatZ_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilZ");

    testStencilHeat_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu_z<dim, AggregateT, blockEdgeSize, chunkSize>>(i, base);
    hipDeviceSynchronize();
}

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
void testStencilHeat3D_perf(unsigned int i, std::string base)
{
    auto testName = "In-place 3D stencil";
//    unsigned int gridEdgeSize = 128;
//    unsigned int gridEdgeSize = 64;
    typedef HeatStencil<SparseGridZ::dims,0,1> StencilT;

    report_sparsegrid_funcs.graphs.put(base + ".dim",3);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*SparseGridZ::blockEdgeSize_);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*SparseGridZ::blockEdgeSize_);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.z",gridEdgeSize*SparseGridZ::blockEdgeSize_);

    unsigned int iterations = 100;

    openfpm::vector<double> measures_gf;
    openfpm::vector<double> measures_tm;

    dim3 gridSize(gridEdgeSize, gridEdgeSize, gridEdgeSize);
    dim3 blockSize(SparseGridZ::blockEdgeSize_, SparseGridZ::blockEdgeSize_, SparseGridZ::blockEdgeSize_);

    typename SparseGridZ::grid_info blockGeometry(gridSize);
    SparseGridZ sparseGrid(blockGeometry);
    mgpu::ofp_context_t ctx;
    sparseGrid.template setBackgroundValue<0>(0);

    unsigned long long numElements = gridEdgeSize*SparseGridZ::blockEdgeSize_
            *gridEdgeSize*SparseGridZ::blockEdgeSize_
            *gridEdgeSize*SparseGridZ::blockEdgeSize_;

    // Initialize the grid
    sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
    CUDA_LAUNCH_DIM3((insertConstantValue<0>),gridSize, blockSize,sparseGrid.toKernel(), 0);
    sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
    dim3 sourcePt(gridSize.x * SparseGridZ::blockEdgeSize_ / 2,
            gridSize.y * SparseGridZ::blockEdgeSize_ / 2,
            gridSize.z * SparseGridZ::blockEdgeSize_ / 2);
    insertOneValue<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), sourcePt, 100);
    sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!

    for (unsigned int iter=0; iter<iterations; ++iter)
    {
        hipDeviceSynchronize();

        timer ts;
        ts.start();

        sparseGrid.template applyStencils<StencilT>(STENCIL_MODE_INPLACE, 0.1);

        hipDeviceSynchronize();
        ts.stop();

        measures_tm.add(ts.getwct());

        float gElemS = numElements / (1e9 * ts.getwct());
        float gFlopsS = gElemS * StencilT::flops;

        measures_gf.add(gFlopsS);
    }

    double mean_tm = 0;
    double deviation_tm = 0;
    standard_deviation(measures_tm,mean_tm,deviation_tm);

    double mean_gf = 0;
    double deviation_gf = 0;
    standard_deviation(measures_gf,mean_gf,deviation_gf);

    // All times above are in ms

    float gElemS = numElements / (1e9 * mean_tm);
    float gFlopsS = gElemS * StencilT::flops;
    std::cout << "Test: " << testName << std::endl;
    std::cout << "Block: " << SparseGridZ::blockEdgeSize_
              << "x" << SparseGridZ::blockEdgeSize_
              << "x" << SparseGridZ::blockEdgeSize_
              << std::endl;
    std::cout << "Grid: " << gridEdgeSize*SparseGridZ::blockEdgeSize_
        << "x" << gridEdgeSize*SparseGridZ::blockEdgeSize_
        << "x" << gridEdgeSize*SparseGridZ::blockEdgeSize_
        << std::endl;
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;

    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeat3D_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 3;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilN3D");

    testStencilHeat3D_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize>>(i, base);
    hipDeviceSynchronize();
}

//template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
//void testStencilHeatSparse_perf(unsigned int i, std::string base)
//{
//    auto testName = "In-place sparse stencil";
////    unsigned int gridEdgeSize = 128;
//    constexpr unsigned int dim = SparseGridZ::dims;
////    const unsigned int blockEdgeSize = SparseGridZ::blockEdgeSize_;
//
//    typedef HeatStencil<dim, 0, 1> Stencil01T;
//    typedef HeatStencil<dim, 1, 0> Stencil10T;
//
////    std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").stencil");
//
//    report_sparsegrid_funcs.graphs.put(base + ".dim",2);
//    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
//    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*blockEdgeSize);
//    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*blockEdgeSize);
//
//    unsigned int iterations = 100;
//
//    openfpm::vector<double> measures_gf;
//    openfpm::vector<double> measures_tm;
//
//    dim3 gridSize(gridEdgeSize, gridEdgeSize);
//    dim3 blockSize(blockEdgeSize,blockEdgeSize);
//    size_t sz[2] = {1000000,1000000};
//    typename SparseGridZ::grid_info blockGeometry(sz);
//    SparseGridZ sparseGrid(blockGeometry);
//    mgpu::ofp_context_t ctx;
//    sparseGrid.template setBackgroundValue<0>(0);
//
//    ///// Insert sparse content, a set of 3 hollow spheres /////
//    constexpr unsigned int rBig = gridEdgeSize * blockEdgeSize / 2;
//    constexpr unsigned int rSmall = rBig/2;
//    constexpr unsigned int rBig2 = rBig;
//    constexpr unsigned int rSmall2 = rBig2 - (rBig2/16);
//    constexpr unsigned int rBig3 = rBig/8;
//    constexpr unsigned int rSmall3 = rBig3 - (rBig3/10);
//    // Sphere 1
//    grid_key_dx<2,int> start1({500000,500000});
//    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
//    CUDA_LAUNCH_DIM3((insertSphere<0>),
//                     gridSize, dim3(blockEdgeSize*blockEdgeSize,1,1),
//                     sparseGrid.toKernel(), start1, rBig, rSmall, 1);
//    hipDeviceSynchronize();
//    sparseGrid.template flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
//    hipDeviceSynchronize();
//
//    // Sphere 2
//    grid_key_dx<2,int> start2({500000+rBig,500000+rBig});
//    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
//    CUDA_LAUNCH_DIM3((insertSphere<0>),
//                     gridSize, dim3(blockEdgeSize*blockEdgeSize,1,1),
//                     sparseGrid.toKernel(), start2, rBig2, rSmall2, 1);
//    hipDeviceSynchronize();
//    sparseGrid.template flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
//    hipDeviceSynchronize();
//
//    // Sphere 3
//    grid_key_dx<2,int> start3({500000+rBig,500000});
//    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
//    CUDA_LAUNCH_DIM3((insertSphere<0>),
//                     gridSize, dim3(blockEdgeSize*blockEdgeSize,1,1),
//                     sparseGrid.toKernel(), start3, rBig3, rSmall3, 1);
//    hipDeviceSynchronize();
//    sparseGrid.template flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
//    hipDeviceSynchronize();
//    ///// /////
//
//    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
//    sparseGrid.tagBoundaries();
//
//    sparseGrid.template deviceToHost<0>(); // NECESSARY as count takes place on Host!
//    auto existingElements = sparseGrid.countExistingElements();
//    auto boundaryElements = sparseGrid.countBoundaryElements();
//    unsigned long long numElements = existingElements - boundaryElements;
//
//    // Now apply some boundary conditions
//    sparseGrid.template applyBoundaryStencils<BoundaryStencilSetXRescaled<dim,0,0>>(
//            500000, 500000+(2*rBig),
//            0.0, 10.0);
//    hipDeviceSynchronize();
//
//    for (unsigned int iter=0; iter<iterations; ++iter)
//    {
//        hipDeviceSynchronize();
//
//        timer ts;
//        ts.start();
//
//        sparseGrid.template applyStencils<Stencil01T>(STENCIL_MODE_INPLACE, 0.1);
//        hipDeviceSynchronize();
//        sparseGrid.template applyStencils<Stencil10T>(STENCIL_MODE_INPLACE, 0.1);
//        hipDeviceSynchronize();
//
//        ts.stop();
//
//        measures_tm.add(ts.getwct());
//
//        float gElemS = numElements / (1e9 * ts.getwct());
//        float gFlopsS = gElemS * Stencil01T::flops;
//
//        measures_gf.add(gFlopsS);
//    }
//
//    double mean_tm = 0;
//    double deviation_tm = 0;
//    standard_deviation(measures_tm,mean_tm,deviation_tm);
//
//    double mean_gf = 0;
//    double deviation_gf = 0;
//    standard_deviation(measures_gf,mean_gf,deviation_gf);
//
//    // All times above are in ms
//
//    float gElemS = numElements / (1e9 * mean_tm);
//    float gFlopsS = gElemS * Stencil01T::flops;
//    std::cout << "Test: " << testName << std::endl;
//    std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << std::endl;
//    std::cout << "Grid: " << gridEdgeSize*blockEdgeSize << "x" << gridEdgeSize*blockEdgeSize << std::endl;
//    double dataOccupancyMean, dataOccupancyDev;
//    sparseGrid.deviceToHost();
//    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
//    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
//    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
//    std::cout << "Iterations: " << iterations << std::endl;
//    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
//    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;
//
//    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
//    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
//    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
//    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);
//
////    // DEBUG
////    sparseGrid.template deviceToHost<0,1>();
////    sparseGrid.write("SparseGridGPU_testStencilHeatSparse_perf_DEBUG.vtk");
//}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
void testStencilHeatSparse_perf(unsigned int i, std::string base, float fillMultiplier=1, float voidMultiplier=1)
{
    auto testName = "In-place sparse stencil";
//    unsigned int gridEdgeSize = 128;
    constexpr unsigned int dim = SparseGridZ::dims;
//    const unsigned int blockEdgeSize = SparseGridZ::blockEdgeSize_;

    typedef HeatStencil<dim, 0, 1> Stencil01T;
    typedef HeatStencil<dim, 1, 0> Stencil10T;

//    std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").stencil");

    report_sparsegrid_funcs.graphs.put(base + ".dim",2);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*blockEdgeSize);

    unsigned int iterations = 100;

    openfpm::vector<double> measures_gf;
    openfpm::vector<double> measures_tm;

    dim3 gridSize(gridEdgeSize, gridEdgeSize);
    dim3 blockSize(blockEdgeSize,blockEdgeSize);
    unsigned int spatialEdgeSize = 1000000;
    size_t sz[2] = {spatialEdgeSize, spatialEdgeSize};
    typename SparseGridZ::grid_info blockGeometry(sz);
    SparseGridZ sparseGrid(blockGeometry);
    mgpu::ofp_context_t ctx;
    sparseGrid.template setBackgroundValue<0>(0);

    ///// Insert sparse content, a set of concentric spheres /////
    float allMultiplier = fillMultiplier + voidMultiplier;
    const unsigned int numSpheres = gridEdgeSize / (2*allMultiplier);
//    const unsigned int numSpheres = 1;
    unsigned int centerPoint = spatialEdgeSize / 2;

    for (int i = 1; i <= numSpheres; ++i)
    {
        unsigned int rBig = allMultiplier*i * blockEdgeSize;
        unsigned int rSmall = (allMultiplier*i - fillMultiplier) * blockEdgeSize;
        // Sphere i-th
        grid_key_dx<dim, int> start1({centerPoint, centerPoint});
        sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
        CUDA_LAUNCH_DIM3((insertSphere<0>),
                         gridSize, dim3(blockEdgeSize * blockEdgeSize, 1, 1),
                         sparseGrid.toKernel(), start1, rBig, rSmall, 5);
        hipDeviceSynchronize();
        sparseGrid.template flush<smax_<0 >>(ctx, flush_type::FLUSH_ON_DEVICE);
        hipDeviceSynchronize();
    }
    ///// /////

    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
    sparseGrid.tagBoundaries();

    sparseGrid.template deviceToHost<0>(); // NECESSARY as count takes place on Host!
    auto existingElements = sparseGrid.countExistingElements();
    auto boundaryElements = sparseGrid.countBoundaryElements();
    unsigned long long numElements = existingElements - boundaryElements;

    // Now apply some boundary conditions
    sparseGrid.template applyBoundaryStencils<BoundaryStencilSetXRescaled<dim,0,0>>(
            centerPoint, centerPoint + 2*blockEdgeSize*gridEdgeSize,
            0.0, 10.0);
    hipDeviceSynchronize();

    for (unsigned int iter=0; iter<iterations; ++iter)
    {
        hipDeviceSynchronize();

        timer ts;
        ts.start();

        sparseGrid.template applyStencils<Stencil01T>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();
        sparseGrid.template applyStencils<Stencil10T>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();

        ts.stop();

        measures_tm.add(ts.getwct());

        float gElemS = numElements / (1e9 * ts.getwct());
        float gFlopsS = gElemS * Stencil01T::flops;

        measures_gf.add(gFlopsS);
    }

    double mean_tm = 0;
    double deviation_tm = 0;
    standard_deviation(measures_tm,mean_tm,deviation_tm);

    double mean_gf = 0;
    double deviation_gf = 0;
    standard_deviation(measures_gf,mean_gf,deviation_gf);

    // All times above are in ms

    float gElemS = numElements / (1e9 * mean_tm);
    float gFlopsS = gElemS * Stencil01T::flops;
    std::cout << "Test: " << testName << std::endl;
    std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << std::endl;
    std::cout << "Grid: " << gridEdgeSize*blockEdgeSize << "x" << gridEdgeSize*blockEdgeSize << std::endl;
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;

    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);

//    // DEBUG
//    sparseGrid.template deviceToHost<0,1>();
//    sparseGrid.write("SparseGridGPU_testStencilHeatSparse_perf_DEBUG.vtk");
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeatSparse_perf(std::string testURI, unsigned int i,
        float fillMultiplier=1, float voidMultiplier=1, std::string occupancyStr="05")
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilNSparse"+occupancyStr);

    testStencilHeatSparse_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize, long int>>(i, base,
                    fillMultiplier, voidMultiplier);
    hipDeviceSynchronize();
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeatSparseZ_perf(std::string testURI, unsigned int i,
                                       float fillMultiplier=1, float voidMultiplier=1, std::string occupancyStr="05")
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilNSparse"+occupancyStr);

    testStencilHeatSparse_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu_z<dim, AggregateT, blockEdgeSize, chunkSize, long int>>(i, base,
                                                                                fillMultiplier, voidMultiplier);
    hipDeviceSynchronize();
}

//template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
//void testStencilHeat3DSparse_perf(unsigned int i, std::string base)
//{
//    auto testName = "In-place 3D sparse stencil";
////    unsigned int gridEdgeSize = 32;
//    constexpr unsigned int dim = SparseGridZ::dims;
////    const unsigned int blockEdgeSize = SparseGridZ::blockEdgeSize_;
//
//    typedef HeatStencil<dim, 0, 1> Stencil01T;
//    typedef HeatStencil<dim, 1, 0> Stencil10T;
//
////    std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").stencil");
//
//    report_sparsegrid_funcs.graphs.put(base + ".dim",dim);
//    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
//    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x", gridEdgeSize * blockEdgeSize);
//    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y", gridEdgeSize * blockEdgeSize);
//    report_sparsegrid_funcs.graphs.put(base + ".gridSize.z", gridEdgeSize * blockEdgeSize);
//
//    unsigned int iterations = 100;
//
//    openfpm::vector<double> measures_gf;
//    openfpm::vector<double> measures_tm;
//
//    dim3 gridSize(gridEdgeSize, gridEdgeSize, gridEdgeSize);
//    dim3 blockSize(blockEdgeSize, blockEdgeSize, blockEdgeSize);
//    unsigned int spatialEdgeSize = 10000;
//    size_t sz[3] = {spatialEdgeSize, spatialEdgeSize, spatialEdgeSize};
//    typename SparseGridZ::grid_info blockGeometry(sz);
//    SparseGridZ sparseGrid(blockGeometry);
//    mgpu::ofp_context_t ctx;
//    sparseGrid.template setBackgroundValue<0>(0);
//
//    ///// Insert sparse content, a set of 3 hollow spheres /////
//    constexpr unsigned int rBig = gridEdgeSize * blockEdgeSize / 2;
//    constexpr unsigned int rSmall = rBig/2;
//    constexpr unsigned int rBig2 = rBig;
//    constexpr unsigned int rSmall2 = rBig2 - (rBig2/3);
//    constexpr unsigned int rBig3 = rBig/4;
//    constexpr unsigned int rSmall3 = rBig3 - (rBig3/4);
//    // Sphere 1
//    unsigned int centerPoint = spatialEdgeSize/2;
//    grid_key_dx<dim,int> start1({centerPoint, centerPoint, centerPoint});
//    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
//    CUDA_LAUNCH_DIM3((insertSphere3D<0>),
//                     gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
//                     sparseGrid.toKernel(), start1, rBig, rSmall, 1);
//    hipDeviceSynchronize();
//    sparseGrid.template flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
//    hipDeviceSynchronize();
//
//    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
//    sparseGrid.tagBoundaries();
//
//    // Sphere 2
//    grid_key_dx<dim,int> start2({centerPoint - rBig, centerPoint - rBig, centerPoint - rBig});
//    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
//    CUDA_LAUNCH_DIM3((insertSphere3D<0>),
//                     gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
//                     sparseGrid.toKernel(), start2, rBig2, rSmall2, 1);
//    hipDeviceSynchronize();
//    sparseGrid.template flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
//    hipDeviceSynchronize();
//
//    // Sphere 3
//    grid_key_dx<dim,int> start3({centerPoint + rBig, centerPoint - rBig, centerPoint - rBig});
//    sparseGrid.setGPUInsertBuffer(gridSize,dim3(1));
//    CUDA_LAUNCH_DIM3((insertSphere3D<0>),
//                     gridSize, dim3(blockEdgeSize*blockEdgeSize*blockEdgeSize,1,1),
//                     sparseGrid.toKernel(), start3, rBig3, rSmall3, 1);
//    hipDeviceSynchronize();
//    sparseGrid.template flush < smax_< 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
//    hipDeviceSynchronize();
//    ///// /////
//
//    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
//    sparseGrid.tagBoundaries();
//
//    sparseGrid.template deviceToHost<0>(); // NECESSARY as count takes place on Host!
//    auto existingElements = sparseGrid.countExistingElements();
//    auto boundaryElements = sparseGrid.countBoundaryElements();
//    unsigned long long numElements = existingElements - boundaryElements;
//
//    // Now apply some boundary conditions
//    sparseGrid.template applyBoundaryStencils<BoundaryStencilSetXRescaled<dim,0,0>>(
//            centerPoint - rBig, centerPoint + (2 * rBig),
//            0.0, 10.0);
//    hipDeviceSynchronize();
//
//    for (unsigned int iter=0; iter<iterations; ++iter)
//    {
//        hipDeviceSynchronize();
//
//        timer ts;
//        ts.start();
//
//        sparseGrid.template applyStencils<Stencil01T>(STENCIL_MODE_INPLACE, 0.1);
//        hipDeviceSynchronize();
//        sparseGrid.template applyStencils<Stencil10T>(STENCIL_MODE_INPLACE, 0.1);
//        hipDeviceSynchronize();
//
//        ts.stop();
//
//        measures_tm.add(ts.getwct());
//
//        float gElemS = numElements / (1e9 * ts.getwct());
//        float gFlopsS = gElemS * Stencil01T::flops;
//
//        measures_gf.add(gFlopsS);
//    }
//
//    double mean_tm = 0;
//    double deviation_tm = 0;
//    standard_deviation(measures_tm,mean_tm,deviation_tm);
//
//    double mean_gf = 0;
//    double deviation_gf = 0;
//    standard_deviation(measures_gf,mean_gf,deviation_gf);
//
//    // All times above are in ms
//
//    float gElemS = numElements / (1e9 * mean_tm);
//    float gFlopsS = gElemS * Stencil01T::flops;
//    std::cout << "Test: " << testName << std::endl;
//    std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << "x" << blockEdgeSize << std::endl;
//    std::cout << "Grid: " << gridEdgeSize * blockEdgeSize
//              << "x" << gridEdgeSize * blockEdgeSize
//              << "x" << gridEdgeSize * blockEdgeSize
//              << std::endl;
//    double dataOccupancyMean, dataOccupancyDev;
//    sparseGrid.deviceToHost();
//    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
//    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
//    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
//    std::cout << "Iterations: " << iterations << std::endl;
//    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
//    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;
//
//    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
//    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
//    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
//    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);
//
////    // DEBUG
////    sparseGrid.template deviceToHost<0,1>();
////    sparseGrid.write("SparseGridGPU_testStencilHeat3DSparse_perf_DEBUG.vtk");
//}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
void testStencilHeat3DSparse_perf(unsigned int i, std::string base, float fillMultiplier=1, float voidMultiplier=1)
{
    auto testName = "In-place 3D sparse stencil";
//    unsigned int gridEdgeSize = 32;
    constexpr unsigned int dim = SparseGridZ::dims;
//    const unsigned int blockEdgeSize = SparseGridZ::blockEdgeSize_;

    typedef HeatStencil<dim, 0, 1> Stencil01T;
    typedef HeatStencil<dim, 1, 0> Stencil10T;

//    std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").stencil");

    report_sparsegrid_funcs.graphs.put(base + ".dim",dim);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x", gridEdgeSize * blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y", gridEdgeSize * blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.z", gridEdgeSize * blockEdgeSize);

    unsigned int iterations = 100;

    openfpm::vector<double> measures_gf;
    openfpm::vector<double> measures_tm;

    dim3 gridSize(gridEdgeSize, gridEdgeSize, gridEdgeSize);
    dim3 blockSize(blockEdgeSize, blockEdgeSize, blockEdgeSize);
    unsigned int spatialEdgeSize = 10000;
    size_t sz[3] = {spatialEdgeSize, spatialEdgeSize, spatialEdgeSize};
    typename SparseGridZ::grid_info blockGeometry(sz);
    SparseGridZ sparseGrid(blockGeometry);
    mgpu::ofp_context_t ctx;
    sparseGrid.template setBackgroundValue<0>(0);

    ///// Insert sparse content, a set of concentric spheres /////
    float allMultiplier = fillMultiplier + voidMultiplier;
    const unsigned int numSpheres = gridEdgeSize / (2*allMultiplier);
    unsigned int centerPoint = spatialEdgeSize / 2;

    for (int i = 1; i <= numSpheres; ++i)
    {
        unsigned int rBig = allMultiplier*i * blockEdgeSize;
        unsigned int rSmall = (allMultiplier*i - fillMultiplier) * blockEdgeSize;
        // Sphere i-th
        grid_key_dx<dim, int> start1({centerPoint, centerPoint, centerPoint});
        sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
        CUDA_LAUNCH_DIM3((insertSphere3D<0>),
                         gridSize, dim3(blockEdgeSize * blockEdgeSize * blockEdgeSize, 1, 1),
                         sparseGrid.toKernel(), start1, rBig, rSmall, 1);
        hipDeviceSynchronize();
        sparseGrid.template flush<smax_<0 >>(ctx, flush_type::FLUSH_ON_DEVICE);
        hipDeviceSynchronize();
    }
    ///// /////

    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
    sparseGrid.tagBoundaries();

    sparseGrid.template deviceToHost<0>(); // NECESSARY as count takes place on Host!
    auto existingElements = sparseGrid.countExistingElements();
    auto boundaryElements = sparseGrid.countBoundaryElements();
    unsigned long long numElements = existingElements - boundaryElements;

    // Now apply some boundary conditions
    sparseGrid.template applyBoundaryStencils<BoundaryStencilSetXRescaled<dim,0,0>>(
            centerPoint, centerPoint + 2*blockEdgeSize*gridEdgeSize,
            0.0, 10.0);
    hipDeviceSynchronize();

    for (unsigned int iter=0; iter<iterations; ++iter)
    {
        hipDeviceSynchronize();

        timer ts;
        ts.start();

        sparseGrid.template applyStencils<Stencil01T>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();
        sparseGrid.template applyStencils<Stencil10T>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();

        ts.stop();

        measures_tm.add(ts.getwct());

        float gElemS = numElements / (1e9 * ts.getwct());
        float gFlopsS = gElemS * Stencil01T::flops;

        measures_gf.add(gFlopsS);
    }

    double mean_tm = 0;
    double deviation_tm = 0;
    standard_deviation(measures_tm,mean_tm,deviation_tm);

    double mean_gf = 0;
    double deviation_gf = 0;
    standard_deviation(measures_gf,mean_gf,deviation_gf);

    // All times above are in ms

    float gElemS = numElements / (1e9 * mean_tm);
    float gFlopsS = gElemS * Stencil01T::flops;
    std::cout << "Test: " << testName << std::endl;
    std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << "x" << blockEdgeSize << std::endl;
    std::cout << "Grid: " << gridEdgeSize * blockEdgeSize
              << "x" << gridEdgeSize * blockEdgeSize
              << "x" << gridEdgeSize * blockEdgeSize
              << std::endl;
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;

    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);

//    // DEBUG
//    sparseGrid.template deviceToHost<0,1>();
//    sparseGrid.write("SparseGridGPU_testStencilHeat3DSparse_perf_DEBUG.vtk");
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeat3DSparse_perf(std::string testURI, unsigned int i,
        float fillMultiplier=1, float voidMultiplier=1)
{
    constexpr unsigned int dim = 3;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilN3DSparse");

    testStencilHeat3DSparse_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize, long int>>(i, base,
                    fillMultiplier, voidMultiplier);
    hipDeviceSynchronize();
}

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
void testStencilHeatHost_perf(unsigned int i, std::string base)
{
    // todo: Make sure to reimplement the host stencil application function to pre-load to a block of memory both content and ghost
    // this way we can avoid binary searches...
    auto testName = "In-place stencil HOST";
    typedef HeatStencil<SparseGridZ::dims,0,1> StencilT;

    constexpr unsigned int dim = 2;

//    std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").stencil");

    report_sparsegrid_funcs.graphs.put(base + ".dim",dim);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*SparseGridZ::blockEdgeSize_);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*SparseGridZ::blockEdgeSize_);

//    unsigned int iterations = 100;
    unsigned int iterations = 10;
//    unsigned int iterations = 2;
//    unsigned int iterations = 1; // Debug

    openfpm::vector<double> measures_gf;
    openfpm::vector<double> measures_tm;

    dim3 gridSize(gridEdgeSize, gridEdgeSize);
    dim3 blockSize(SparseGridZ::blockEdgeSize_,SparseGridZ::blockEdgeSize_);
    typename SparseGridZ::grid_info blockGeometry(gridSize);
    SparseGridZ sparseGrid(blockGeometry);
    mgpu::ofp_context_t ctx;
    sparseGrid.template setBackgroundValue<0>(0);

    unsigned long long numElements = gridEdgeSize*SparseGridZ::blockEdgeSize_*gridEdgeSize*SparseGridZ::blockEdgeSize_;

    // Initialize the grid
    sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
    CUDA_LAUNCH_DIM3((insertConstantValue<0>),gridSize, blockSize,sparseGrid.toKernel(), 0);
    sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
    dim3 sourcePt(gridSize.x * SparseGridZ::blockEdgeSize_ / 2, gridSize.y * SparseGridZ::blockEdgeSize_ / 2, 0);
    insertOneValue<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), sourcePt, 100);
    sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
    hipDeviceSynchronize();

    sparseGrid.template deviceToHost<0>();

    for (unsigned int iter=0; iter<iterations; ++iter)
    {
        hipDeviceSynchronize();

        timer ts;
        ts.start();

        sparseGrid.template applyStencilsHost<StencilT>(STENCIL_MODE_INPLACE, 0.1);

        hipDeviceSynchronize();
        ts.stop();

        measures_tm.add(ts.getwct());

        float gElemS = numElements / (1e9 * ts.getwct());
        float gFlopsS = gElemS * StencilT::flops;

        measures_gf.add(gFlopsS);
    }

    double mean_tm = 0;
    double deviation_tm = 0;
    standard_deviation(measures_tm,mean_tm,deviation_tm);

    double mean_gf = 0;
    double deviation_gf = 0;
    standard_deviation(measures_gf,mean_gf,deviation_gf);

    // All times above are in ms

    float gElemS = numElements / (1e9 * mean_tm);
    float gFlopsS = gElemS * StencilT::flops;

    std::cout << "Test: " << testName << std::endl;
    std::cout << "Host: " << SparseGridZ::blockEdgeSize_ << "x" << SparseGridZ::blockEdgeSize_ << std::endl;
    std::cout << "Grid: " << gridEdgeSize*SparseGridZ::blockEdgeSize_ << "x" << gridEdgeSize*SparseGridZ::blockEdgeSize_ << std::endl;
    double dataOccupancyMean=0, dataOccupancyDev=0;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl
                << "\t " << gFlopsS << " GFlops/s" << std::endl;

    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);
}
template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeatHost_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilN_Host");

    testStencilHeatHost_perf<blockEdgeSize, gridEdgeSize,
        SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize>>(i, base);
}

BOOST_AUTO_TEST_SUITE(performance, *boost::unit_test::fixture<Fixture>())

BOOST_AUTO_TEST_SUITE(SparseGridGpu_test)

//BOOST_AUTO_TEST_CASE(testStencilHeatHost_gridScaling)
//{
//    std::string testURI = suiteURI + ".host.stencil.dense.N.2D.gridScaling";
//    unsigned int counter = 0;
//    launch_testStencilHeatHost_perf<8, 128>(testURI, counter++);
//    launch_testStencilHeatHost_perf<8, 256>(testURI, counter++);
//    launch_testStencilHeatHost_perf<8, 512>(testURI, counter++);
//    launch_testStencilHeatHost_perf<8, 1024>(testURI, counter++);
////    launch_testStencilHeatHost_perf<8, 2048>(testURI, counter++);

//    testSet.insert(testURI);
//}
//
//BOOST_AUTO_TEST_CASE(testStencilHeatHost_blockScaling)
//{
//    std::string testURI = suiteURI + ".host.stencil.dense.N.2D.blockScaling";
//    unsigned int counter = 0;
//    launch_testStencilHeatHost_perf<4, 2048>(testURI, counter++);
//    launch_testStencilHeatHost_perf<8, 1024>(testURI, counter++);
//    launch_testStencilHeatHost_perf<16, 512>(testURI, counter++);
//    launch_testStencilHeatHost_perf<32, 256>(testURI, counter++);

//    testSet.insert(testURI);
//}

BOOST_AUTO_TEST_CASE(testStencilHeat_gridScaling)
//BOOST_AUTO_TEST_CASE(testStencilHeatSparse10_gridScaling)
{
    std::string testURI = suiteURI + ".device.stencil.dense.N.2D.gridScaling";
    unsigned int counter = 0;
    constexpr unsigned int blockEdgeSize = 8;
    launch_testStencilHeat_perf<blockEdgeSize, 128>(testURI, counter++);
    launch_testStencilHeat_perf<blockEdgeSize, 256>(testURI, counter++);
    launch_testStencilHeat_perf<blockEdgeSize, 512>(testURI, counter++);
    launch_testStencilHeat_perf<blockEdgeSize, 1024>(testURI, counter++);
//    launch_testStencilHeat_perf<blockEdgeSize, 2048>(testURI, counter++);

    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeat_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencil.dense.N.2D.blockScaling";
    unsigned int counter = 0;
    // Note - blockEdgeSize == 2 doesn't work
    launch_testStencilHeat_perf<4, 2048>(testURI, counter++);
    launch_testStencilHeat_perf<8, 1024>(testURI, counter++);
    launch_testStencilHeat_perf<16, 512>(testURI, counter++);
    launch_testStencilHeat_perf<32, 256>(testURI, counter++);

    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeatZ_gridScaling)
{
    std::string testURI = suiteURI + ".device.stencil.dense.Z.2D.gridScaling";
    unsigned int counter = 0;
    constexpr unsigned int blockEdgeSize = 8;
    launch_testStencilHeatZ_perf<blockEdgeSize, 128>(testURI, counter++);
    launch_testStencilHeatZ_perf<blockEdgeSize, 256>(testURI, counter++);
    launch_testStencilHeatZ_perf<blockEdgeSize, 512>(testURI, counter++);
    launch_testStencilHeatZ_perf<blockEdgeSize, 1024>(testURI, counter++);
//    launch_testStencilHeatZ_perf<blockEdgeSize, 2048>(testURI, counter++);

    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeatZ_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencil.dense.Z.2D.blockScaling";
    unsigned int counter = 0;
    // Note - blockEdgeSize == 2 doesn't work
    launch_testStencilHeatZ_perf<4, 2048>(testURI, counter++);
    launch_testStencilHeatZ_perf<8, 1024>(testURI, counter++);
    launch_testStencilHeatZ_perf<16, 512>(testURI, counter++);
    launch_testStencilHeatZ_perf<32, 256>(testURI, counter++);

    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeat3D_gridScaling)
{
    std::string testURI = suiteURI + ".device.stencil.dense.N.3D.gridScaling";
    unsigned int counter = 0;
    constexpr unsigned int blockEdgeSize = 8;
    launch_testStencilHeat3D_perf<blockEdgeSize, 8>(testURI, counter++);
    launch_testStencilHeat3D_perf<blockEdgeSize, 16>(testURI, counter++);
    launch_testStencilHeat3D_perf<blockEdgeSize, 32>(testURI, counter++);
    launch_testStencilHeat3D_perf<blockEdgeSize, 64>(testURI, counter++);
//    launch_testStencilHeat3D_perf<blockEdgeSize, 128>(testURI, counter++);

    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeat3D_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencil.dense.N.3D.blockScaling";
    unsigned int counter = 0;
    launch_testStencilHeat3D_perf<2, 128>(testURI, counter++);
    launch_testStencilHeat3D_perf<4, 64>(testURI, counter++);
    launch_testStencilHeat3D_perf<8, 32>(testURI, counter++);
//    launch_testStencilHeat3D_perf<16, 16>(testURI, counter++); // Too big, it doesn't work

    testSet.insert(testURI);
}

//BOOST_AUTO_TEST_CASE(testStencilHeatZ3D)
//{
//    constexpr unsigned int dim = 3;
//    constexpr unsigned int blockEdgeSize = 4;
//
//    typedef aggregate<float,float> AggregateT;
//    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;
//
//    report_sparsegrid_funcs.graphs.put("performance.SparseGridGpu(1).stencil.test.name","StencilZ3D");
//
//    testStencilHeat3D_perf<SparseGridGpu_z<dim, AggregateT, blockEdgeSize, chunkSize>>(1);
//}

BOOST_AUTO_TEST_CASE(testStencilHeatSparse05_gridScaling)
{
    std::string testURI = suiteURI + ".device.stencil.sparse.N.2D.05.gridScaling";
    unsigned int counter = 0;
    constexpr unsigned int blockEdgeSize = 8;
    launch_testStencilHeatSparse_perf<blockEdgeSize, 128>(testURI, counter++, 1, 1, "05");
    launch_testStencilHeatSparse_perf<blockEdgeSize, 256>(testURI, counter++, 1, 1, "05");
    launch_testStencilHeatSparse_perf<blockEdgeSize, 512>(testURI, counter++, 1, 1, "05");
    launch_testStencilHeatSparse_perf<blockEdgeSize, 1024>(testURI, counter++, 1, 1, "05");
////    launch_testStencilHeatSparse_perf<blockEdgeSize, 2048>(testURI, counter++, 1, 1, "05);

    testSet.insert(testURI);
}
BOOST_AUTO_TEST_CASE(testStencilHeatSparse05_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencil.sparse.N.2D.05.blockScaling";
    unsigned int counter = 0;
    // Note - blockEdgeSize == 2 doesn't work
    launch_testStencilHeatSparse_perf<2, 2048>(testURI, counter++, 1, 1, "05");
    launch_testStencilHeatSparse_perf<4, 1024>(testURI, counter++, 1, 1, "05");
    launch_testStencilHeatSparse_perf<8, 512>(testURI, counter++, 1, 1, "05");
    launch_testStencilHeatSparse_perf<16, 256>(testURI, counter++, 1, 1, "05");
    launch_testStencilHeatSparse_perf<32, 128>(testURI, counter++, 1, 1, "05");

    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeatSparse08_gridScaling)
{
    std::string testURI = suiteURI + ".device.stencil.sparse.N.2D.08.gridScaling";
    unsigned int counter = 0;
    constexpr unsigned int blockEdgeSize = 8;
    launch_testStencilHeatSparse_perf<blockEdgeSize, 128>(testURI, counter++, 1, 0.25, "08");
    launch_testStencilHeatSparse_perf<blockEdgeSize, 256>(testURI, counter++, 1, 0.25, "08");
    launch_testStencilHeatSparse_perf<blockEdgeSize, 512>(testURI, counter++, 1, 0.25, "08");
    launch_testStencilHeatSparse_perf<blockEdgeSize, 1024>(testURI, counter++, 1, 0.25, "08");
////    launch_testStencilHeatSparse_perf<blockEdgeSize, 2048>(testURI, counter++, 1, 0.25, "08");

    testSet.insert(testURI);
}
BOOST_AUTO_TEST_CASE(testStencilHeatSparse08_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencil.sparse.N.2D.08.blockScaling";
    unsigned int counter = 0;
    launch_testStencilHeatSparse_perf<2, 2048>(testURI, counter++, 1, 0.25, "08");
    launch_testStencilHeatSparse_perf<4, 1024>(testURI, counter++, 1, 0.25, "08");
    launch_testStencilHeatSparse_perf<8, 512>(testURI, counter++, 1, 0.25, "08");
    launch_testStencilHeatSparse_perf<16, 256>(testURI, counter++, 1, 0.25, "08");
    launch_testStencilHeatSparse_perf<32, 128>(testURI, counter++, 1, 0.25, "08");

    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeatSparse09_gridScaling)
{
    std::string testURI = suiteURI + ".device.stencil.sparse.N.2D.09.gridScaling";
    unsigned int counter = 0;
    constexpr unsigned int blockEdgeSize = 8;
    launch_testStencilHeatSparse_perf<blockEdgeSize, 128>(testURI, counter++, 1, 0.1, "09");
    launch_testStencilHeatSparse_perf<blockEdgeSize, 256>(testURI, counter++, 1, 0.1, "09");
    launch_testStencilHeatSparse_perf<blockEdgeSize, 512>(testURI, counter++, 1, 0.1, "09");
    launch_testStencilHeatSparse_perf<blockEdgeSize, 1024>(testURI, counter++, 1, 0.1, "09");
////    launch_testStencilHeatSparse_perf<blockEdgeSize, 2048>(testURI, counter++, 1, 0.1, "09");

    testSet.insert(testURI);
}
BOOST_AUTO_TEST_CASE(testStencilHeatSparse09_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencil.sparse.N.2D.09.blockScaling";
    unsigned int counter = 0;
    // Note - blockEdgeSize == 2 doesn't work
    launch_testStencilHeatSparse_perf<2, 2048>(testURI, counter++, 1, 0.1, "09");
    launch_testStencilHeatSparse_perf<4, 1024>(testURI, counter++, 1, 0.1, "09");
    launch_testStencilHeatSparse_perf<8, 512>(testURI, counter++, 1, 0.1, "09");
    launch_testStencilHeatSparse_perf<16, 256>(testURI, counter++, 1, 0.1, "09");
    launch_testStencilHeatSparse_perf<32, 128>(testURI, counter++, 1, 0.1, "09");

    testSet.insert(testURI);
}

        BOOST_AUTO_TEST_CASE(testStencilHeatSparseZ05_gridScaling)
        {
            std::string testURI = suiteURI + ".device.stencil.sparse.Z.2D.05.gridScaling";
            unsigned int counter = 0;
            constexpr unsigned int blockEdgeSize = 8;
            launch_testStencilHeatSparseZ_perf<blockEdgeSize, 128>(testURI, counter++, 1, 1, "05");
            launch_testStencilHeatSparseZ_perf<blockEdgeSize, 256>(testURI, counter++, 1, 1, "05");
            launch_testStencilHeatSparseZ_perf<blockEdgeSize, 512>(testURI, counter++, 1, 1, "05");
            launch_testStencilHeatSparseZ_perf<blockEdgeSize, 1024>(testURI, counter++, 1, 1, "05");
////    launch_testStencilHeatSparseZ_perf<blockEdgeSize, 2048>(testURI, counter++, 1, 1, "05);

            testSet.insert(testURI);
        }
        BOOST_AUTO_TEST_CASE(testStencilHeatSparseZ05_blockScaling)
        {
            std::string testURI = suiteURI + ".device.stencil.sparse.Z.2D.05.blockScaling";
            unsigned int counter = 0;
            // Note - blockEdgeSize == 2 doesn't work
            launch_testStencilHeatSparseZ_perf<2, 2048>(testURI, counter++, 1, 1, "05");
            launch_testStencilHeatSparseZ_perf<4, 1024>(testURI, counter++, 1, 1, "05");
            launch_testStencilHeatSparseZ_perf<8, 512>(testURI, counter++, 1, 1, "05");
            launch_testStencilHeatSparseZ_perf<16, 256>(testURI, counter++, 1, 1, "05");
            launch_testStencilHeatSparseZ_perf<32, 128>(testURI, counter++, 1, 1, "05");

            testSet.insert(testURI);
        }

        BOOST_AUTO_TEST_CASE(testStencilHeatSparseZ08_gridScaling)
        {
            std::string testURI = suiteURI + ".device.stencil.sparse.Z.2D.08.gridScaling";
            unsigned int counter = 0;
            constexpr unsigned int blockEdgeSize = 8;
            launch_testStencilHeatSparseZ_perf<blockEdgeSize, 128>(testURI, counter++, 1, 0.25, "08");
            launch_testStencilHeatSparseZ_perf<blockEdgeSize, 256>(testURI, counter++, 1, 0.25, "08");
            launch_testStencilHeatSparseZ_perf<blockEdgeSize, 512>(testURI, counter++, 1, 0.25, "08");
            launch_testStencilHeatSparseZ_perf<blockEdgeSize, 1024>(testURI, counter++, 1, 0.25, "08");
////    launch_testStencilHeatSparseZ_perf<blockEdgeSize, 2048>(testURI, counter++, 1, 0.25, "08");

            testSet.insert(testURI);
        }
        BOOST_AUTO_TEST_CASE(testStencilHeatSparseZ08_blockScaling)
        {
            std::string testURI = suiteURI + ".device.stencil.sparse.Z.2D.08.blockScaling";
            unsigned int counter = 0;
            launch_testStencilHeatSparseZ_perf<2, 2048>(testURI, counter++, 1, 0.25, "08");
            launch_testStencilHeatSparseZ_perf<4, 1024>(testURI, counter++, 1, 0.25, "08");
            launch_testStencilHeatSparseZ_perf<8, 512>(testURI, counter++, 1, 0.25, "08");
            launch_testStencilHeatSparseZ_perf<16, 256>(testURI, counter++, 1, 0.25, "08");
            launch_testStencilHeatSparseZ_perf<32, 128>(testURI, counter++, 1, 0.25, "08");

            testSet.insert(testURI);
        }

        BOOST_AUTO_TEST_CASE(testStencilHeatSparseZ09_gridScaling)
        {
            std::string testURI = suiteURI + ".device.stencil.sparse.Z.2D.09.gridScaling";
            unsigned int counter = 0;
            constexpr unsigned int blockEdgeSize = 8;
            launch_testStencilHeatSparseZ_perf<blockEdgeSize, 128>(testURI, counter++, 1, 0.1, "09");
            launch_testStencilHeatSparseZ_perf<blockEdgeSize, 256>(testURI, counter++, 1, 0.1, "09");
            launch_testStencilHeatSparseZ_perf<blockEdgeSize, 512>(testURI, counter++, 1, 0.1, "09");
            launch_testStencilHeatSparseZ_perf<blockEdgeSize, 1024>(testURI, counter++, 1, 0.1, "09");
////    launch_testStencilHeatSparseZ_perf<blockEdgeSize, 2048>(testURI, counter++, 1, 0.1, "09");

            testSet.insert(testURI);
        }
        BOOST_AUTO_TEST_CASE(testStencilHeatSparseZ09_blockScaling)
        {
            std::string testURI = suiteURI + ".device.stencil.sparse.Z.2D.09.blockScaling";
            unsigned int counter = 0;
            // Note - blockEdgeSize == 2 doesn't work
            launch_testStencilHeatSparseZ_perf<2, 2048>(testURI, counter++, 1, 0.1, "09");
            launch_testStencilHeatSparseZ_perf<4, 1024>(testURI, counter++, 1, 0.1, "09");
            launch_testStencilHeatSparseZ_perf<8, 512>(testURI, counter++, 1, 0.1, "09");
            launch_testStencilHeatSparseZ_perf<16, 256>(testURI, counter++, 1, 0.1, "09");
            launch_testStencilHeatSparseZ_perf<32, 128>(testURI, counter++, 1, 0.1, "09");

            testSet.insert(testURI);
        }

BOOST_AUTO_TEST_CASE(testStencilHeat3DSparse_gridScaling)
{
    std::string testURI = suiteURI + ".device.stencil.sparse.N.3D.gridScaling";
    unsigned int counter = 0;
    constexpr unsigned int blockEdgeSize = 8;
    launch_testStencilHeat3DSparse_perf<blockEdgeSize, 8>(testURI, counter++, 1, 1);
    launch_testStencilHeat3DSparse_perf<blockEdgeSize, 16>(testURI, counter++, 1, 1);
    launch_testStencilHeat3DSparse_perf<blockEdgeSize, 32>(testURI, counter++, 1, 1);
    launch_testStencilHeat3DSparse_perf<blockEdgeSize, 64>(testURI, counter++, 1, 1);

    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeat3DSparse_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencil.sparse.N.3D.blockScaling";
    unsigned int counter = 0;
    launch_testStencilHeat3DSparse_perf<2, 128>(testURI, counter++, 1, 1);
    launch_testStencilHeat3DSparse_perf<4, 64>(testURI, counter++, 1, 1);
    launch_testStencilHeat3DSparse_perf<8, 32>(testURI, counter++, 1, 1);
//    launch_testStencilHeat3DSparse_perf<16, 16>(testURI, counter++, 1, 1); // Too big, it doesn't work

    testSet.insert(testURI);
}

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void testInsertStencil(std::string testURI, unsigned int i)
{
	auto testName = "Insert stencil";
	constexpr unsigned int dim = 2;
//	constexpr unsigned int blockEdgeSize = 8;
	constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;
	typedef aggregate<float,float> AggregateT;
	typedef HeatStencil<dim,0,1> StencilT;

	unsigned int iterations = 10;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilInsertN");

    report_sparsegrid_funcs.graphs.put(base + ".dim",2);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*blockEdgeSize);

	dim3 gridSize(gridEdgeSize, gridEdgeSize);
	dim3 blockSize(blockEdgeSize, blockEdgeSize);
	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	// Initialize the grid
	sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
	CUDA_LAUNCH_DIM3((insertConstantValue<0>),gridSize, blockSize,sparseGrid.toKernel(), 0);
	sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
	dim3 sourcePt(gridSize.x * blockEdgeSize / 2, gridSize.y * blockEdgeSize / 2, 0);
	insertOneValue<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), sourcePt, 100);
	sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

	sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!

	unsigned long long numElements = gridEdgeSize*blockEdgeSize*gridEdgeSize*blockEdgeSize;

	for (unsigned int iter=0; iter<5; ++iter)
	{
		sparseGrid.template applyStencils<StencilT>(STENCIL_MODE_INSERT, 0.1);
		sparseGrid.template flush<smax_<0>>(ctx, flush_type::FLUSH_ON_DEVICE);
	}

    openfpm::vector<double> gElemSMeasures;
    openfpm::vector<double> gFlopsSMeasures;

    for (unsigned int iter=0; iter<iterations; ++iter)
	{
		hipDeviceSynchronize();

		timer ts;
		ts.start();

		sparseGrid.template applyStencils<StencilT>(STENCIL_MODE_INSERT, 0.1);
		sparseGrid.template flush<smax_<0>>(ctx, flush_type::FLUSH_ON_DEVICE);

		hipDeviceSynchronize();

		ts.stop();

		float gElemS = numElements / (1e9 * ts.getwct());
		float gFlopsS = gElemS * StencilT::flops;

		gElemSMeasures.add(gElemS);
		gFlopsSMeasures.add(gFlopsS);
	}


	double elemMean=0, elemDeviation=0;
	standard_deviation(gElemSMeasures, elemMean, elemDeviation);
    report_sparsegrid_funcs.graphs.put(base + ".GElems.mean",elemMean);
    report_sparsegrid_funcs.graphs.put(base +".GElems.dev",elemDeviation);
    double flopsMean=0, flopsDeviation=0;
    standard_deviation(gFlopsSMeasures, flopsMean, flopsDeviation);
    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",flopsMean);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",flopsDeviation);

	std::cout << "Test: " << testName << "\n";
	std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << "\n";
	std::cout << "Grid: " << gridEdgeSize*blockEdgeSize << "x" << gridEdgeSize*blockEdgeSize << "\n";
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << "\n";
	std::cout << "Throughput:\n\t" << elemMean << " GElem/s dev: " << elemDeviation << " GElem/s" << std::endl
	            << "\t" << flopsMean << " GFlops/s dev: " << flopsDeviation << " GFlops/s" << std::endl;
}

BOOST_AUTO_TEST_CASE(testStencilHeatInsert_gridScaling)
{
    std::string testURI = suiteURI + ".device.stencilInsert.dense.N.2D.gridScaling";
    unsigned int counter = 0;
    testInsertStencil<8, 64>(testURI, counter++);
	testInsertStencil<8, 128>(testURI, counter++);
	testInsertStencil<8, 256>(testURI, counter++);
	testInsertStencil<8, 512>(testURI, counter++);
	testInsertStencil<8, 1024>(testURI, counter++);

    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeatInsert_blockScaling)
{
    std::string testURI = suiteURI + ".device.stencilInsert.dense.N.2D.blockScaling";
    unsigned int counter = 0;
    testInsertStencil<4, 1024>(testURI, counter++);
    testInsertStencil<8, 512>(testURI, counter++);
    testInsertStencil<16, 256>(testURI, counter++);
    testInsertStencil<32, 128>(testURI, counter++);

    testSet.insert(testURI);
}

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void testInsertSingle(std::string testURI, unsigned int i)
{
	auto testName = "Insert single (one chunk per element)";
	constexpr unsigned int dim = 2;
//	constexpr unsigned int blockEdgeSize = 8;
	constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;
	typedef aggregate<float> AggregateT;

	unsigned int iterations = 10;
	bool prePopulateGrid = true;

//    std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").insertSingle");
    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","InsertSingle");

    report_sparsegrid_funcs.graphs.put(base + ".dim",dim);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*blockEdgeSize);

	dim3 gridSize(gridEdgeSize, gridEdgeSize);
	dim3 blockSize(blockEdgeSize, blockEdgeSize);
	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	if (prePopulateGrid)
	{
		// Pre-populate grid
		sparseGrid.setGPUInsertBuffer(gridSize, blockSize);
		insertValues2D<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), 0, 0);
		sparseGrid.template flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
		hipDeviceSynchronize();
		///
	}

	for (unsigned int iter=0; iter<5; ++iter)
	{
		auto offset = 0;
		sparseGrid.setGPUInsertBuffer(gridSize, blockSize);
		insertValues2D<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), offset, offset);
		sparseGrid.template flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
		hipDeviceSynchronize();
	}

	unsigned long long numElements = gridEdgeSize*blockEdgeSize*gridEdgeSize*blockEdgeSize;
	openfpm::vector<double> measures;

	for (unsigned int iter=0; iter<iterations; ++iter)
	{
		auto offset = 0;

		hipDeviceSynchronize();

		timer ts;
		ts.start();

		sparseGrid.setGPUInsertBuffer(gridSize, blockSize);
		insertValues2D<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), offset, offset);
		sparseGrid.template flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
		hipDeviceSynchronize();

		ts.stop();

		float mElemS = numElements / (1e6 * ts.getwct());
		measures.add(mElemS);
	}

	double mean = 0;
	double deviation = 0;
	standard_deviation(measures,mean,deviation);

    report_sparsegrid_funcs.graphs.put(base + ".Minsert.mean",mean);
    report_sparsegrid_funcs.graphs.put(base +".Minsert.dev",deviation);

	// All times above are in ms

	std::cout << "Test: " << testName << "\n";
    std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << "\n";
    std::cout << "Grid: " << gridEdgeSize*blockEdgeSize << "x" << gridEdgeSize*blockEdgeSize << "\n";
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << "\n";
	std::cout << "Throughput:\n\t" << mean << "M/s" << "\n";
}

BOOST_AUTO_TEST_CASE(testInsert_gridScaling)
{
    std::string testURI = suiteURI + ".device.insert.dense.single.2D.gridScaling";
    unsigned int counter = 0;
    testInsertSingle<8, 64>(testURI, counter++);
    testInsertSingle<8, 128>(testURI, counter++);
    testInsertSingle<8, 256>(testURI, counter++);
//    testInsertSingle<8, 512>(testURI, counter++);
//    testInsertSingle<8, 1024>(testURI, counter++);

    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testInsert_blockScaling)
{
    std::string testURI = suiteURI + ".device.insert.dense.single.2D.blockScaling";
    unsigned int counter = 0;
    testInsertSingle<2, 1024>(testURI, counter++);
    testInsertSingle<4, 512>(testURI, counter++);
    testInsertSingle<8, 256>(testURI, counter++);
//    testInsertSingle<16, 128>(testURI, counter++);
//    testInsertSingle<32, 64>(testURI, counter++);

    testSet.insert(testURI);
}

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void test_insert_block(std::string testURI, unsigned int i)
{
	auto testName = "Insert (one chunk per block)";
	constexpr unsigned int dim = 2;
//	constexpr unsigned int blockEdgeSize = 8;
	constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;
	typedef aggregate<float> AggregateT;

//	std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").insert");
    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","InsertBlock");

	report_sparsegrid_funcs.graphs.put(base + ".name","Block insert");
    report_sparsegrid_funcs.graphs.put(base + ".dim",dim);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*blockEdgeSize);

	unsigned int iterations = 10;

	openfpm::vector<double> measures;

	unsigned long long numElements = gridEdgeSize*blockEdgeSize*gridEdgeSize*blockEdgeSize;
	dim3 gridSize(gridEdgeSize, gridEdgeSize);
	dim3 blockSize(blockEdgeSize, blockEdgeSize);
	dim3 blockSizeBlockedInsert(1, 1);
	grid_smb<dim, blockEdgeSize> blockGeometry(gridSize);
	SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize> sparseGrid(blockGeometry);
	mgpu::ofp_context_t ctx;
	sparseGrid.template setBackgroundValue<0>(0);

	// Warmup
	for (unsigned int iter=0; iter<5; ++iter)
	{
		auto offset = 0;
		sparseGrid.setGPUInsertBuffer(gridSize, blockSizeBlockedInsert);
		insertValues2DBlocked<0, 1, blockEdgeSize> << < gridSize, blockSize >> >
				(sparseGrid.toKernel(), offset, offset);
		sparseGrid.template flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);
	}


	hipDeviceSynchronize();


	for (unsigned int iter=0; iter<iterations; ++iter)
	{
		auto offset = 0;

		hipDeviceSynchronize();

		timer ts;
		ts.start();

		sparseGrid.setGPUInsertBuffer(gridSize, blockSizeBlockedInsert);
		insertValues2DBlocked<0, 1, blockEdgeSize> << < gridSize, blockSize >> >
				(sparseGrid.toKernel(), offset, offset);
		sparseGrid.template flush < smax_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

		hipDeviceSynchronize();

		ts.stop();

		float mElemS = numElements / (1e6 * ts.getwct());
		measures.add(mElemS);
	}

	double mean = 0;
	double deviation = 0;
	standard_deviation(measures,mean,deviation);

    report_sparsegrid_funcs.graphs.put(base + ".Minsert.mean",mean);
    report_sparsegrid_funcs.graphs.put(base +".Minsert.dev",deviation);

	// All times above are in ms

	std::cout << "Test: " << testName << "\n";
    std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << "\n";
    std::cout << "Grid: " << gridEdgeSize*blockEdgeSize << "x" << gridEdgeSize*blockEdgeSize << "\n";
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << "\n";
	std::cout << "\tInsert: " << mean << " dev: " << deviation << " s" << std::endl;
	std::cout << "Throughput:\n\t" << mean << " MElem/s\n";
}

BOOST_AUTO_TEST_CASE(testInsertBlocked_gridScaling)
{
    std::string testURI = suiteURI + ".device.insert.dense.block.2D.gridScaling";
    unsigned int counter = 0;
    test_insert_block<8,64>(testURI, counter++);
    test_insert_block<8,128>(testURI, counter++);
    test_insert_block<8,256>(testURI, counter++);
    test_insert_block<8,512>(testURI, counter++);
    test_insert_block<8,1024>(testURI, counter++);
//    test_insert_block<8,2048>(testURI, counter++); // Out of memory

    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testInsertBlocked_blockScaling)
{
    std::string testURI = suiteURI + ".device.insert.dense.block.2D.blockScaling";
    unsigned int counter = 0;
    test_insert_block<2,2048>(testURI, counter++);
    test_insert_block<4,1024>(testURI, counter++);
    test_insert_block<8,512>(testURI, counter++);
    test_insert_block<16,256>(testURI, counter++);
    test_insert_block<32,128>(testURI, counter++);

    testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(write_teport)
{
    write_test_report(report_sparsegrid_funcs, testSet);
}

BOOST_AUTO_TEST_SUITE_END()

BOOST_AUTO_TEST_SUITE_END()
