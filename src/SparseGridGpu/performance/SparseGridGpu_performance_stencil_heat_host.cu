/*
 * SparseGridGpu_performance_host.cu
 *
 *  Created on: Sep 10, 2019
 *      Author: i-bird
 */

#define SCAN_WITH_CUB
#define BOOST_TEST_DYN_LINK
#define OPENFPM_DATA_ENABLE_IO_MODULE
#define DISABLE_MPI_WRITTERS

#include <boost/test/unit_test.hpp>
#include "performancePlots.hpp"
#include <iostream>
#include "SparseGridGpu/SparseGridGpu.hpp"
#include "SparseGridGpu/tests/utils/SparseGridGpu_util_test.cuh"

extern std::string suiteURI;
extern report_sparse_grid_tests report_sparsegrid_funcs;
extern std::set<std::string> testSet;

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
void testStencilHeatSparseHost_perf(unsigned int i, std::string base, float fillMultiplier=1, float voidMultiplier=1)
{
    auto testName = "In-place sparse stencil";
//    unsigned int gridEdgeSize = 128;
    constexpr unsigned int dim = SparseGridZ::dims;
//    const unsigned int blockEdgeSize = SparseGridZ::blockEdgeSize_;

    typedef HeatStencil<dim, 0, 1> Stencil01T;
    typedef HeatStencil<dim, 1, 0> Stencil10T;

//    std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").stencil");

    report_sparsegrid_funcs.graphs.put(base + ".dim",2);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*blockEdgeSize);

    unsigned int iterations = 100;

    openfpm::vector<double> measures_gf;
    openfpm::vector<double> measures_tm;

    dim3 gridSize(gridEdgeSize, gridEdgeSize);
    dim3 blockSize(blockEdgeSize,blockEdgeSize);
    unsigned int spatialEdgeSize = 1000000;
    size_t sz[2] = {spatialEdgeSize, spatialEdgeSize};
    typename SparseGridZ::grid_info blockGeometry(sz);
    SparseGridZ sparseGrid(blockGeometry);
    mgpu::ofp_context_t ctx;
    sparseGrid.template setBackgroundValue<0>(0);

    ///// Insert sparse content, a set of concentric spheres /////
    float allMultiplier = fillMultiplier + voidMultiplier;
    const unsigned int numSpheres = gridEdgeSize / (2*allMultiplier);
//    const unsigned int numSpheres = 1;
    unsigned int centerPoint = spatialEdgeSize / 2;

    for (int i = 1; i <= numSpheres; ++i)
    {
        unsigned int rBig = allMultiplier*i * blockEdgeSize;
        unsigned int rSmall = (allMultiplier*i - fillMultiplier) * blockEdgeSize;
        // Sphere i-th
        grid_key_dx<dim, int> start1({centerPoint, centerPoint});
        sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
        CUDA_LAUNCH_DIM3((insertSphere<0>),
                         gridSize, dim3(blockEdgeSize * blockEdgeSize, 1, 1),
                         sparseGrid.toKernel(), start1, rBig, rSmall, 5);
        hipDeviceSynchronize();
        sparseGrid.template flush<smax_<0 >>(ctx, flush_type::FLUSH_ON_DEVICE);
        hipDeviceSynchronize();
    }
    ///// /////

    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
    sparseGrid.tagBoundaries();

    sparseGrid.template deviceToHost<0>(); // NECESSARY as count takes place on Host!
    auto existingElements = sparseGrid.countExistingElements();
    auto boundaryElements = sparseGrid.countBoundaryElements();
    unsigned long long numElements = existingElements - boundaryElements;

    // Now apply some boundary conditions
    sparseGrid.template applyBoundaryStencils<BoundaryStencilSetXRescaled<dim,0,0>>(
            centerPoint, centerPoint + 2*blockEdgeSize*gridEdgeSize,
            0.0, 10.0);
    hipDeviceSynchronize();

    sparseGrid.template deviceToHost<0>(); // NECESSARY as stencils are applied on Host!

    iterations /= 2;
    for (unsigned int iter=0; iter<iterations; ++iter)
    {
        hipDeviceSynchronize();

        timer ts;
        ts.start();

        sparseGrid.template applyStencilsHost<Stencil01T>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();
        sparseGrid.template applyStencilsHost<Stencil10T>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();

        ts.stop();

        measures_tm.add(ts.getwct());

        float gElemS = 2 * numElements / (1e9 * ts.getwct());
        float gFlopsS = gElemS * Stencil01T::flops;

        measures_gf.add(gFlopsS);
    }

    double mean_tm = 0;
    double deviation_tm = 0;
    standard_deviation(measures_tm,mean_tm,deviation_tm);

    double mean_gf = 0;
    double deviation_gf = 0;
    standard_deviation(measures_gf,mean_gf,deviation_gf);

    // All times above are in ms

    float gElemS =  2 * numElements / (1e9 * mean_tm);
    float gFlopsS = gElemS * Stencil01T::flops;
    std::cout << "Test: " << testName << std::endl;
    std::cout << "Block: " << blockEdgeSize << "x" << blockEdgeSize << std::endl;
    std::cout << "Grid: " << gridEdgeSize*blockEdgeSize << "x" << gridEdgeSize*blockEdgeSize << std::endl;
    double dataOccupancyMean, dataOccupancyDev;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl << "\t " << gFlopsS << " GFlops/s" << std::endl;

    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);
}


template<unsigned int blockEdgeSize, unsigned int gridEdgeSize, typename SparseGridZ>
void testStencilHeatHost_perf(unsigned int i, std::string base)
{
    // todo: Make sure to reimplement the host stencil application function to pre-load to a block of memory both content and ghost
    // this way we can avoid binary searches...
    auto testName = "In-place stencil HOST";
    typedef HeatStencil<SparseGridZ::dims,0,1> Stencil01T;
    typedef HeatStencil<SparseGridZ::dims,1,0> Stencil10T;


    constexpr unsigned int dim = 2;

//    std::string base("performance.SparseGridGpu(" + std::to_string(i) + ").stencil");

    report_sparsegrid_funcs.graphs.put(base + ".dim",dim);
    report_sparsegrid_funcs.graphs.put(base + ".blockSize",blockEdgeSize);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.x",gridEdgeSize*SparseGridZ::blockEdgeSize_);
    report_sparsegrid_funcs.graphs.put(base + ".gridSize.y",gridEdgeSize*SparseGridZ::blockEdgeSize_);

//    unsigned int iterations = 100;
    unsigned int iterations = 10;
//    unsigned int iterations = 2;
//    unsigned int iterations = 1; // Debug

    openfpm::vector<double> measures_gf;
    openfpm::vector<double> measures_tm;

    dim3 gridSize(gridEdgeSize, gridEdgeSize);
    dim3 blockSize(SparseGridZ::blockEdgeSize_,SparseGridZ::blockEdgeSize_);
    typename SparseGridZ::grid_info blockGeometry(gridSize);
    SparseGridZ sparseGrid(blockGeometry);
    mgpu::ofp_context_t ctx;
    sparseGrid.template setBackgroundValue<0>(0);

    unsigned long long numElements = gridEdgeSize*SparseGridZ::blockEdgeSize_*gridEdgeSize*SparseGridZ::blockEdgeSize_;

    // Initialize the grid
    sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
    CUDA_LAUNCH_DIM3((insertConstantValue<0>),gridSize, blockSize,sparseGrid.toKernel(), 0);
    sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.setGPUInsertBuffer(gridSize, dim3(1));
    dim3 sourcePt(gridSize.x * SparseGridZ::blockEdgeSize_ / 2, gridSize.y * SparseGridZ::blockEdgeSize_ / 2, 0);
    insertOneValue<0> << < gridSize, blockSize >> > (sparseGrid.toKernel(), sourcePt, 100);
    sparseGrid.template flush < sRight_ < 0 >> (ctx, flush_type::FLUSH_ON_DEVICE);

    sparseGrid.findNeighbours(); // Pre-compute the neighbours pos for each block!
    hipDeviceSynchronize();

    sparseGrid.template deviceToHost<0,1>();

    iterations /= 2;
    for (unsigned int iter=0; iter<iterations; ++iter)
    {
        hipDeviceSynchronize();

        timer ts;
        ts.start();

        sparseGrid.template applyStencilsHost<Stencil01T>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();
        sparseGrid.template applyStencilsHost<Stencil10T>(STENCIL_MODE_INPLACE, 0.1);
        hipDeviceSynchronize();

        ts.stop();

        measures_tm.add(ts.getwct());

        float gElemS = 2 * numElements / (1e9 * ts.getwct());
        float gFlopsS = gElemS * Stencil01T::flops;

        measures_gf.add(gFlopsS);
    }

    double mean_tm = 0;
    double deviation_tm = 0;
    standard_deviation(measures_tm,mean_tm,deviation_tm);

    double mean_gf = 0;
    double deviation_gf = 0;
    standard_deviation(measures_gf,mean_gf,deviation_gf);

    // All times above are in ms

    float gElemS = 2 * numElements / (1e9 * mean_tm);
    float gFlopsS = gElemS * Stencil01T::flops;

    std::cout << "Test: " << testName << std::endl;
    std::cout << "Host: " << SparseGridZ::blockEdgeSize_ << "x" << SparseGridZ::blockEdgeSize_ << std::endl;
    std::cout << "Grid: " << gridEdgeSize*SparseGridZ::blockEdgeSize_ << "x" << gridEdgeSize*SparseGridZ::blockEdgeSize_ << std::endl;
    double dataOccupancyMean=0, dataOccupancyDev=0;
    sparseGrid.deviceToHost();
    sparseGrid.measureBlockOccupancy(dataOccupancyMean, dataOccupancyDev);std::cout << "Data Occupancy: " << dataOccupancyMean << " dev:" << dataOccupancyDev << std::endl;
    report_sparsegrid_funcs.graphs.put(base + ".dataOccupancy.mean",dataOccupancyMean);
    report_sparsegrid_funcs.graphs.put(base +".dataOccupancy.dev",dataOccupancyDev);
    std::cout << "Iterations: " << iterations << std::endl;
    std::cout << "\tStencil: " << mean_gf << " dev:" << deviation_gf << " s" << std::endl;
    std::cout << "Throughput: " << std::endl << "\t " << gElemS << " GElem/s " << std::endl
                << "\t " << gFlopsS << " GFlops/s" << std::endl;

    report_sparsegrid_funcs.graphs.put(base + ".GFlops.mean",mean_gf);
    report_sparsegrid_funcs.graphs.put(base +".GFlops.dev",deviation_gf);
    report_sparsegrid_funcs.graphs.put(base + ".time.mean",mean_tm);
    report_sparsegrid_funcs.graphs.put(base +".time.dev",deviation_tm);
}

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeatSparseHost_perf(std::string testURI, unsigned int i,
        float fillMultiplier=1, float voidMultiplier=1, std::string occupancyStr="05")
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilNSparseHost"+occupancyStr);

    testStencilHeatSparseHost_perf<blockEdgeSize, gridEdgeSize,
            SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize, long int>>(i, base,
                    fillMultiplier, voidMultiplier);
    hipDeviceSynchronize();
}

template<unsigned int blockEdgeSize, unsigned int gridEdgeSize>
void launch_testStencilHeatHost_perf(std::string testURI, unsigned int i)
{
    constexpr unsigned int dim = 2;
    typedef aggregate<float,float> AggregateT;
    constexpr unsigned int chunkSize = IntPow<blockEdgeSize,dim>::value;

    std::string base(testURI + "(" + std::to_string(i) + ")");
    report_sparsegrid_funcs.graphs.put(base + ".test.name","StencilN_Host");

    testStencilHeatHost_perf<blockEdgeSize, gridEdgeSize,
        SparseGridGpu<dim, AggregateT, blockEdgeSize, chunkSize>>(i, base);
}

BOOST_AUTO_TEST_SUITE(performance)

BOOST_AUTO_TEST_SUITE(SparseGridGpu_test)

BOOST_AUTO_TEST_CASE(testStencilHeatHost_gridScaling)
{
   std::string testURI = suiteURI + ".host.stencil.dense.N.2D.gridScaling";
   unsigned int counter = 0;
   launch_testStencilHeatHost_perf<8, 128>(testURI, counter++);
   launch_testStencilHeatHost_perf<8, 256>(testURI, counter++);
   launch_testStencilHeatHost_perf<8, 512>(testURI, counter++);
//   launch_testStencilHeatHost_perf<8, 1024>(testURI, counter++);

   testSet.insert(testURI);
}
BOOST_AUTO_TEST_CASE(testStencilHeatHost_blockScaling)
{
   std::string testURI = suiteURI + ".host.stencil.dense.N.2D.blockScaling";
   unsigned int counter = 0;
   launch_testStencilHeatHost_perf<4, 2048>(testURI, counter++);
   launch_testStencilHeatHost_perf<8, 1024>(testURI, counter++);
   launch_testStencilHeatHost_perf<16, 512>(testURI, counter++);
//   launch_testStencilHeatHost_perf<32, 256>(testURI, counter++);

   testSet.insert(testURI);
}

BOOST_AUTO_TEST_CASE(testStencilHeatSparseHost_gridScaling)
{
   std::string testURI = suiteURI + ".host.stencil.sparse.N.2D.05.gridScaling";
   unsigned int counter = 0;
   constexpr unsigned int blockSize = 32;
   constexpr unsigned int __referenceBlockSize = 8;
   constexpr unsigned int __bsFactor = blockSize / __referenceBlockSize;
   launch_testStencilHeatSparseHost_perf<blockSize, 128/__bsFactor>(testURI, counter++, 1.45, 1, "05");
   launch_testStencilHeatSparseHost_perf<blockSize, 256/__bsFactor>(testURI, counter++, 1.45, 1, "05");
   launch_testStencilHeatSparseHost_perf<blockSize, 512/__bsFactor>(testURI, counter++, 1.45, 1, "05");
   launch_testStencilHeatSparseHost_perf<blockSize, 1024/__bsFactor>(testURI, counter++, 1.45, 1, "05");

   testSet.insert(testURI);
}
BOOST_AUTO_TEST_CASE(testStencilHeatSparseHost_blockScaling)
{
   std::string testURI = suiteURI + ".host.stencil.sparse.N.2D.05.blockScaling";
   unsigned int counter = 0;
   launch_testStencilHeatSparseHost_perf<4, 2048>(testURI, counter++, 1.45, 1, "05");
   launch_testStencilHeatSparseHost_perf<8, 1024>(testURI, counter++, 1.45, 1, "05");
   launch_testStencilHeatSparseHost_perf<16, 512>(testURI, counter++, 1.45, 1, "05");
   launch_testStencilHeatSparseHost_perf<32, 256>(testURI, counter++, 1.45, 1, "05");

   testSet.insert(testURI);
}

BOOST_AUTO_TEST_SUITE_END()

BOOST_AUTO_TEST_SUITE_END()

